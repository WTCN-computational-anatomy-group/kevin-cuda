#include "hip/hip_runtime.h"


/*
 * This CUDA-Cusparse code can handle/work with  any type of the input mxArrays, 
 * GPUarray or standard matlab CPU array as input {prhs[0]/prhs[1] := mxGPUArray or CPU Array}[double/complex double]
 * Sparse/Dense vector-sparse/dense vector addition   Z=CuMatlab_addV(Sparse/Dense(X),Sparse/Dense(Y), alpha).
 * Z= alpha*X+Y
 * Developed at UCL, Institute of Neurology, 12 Queen Square, WC1N 3AR, London
 * Wellcome Trust Centre for Neuroimaging
 * Part of the project SPM(http://www.fil.ion.ucl.ac.uk/spm)
 * Copyright 2018
 * Kevin Bronik
 */

#include "matrix.h"
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <hipsparse.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "SPARSEHELPER.h"
#include "ERRORCHK.h"
#include <omp.h>

// Input Arguments
#define	INPUTSPARSEA   prhs[0]
#define	INPUTDENSEB   prhs[1]
#define	ALPHA   prhs[2]
//#define	BETA    prhs[3]
// Output Arguments
#define	OUTPUTMATRIX  plhs[0]



  
    
extern "C" static void mexCuMatlab_sparseSDC(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
 int nDevices;
hipError_t errCode =hipGetDeviceCount(&nDevices); 
//int nDevices;
//hipGetDeviceCount(&nDevices);

if (errCode != hipSuccess){
printf("Error! No CUDA devices found! \n");
return;
}

    char const * const InputErrMsg = "Invalid input to MEX file, number of input arguments must be three.";
    char const * const OutputErrMsg = "Invalid output to MEX file, number of output arguments must be one.";
   if ((nrhs!=3)) {
        mexErrMsgIdAndTxt("MATLAB:mexatexit:invalidInput", InputErrMsg);
    }
   if ((nlhs!=1)) {
        mexErrMsgIdAndTxt("MATLAB:mexatexit:invalidInput", OutputErrMsg);
    }
 char *input_buf0;
 input_buf0 = mxArrayToString(INPUTSPARSEA);

      if ((mxIsChar(INPUTSPARSEA))){
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Input(FIRST ARGUMENT) must be array, or gpuArray object not  %s\n",input_buf0);
    }
    
 char *input_buf1;
 input_buf1 = mxArrayToString(INPUTDENSEB);

      if ((mxIsChar(INPUTDENSEB))){
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Input(SECOND ARGUMENT) must be array, or gpuArray object not  %s\n",input_buf1);
    } 
 char *input_buf2;
 input_buf2 = mxArrayToString(ALPHA);

      if ((mxIsChar(ALPHA))){
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Input(THIRD ARGUMENT) must be scalar not  %s\n",input_buf2);
    } 

if (mxIsGPUArray(INPUTSPARSEA) && mxIsGPUArray(INPUTDENSEB)) {
    
    mxGPUArray const *INPUTSPARSEGPUA;
    mxGPUArray const *INPUTDENSEGPUB;
    
    
    /* Initialize the MathWorks GPU API. */
    mxInitGPU();
    INPUTSPARSEGPUA = mxGPUCreateFromMxArray(INPUTSPARSEA);
    INPUTDENSEGPUB = mxGPUCreateFromMxArray(INPUTDENSEB);
    
   
	
    if((mxGPUIsSparse(INPUTSPARSEGPUA))&& (!mxGPUIsSparse(INPUTDENSEGPUB)) ){
        
    const mwSize *dimsGPUSA;
    dimsGPUSA=mxGPUGetDimensions(INPUTSPARSEGPUA);
    int numARows, numAColumns;
    numARows = (int)dimsGPUSA[0]; /* gets number of rows of A */
    numAColumns = (int)dimsGPUSA[1]; /* gets number of columns of A */
    
    const mwSize *dimsGPUSB;
    dimsGPUSB=mxGPUGetDimensions(INPUTDENSEGPUB);
    int numBRows, numBColumns;
    numBRows = (int)dimsGPUSB[0]; /* gets number of rows of B */
    numBColumns = (int)dimsGPUSB[1]; /* gets number of columns of B */
   if ( (((numARows!= 1) && (numAColumns!= 1))) ||(((numBRows!= 1) && (numBColumns!= 1)))) {
              mxGPUDestroyGPUArray(INPUTSPARSEGPUA);
              mxGPUDestroyGPUArray(INPUTDENSEGPUB);   
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                        "Invalid input to MEX file, first/second arguments must be sparse/dense column vector.");
             
    }
    
     if ( mxGPUGetNumberOfElements(INPUTSPARSEGPUA)!=mxGPUGetNumberOfElements(INPUTDENSEGPUB)) {
              mxGPUDestroyGPUArray(INPUTSPARSEGPUA);
              mxGPUDestroyGPUArray(INPUTDENSEGPUB);  
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                        "Invalid input to MEX file, row number of sparse vector (first argument) must be equal to row number of dense vector(second argument).");
             
    }
     if ( (numARows!=numBRows)&& (numAColumns!=numBColumns)  ) {
              mxGPUDestroyGPUArray(INPUTSPARSEGPUA);
              mxGPUDestroyGPUArray(INPUTDENSEGPUB);    
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                        "Invalid input to MEX file, sparse vector (first argument) and  dense vector(second argument) must be both row or column vectors.");
             
    }
       mxComplexDouble*  al= (mxComplexDouble *)mxGetComplexDoubles(ALPHA);
       const hipDoubleComplex alpha = make_hipDoubleComplex(al[0].real, al[0].imag);
      
    
    mwIndex nnz1;
    
    mxArray * tempx = mxGPUCreateMxArrayOnCPU(INPUTSPARSEGPUA);
    nnz1 = *(mxGetJc(tempx) + numAColumns);
     //nnz1=(mwSize)ceil(numARows*numAColumns);
   int nnz= static_cast<int> (nnz1);
    int *pointerrow =0;
    mxArray *row_sort;
   if (numAColumns == 1) {
    row_sort =mxCreateNumericMatrix(nnz, 1, mxINT32_CLASS, mxREAL);
    pointerrow = (int *)mxGetInt32s(row_sort);
   
    Ir_DataGetSetIXY(tempx , pointerrow, nnz);
    }
    
   if (numARows == 1) {

   
    row_sort =mxCreateNumericMatrix(nnz, 1, mxINT32_CLASS, mxREAL);
    pointerrow = (int *)mxGetInt32s(row_sort);
    
    Jc_GetSetIXY(tempx , pointerrow);
   
        } 
   
    hipDoubleComplex  *pointerval = (hipDoubleComplex *)mxGetComplexDoubles(tempx);
            
   size_t pivot_dimensionsrow[1] = {nnz};
   
   size_t pivot_dimensionsvalue[1] = {nnz};    
      mxGPUArray *row_sortA = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionsrow, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);  
          
        int *xrow_sortA=(int *)mxGPUGetData(row_sortA);
 gpuErrchk(hipMemcpy(xrow_sortA, pointerrow, nnz * sizeof(*xrow_sortA), hipMemcpyHostToDevice));
       
      mxGPUArray *val_sortA = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionsvalue, mxDOUBLE_CLASS, mxCOMPLEX, MX_GPU_DO_NOT_INITIALIZE);  
          
        hipDoubleComplex *xval_sortA=(hipDoubleComplex*)mxGPUGetData(val_sortA);
 gpuErrchk(hipMemcpy(xval_sortA, pointerval, nnz * sizeof(*xval_sortA), hipMemcpyHostToDevice));    
          
    
    hipDoubleComplex const *d_B_dense;
   d_B_dense = (hipDoubleComplex const *)(mxGPUGetDataReadOnly(INPUTDENSEGPUB));
    
        mxGPUDestroyGPUArray(INPUTSPARSEGPUA);
        mxGPUDestroyGPUArray(INPUTDENSEGPUB);
        mxDestroyArray(row_sort);
        mxDestroyArray(tempx);  
    hipsparseHandle_t handle;    cusparseSafeCall(hipsparseCreate(&handle));

	hipsparseMatDescr_t descrA;      cusparseSafeCall(hipsparseCreateMatDescr(&descrA));
	hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE);

    

    hipDoubleComplex  *VALOUT;
    mxGPUArray *VAL;
if (numBColumns == 1) {	
    
    size_t pivot_dimensionsvalueV[1] = {numARows};

    VAL = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionsvalueV, mxDOUBLE_CLASS, mxCOMPLEX, MX_GPU_DO_NOT_INITIALIZE);
    VALOUT = (hipDoubleComplex *)mxGPUGetData(VAL);
       gpuErrchk(hipMemcpy(VALOUT, d_B_dense, sizeof(hipDoubleComplex) * numARows , hipMemcpyDeviceToDevice));
    }  

if (numBRows == 1) {
	
   size_t  pivot_dimensionsvalueV[2] = {1,numBColumns};
    VAL = mxGPUCreateGPUArray(2, (mwSize*) pivot_dimensionsvalueV, mxDOUBLE_CLASS, mxCOMPLEX, MX_GPU_DO_NOT_INITIALIZE);
    VALOUT = (hipDoubleComplex *)mxGPUGetData(VAL);
       gpuErrchk(hipMemcpy(VALOUT, d_B_dense, sizeof(hipDoubleComplex) * numBColumns , hipMemcpyDeviceToDevice));
   
}       
    
  cusparseSafeCall(hipsparseZaxpyi( handle,  nnz, 
               &alpha, 
               xval_sortA, xrow_sortA, 
               VALOUT, HIPSPARSE_INDEX_BASE_ONE));
               
 
        mxGPUDestroyGPUArray(row_sortA);
        mxGPUDestroyGPUArray(val_sortA);

               
  OUTPUTMATRIX = mxGPUCreateMxArrayOnGPU(VAL);             

       
        mxGPUDestroyGPUArray(VAL);
  
        hipsparseDestroyMatDescr(descrA);   
		hipsparseDestroy(handle);   
    
        }
    
        else{
        mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Incorrect input arguments! %s\n");    
        }
    
   }
     
////////////////////////////////////////////////////////////////////////////////////  
    else if (!(mxIsGPUArray(INPUTSPARSEA)) && !(mxIsGPUArray(INPUTDENSEB))){
   
     // if ((mxGetClassID(INPUTSPARSEA) != mxDOUBLE_CLASS) || (mxGetClassID(INPUTSPARSEB) != mxDOUBLE_CLASS)) {
       //  mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
           //     "Invalid input to MEX file, input(FIRST and SECOND  ARGUMENTS) must be  hipDoubleComplex precision.");
             
   // }
    if((mxIsSparse(INPUTSPARSEA))&& (!mxIsSparse(INPUTDENSEB)) ){
    
     mxInitGPU();
    const mwSize *dimsCPUA;
    dimsCPUA=mxGetDimensions(INPUTSPARSEA);
    
    int  numARows = (int)dimsCPUA[0]; /* gets number of rows of A */
    int  numAColumns = (int)dimsCPUA[1]; /* gets number of columns of A */
   
    const mwSize *dimsCPUB;
    dimsCPUB=mxGetDimensions(INPUTDENSEB);
    
    int  numBRows = (int)dimsCPUB[0]; /* gets number of rows of B */
    int  numBColumns = (int)dimsCPUB[1]; /* gets number of columns of B */
   if ( (((numARows!= 1) && (numAColumns!= 1))) ||(((numBRows!= 1) && (numBColumns!= 1)))) {
       
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                        "Invalid input to MEX file, first/second arguments must be sparse/dense column vector.");
             
    }
    
     if ( mxGetNumberOfElements(INPUTSPARSEA)!=mxGetNumberOfElements(INPUTDENSEB)) {
  
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                        "Invalid input to MEX file, row number of sparse vector (first argument) must be equal to row number of dense vector(second argument).");
             
    }
     if ( (numARows!=numBRows)&& (numAColumns!=numBColumns)  ) {
     
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                        "Invalid input to MEX file, sparse vector (first argument) and  dense vector(second argument) must be both row or column vectors.");
             
    }
       mxComplexDouble*  al= (mxComplexDouble *)mxGetComplexDoubles(ALPHA);
       const hipDoubleComplex alpha = make_hipDoubleComplex(al[0].real, al[0].imag);
    
    mwIndex nnz1;
 
    nnz1 = *(mxGetJc(INPUTSPARSEA) + numAColumns);
    int nnz= static_cast<int> (nnz1);
    
    int *pointerrow =0;
    mxArray *row_sort;
   if (numAColumns == 1) {
    row_sort =mxCreateNumericMatrix(nnz, 1, mxINT32_CLASS, mxREAL);
    pointerrow = (int *)mxGetInt32s(row_sort);
   
    Ir_DataGetSetIXY(INPUTSPARSEA , pointerrow, nnz);
    
    }
    
   if (numARows == 1) {

   
    row_sort =mxCreateNumericMatrix(nnz, 1, mxINT32_CLASS, mxREAL);
    pointerrow = (int *)mxGetInt32s(row_sort);
    
    Jc_GetSetIXY(INPUTSPARSEA , pointerrow);
   
        }
   
    hipDoubleComplex  *pointerval = (hipDoubleComplex *)mxGetComplexDoubles(INPUTSPARSEA);
            
   size_t pivot_dimensionsrow[1] = {nnz};
   
   size_t pivot_dimensionsvalue[1] = {nnz};    
      mxGPUArray *row_sortA = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionsrow, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);  
          
        int *xrow_sortA=(int *)mxGPUGetData(row_sortA);
 gpuErrchk(hipMemcpy(xrow_sortA, pointerrow, nnz * sizeof(*xrow_sortA), hipMemcpyHostToDevice));
       
      mxGPUArray *val_sortA = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionsvalue, mxDOUBLE_CLASS, mxCOMPLEX, MX_GPU_DO_NOT_INITIALIZE);  
          
        hipDoubleComplex *xval_sortA=(hipDoubleComplex*)mxGPUGetData(val_sortA);
 gpuErrchk(hipMemcpy(xval_sortA, pointerval, nnz * sizeof(*xval_sortA), hipMemcpyHostToDevice));    
   
           
    hipsparseHandle_t handle;    cusparseSafeCall(hipsparseCreate(&handle));

	hipsparseMatDescr_t descrA;      cusparseSafeCall(hipsparseCreateMatDescr(&descrA));
	hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE);

   hipDoubleComplex *h_B_dense1;
   h_B_dense1 = (hipDoubleComplex *)mxGetComplexDoubles(INPUTDENSEB);
	

   hipDoubleComplex  *VALOUT;
   mxGPUArray *VAL;
if (numBColumns == 1) {	
    
   size_t pivot_dimensionsvalueV[1] = {numARows};

    VAL = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionsvalueV, mxDOUBLE_CLASS, mxCOMPLEX, MX_GPU_DO_NOT_INITIALIZE);
    VALOUT = (hipDoubleComplex *)mxGPUGetData(VAL);
       gpuErrchk(hipMemcpy(VALOUT, h_B_dense1, sizeof(hipDoubleComplex) * numARows , hipMemcpyHostToDevice));
    }  

if (numBRows == 1) {
	
   size_t  pivot_dimensionsvalueV[2] = {1,numBColumns};
    VAL = mxGPUCreateGPUArray(2, (mwSize*) pivot_dimensionsvalueV, mxDOUBLE_CLASS, mxCOMPLEX, MX_GPU_DO_NOT_INITIALIZE);
    VALOUT = (hipDoubleComplex *)mxGPUGetData(VAL);
       gpuErrchk(hipMemcpy(VALOUT, h_B_dense1, sizeof(hipDoubleComplex) * numBColumns , hipMemcpyHostToDevice));
   
}      
    
  cusparseSafeCall(hipsparseZaxpyi( handle,  nnz, 
               &alpha, 
               xval_sortA, xrow_sortA, 
               VALOUT, HIPSPARSE_INDEX_BASE_ONE));
                
        mxGPUDestroyGPUArray(row_sortA);
        mxGPUDestroyGPUArray(val_sortA);
        mxDestroyArray(row_sort);
       
               
  OUTPUTMATRIX = mxGPUCreateMxArrayOnGPU(VAL);             

       
        mxGPUDestroyGPUArray(VAL);
  
        hipsparseDestroyMatDescr(descrA);   
		hipsparseDestroy(handle);
    
	
    }
    else{
        mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Incorrect input arguments! %s\n");    
        }
    
 }
        //
    else{
        mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Incorrect input arguments! %s\n");    
        }

}
