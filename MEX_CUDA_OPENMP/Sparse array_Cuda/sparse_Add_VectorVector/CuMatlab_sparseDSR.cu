

/*
 * This CUDA-Cusparse code can handle/work with  any type of the input mxArrays, 
 * GPUarray or standard matlab CPU array as input {prhs[0]/prhs[1] := mxGPUArray or CPU Array}[double/complex double]
 * Sparse/Dense vector-sparse/dense vector addition   Z=CuMatlab_addV(Sparse/Dense(X),Sparse/Dense(Y), alpha).
 * Z= alpha*X+Y
 * Developed at UCL, Institute of Neurology, 12 Queen Square, WC1N 3AR, London
 * Wellcome Trust Centre for Neuroimaging
 * Part of the project SPM(http://www.fil.ion.ucl.ac.uk/spm)
 * Copyright 2018
 * Kevin Bronik
 */

#include "matrix.h"
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <hipsparse.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "SPARSEHELPER.h"
#include "ERRORCHK.h"
#include <omp.h>

// Input Arguments
#define	INPUTDENSEA   prhs[0]
#define	INPUTSPARSEB   prhs[1]
#define	ALPHA   prhs[2]
//#define	BETA    prhs[3]
// Output Arguments
#define	OUTPUTMATRIX  plhs[0]



  
    
extern "C" static void mexCuMatlab_sparseDSR(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
 int nDevices;
hipError_t errCode =hipGetDeviceCount(&nDevices); 
//int nDevices;
//hipGetDeviceCount(&nDevices);

if (errCode != hipSuccess){
printf("Error! No CUDA devices found! \n");
return;
}

    char const * const InputErrMsg = "Invalid input to MEX file, number of input arguments must be three.";
    char const * const OutputErrMsg = "Invalid output to MEX file, number of output arguments must be one.";
   if ((nrhs!=3)) {
        mexErrMsgIdAndTxt("MATLAB:mexatexit:invalidInput", InputErrMsg);
    }
   if ((nlhs!=1)) {
        mexErrMsgIdAndTxt("MATLAB:mexatexit:invalidInput", OutputErrMsg);
    }
 char *input_buf0;
 input_buf0 = mxArrayToString(INPUTDENSEA);

      if ((mxIsChar(INPUTDENSEA))){
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Input(FIRST ARGUMENT) must be array, or gpuArray object not  %s\n",input_buf0);
    }
    
 char *input_buf1;
 input_buf1 = mxArrayToString(INPUTSPARSEB);

      if ((mxIsChar(INPUTSPARSEB))){
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Input(SECOND ARGUMENT) must be array, or gpuArray object not  %s\n",input_buf1);
    } 

 char *input_buf2;
 input_buf2 = mxArrayToString(ALPHA);

      if ((mxIsChar(ALPHA))){
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Input(THIRD ARGUMENT) must be scalar not  %s\n",input_buf2);
    } 


if (mxIsGPUArray(INPUTDENSEA) && mxIsGPUArray(INPUTSPARSEB)) {
    
    mxGPUArray const *INPUTDENSEGPUA;
    mxGPUArray const *INPUTSPARSEGPUB;
    
    
    /* Initialize the MathWorks GPU API. */
    mxInitGPU();
    INPUTDENSEGPUA = mxGPUCreateFromMxArray(INPUTDENSEA);
    INPUTSPARSEGPUB = mxGPUCreateFromMxArray(INPUTSPARSEB);
    
   
	
    if((!mxGPUIsSparse(INPUTDENSEGPUA))&& (mxGPUIsSparse(INPUTSPARSEGPUB)) ){
        
    const mwSize *dimsGPUSA;
    dimsGPUSA=mxGPUGetDimensions(INPUTDENSEGPUA);
    int numARows, numAColumns;
    numARows = (int)dimsGPUSA[0]; /* gets number of rows of A */
    numAColumns = (int)dimsGPUSA[1]; /* gets number of columns of A */
    
    const mwSize *dimsGPUSB;
    dimsGPUSB=mxGPUGetDimensions(INPUTSPARSEGPUB);
    int numBRows, numBColumns;
    numBRows = (int)dimsGPUSB[0]; /* gets number of rows of B */
    numBColumns = (int)dimsGPUSB[1]; /* gets number of columns of B */
    if ( (((numARows!= 1) && (numAColumns!= 1))) ||(((numBRows!= 1) && (numBColumns!= 1)))) {
              mxGPUDestroyGPUArray(INPUTDENSEGPUA);
              mxGPUDestroyGPUArray(INPUTSPARSEGPUB);   
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                        "Invalid input to MEX file, first/second arguments must be dense/sparse column vector.");
             
    }
    
     if ( mxGPUGetNumberOfElements(INPUTDENSEGPUA)!=mxGPUGetNumberOfElements(INPUTSPARSEGPUB)) {
              mxGPUDestroyGPUArray(INPUTDENSEGPUA);
              mxGPUDestroyGPUArray(INPUTSPARSEGPUB);    
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                        "Invalid input to MEX file, row number of dense vector(first argument) must be equal to row numbers of sparse vector(second argument).");
             
    }
     if ( (numARows!=numBRows)&& (numAColumns!=numBColumns)  ) {
              mxGPUDestroyGPUArray(INPUTDENSEGPUA);
              mxGPUDestroyGPUArray(INPUTSPARSEGPUB);    
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                        "Invalid input to MEX file, dense vector (first argument) and  sparse vector(second argument) must be both row or column vectors.");
             
    } 	
	
	
      const  double alpha= mxGetScalar(ALPHA);
      
  
    mwIndex nnz2;
    mxArray * tempx = mxGPUCreateMxArrayOnCPU(INPUTSPARSEGPUB);
    nnz2 = *(mxGetJc(tempx) + numBColumns);
    
   int nnz= static_cast<int> (nnz2); 
    int *pointerrow =0;
    mxArray *row_sort;
   if (numBColumns == 1) {
    row_sort =mxCreateNumericMatrix(nnz, 1, mxINT32_CLASS, mxREAL);
    pointerrow = (int *)mxGetInt32s(row_sort);
   
    Ir_DataGetSetIXY(tempx , pointerrow, nnz);
    }
    
   if (numBRows == 1) {

   
    row_sort =mxCreateNumericMatrix(nnz, 1, mxINT32_CLASS, mxREAL);
    pointerrow = (int *)mxGetInt32s(row_sort);
    
    Jc_GetSetIXY(tempx , pointerrow);
   
        }
   
    double  *pointerval = (double *)mxGetDoubles(tempx);
            
   size_t pivot_dimensionsrow[1] = {nnz};
   
   size_t pivot_dimensionsvalue[1] = {nnz};    
      mxGPUArray *row_sortA = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionsrow, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);  
          
        int *xrow_sortA=(int *)mxGPUGetData(row_sortA);
 gpuErrchk(hipMemcpy(xrow_sortA, pointerrow, nnz * sizeof(*xrow_sortA), hipMemcpyHostToDevice));
       
      mxGPUArray *val_sortA = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionsvalue, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);  
          
        double *xval_sortA=(double*)mxGPUGetData(val_sortA);
 gpuErrchk(hipMemcpy(xval_sortA, pointerval, nnz * sizeof(*xval_sortA), hipMemcpyHostToDevice));    
   
       
     hipsparseHandle_t handle;    cusparseSafeCall(hipsparseCreate(&handle));

	hipsparseMatDescr_t descrA;      cusparseSafeCall(hipsparseCreateMatDescr(&descrA));
	hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE);        
    
   double const *d_A_dense;
   d_A_dense = (double const *)(mxGPUGetDataReadOnly(INPUTDENSEGPUA));  
    
              mxGPUDestroyGPUArray(INPUTDENSEGPUA);
              mxGPUDestroyGPUArray(INPUTSPARSEGPUB); 
              mxDestroyArray(row_sort);
              mxDestroyArray(tempx); 

   double  *VALOUT=0;
   mxGPUArray *VAL;
if (numAColumns == 1) {	
    
    size_t pivot_dimensionsvalueV[1] = {numARows};

    VAL = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionsvalueV, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    VALOUT = (double *)mxGPUGetData(VAL);
       gpuErrchk(hipMemcpy(VALOUT, d_A_dense, sizeof(double) * numARows , hipMemcpyDeviceToDevice));
    }  

if (numARows == 1) {
	
   size_t  pivot_dimensionsvalueV[2] = {1,numBColumns};
    VAL = mxGPUCreateGPUArray(2, (mwSize*) pivot_dimensionsvalueV, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    VALOUT = (double *)mxGPUGetData(VAL);
       gpuErrchk(hipMemcpy(VALOUT, d_A_dense, sizeof(double) * numBColumns , hipMemcpyDeviceToDevice));
   
}	     
    
  cusparseSafeCall(hipsparseDaxpyi( handle,  nnz, 
               &alpha, 
               xval_sortA, xrow_sortA, 
               VALOUT, HIPSPARSE_INDEX_BASE_ONE));
               
 
        mxGPUDestroyGPUArray(row_sortA);
        mxGPUDestroyGPUArray(val_sortA);

               
  OUTPUTMATRIX = mxGPUCreateMxArrayOnGPU(VAL);             

       
        mxGPUDestroyGPUArray(VAL);
  
        hipsparseDestroyMatDescr(descrA);   
		hipsparseDestroy(handle);
        
        }
    
        else{
        mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Incorrect input arguments! %s\n");    
        }
    
   }
     
////////////////////////////////////////////////////////////////////////////////////  
    else if (!(mxIsGPUArray(INPUTDENSEA)) && !(mxIsGPUArray(INPUTSPARSEB))){
   
     // if ((mxGetClassID(INPUTSPARSEA) != mxDOUBLE_CLASS) || (mxGetClassID(INPUTSPARSEB) != mxDOUBLE_CLASS)) {
       //  mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
           //     "Invalid input to MEX file, input(FIRST and SECOND  ARGUMENTS) must be  double precision.");
             
   // }
    if((!mxIsSparse(INPUTDENSEA))&& (mxIsSparse(INPUTSPARSEB)) ){
    
     mxInitGPU();
    const mwSize *dimsCPUA;
    dimsCPUA=mxGetDimensions(INPUTDENSEA);
    
    int  numARows = (int)dimsCPUA[0]; /* gets number of rows of A */
    int  numAColumns = (int)dimsCPUA[1]; /* gets number of columns of A */
   
    const mwSize *dimsCPUB;
    dimsCPUB=mxGetDimensions(INPUTSPARSEB);
    
    int  numBRows = (int)dimsCPUB[0]; /* gets number of rows of B */
    int  numBColumns = (int)dimsCPUB[1]; /* gets number of columns of B */
    if ( (((numARows!= 1) && (numAColumns!= 1))) ||(((numBRows!= 1) && (numBColumns!= 1)))) {
   
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                        "Invalid input to MEX file, first/second arguments must be dense/sparse column vector.");
             
    }
    
     if ( mxGetNumberOfElements(INPUTDENSEA)!=mxGetNumberOfElements(INPUTSPARSEB)) {
    
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                        "Invalid input to MEX file, row number of dense vector(first argument) must be equal to row numbers of sparse vector(second argument).");
             
    }
     if ( (numARows!=numBRows)&& (numAColumns!=numBColumns)  ) {
    
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                        "Invalid input to MEX file, dense vector (first argument) and  sparse vector(second argument) must be both row or column vectors.");
             
    } 	
    
      const  double alpha= mxGetScalar(ALPHA);
   
    
	mwIndex nnz2;
 
    nnz2 = *(mxGetJc(INPUTSPARSEB) + numBColumns);
    int nnz= static_cast<int> (nnz2);
    
    int *pointerrow =0;
    mxArray *row_sort;
   if (numBColumns == 1) {
    row_sort =mxCreateNumericMatrix(nnz, 1, mxINT32_CLASS, mxREAL);
    pointerrow = (int *)mxGetInt32s(row_sort);
   
    Ir_DataGetSetIXY(INPUTSPARSEB , pointerrow, nnz);
    
    }
    
   if (numBRows == 1) {

   
    row_sort =mxCreateNumericMatrix(nnz, 1, mxINT32_CLASS, mxREAL);
    pointerrow = (int *)mxGetInt32s(row_sort);
    
    Jc_GetSetIXY(INPUTSPARSEB , pointerrow);
   
        }
   
    double  *pointerval = (double *)mxGetDoubles(INPUTSPARSEB);
            
   size_t pivot_dimensionsrow[1] = {nnz};
   
   size_t pivot_dimensionsvalue[1] = {nnz};    
      mxGPUArray *row_sortA = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionsrow, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);  
          
        int *xrow_sortA=(int *)mxGPUGetData(row_sortA);
       gpuErrchk(hipMemcpy(xrow_sortA, pointerrow, nnz * sizeof(*xrow_sortA), hipMemcpyHostToDevice));
       
      mxGPUArray *val_sortA = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionsvalue, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);  
          
        double *xval_sortA=(double*)mxGPUGetData(val_sortA);
      gpuErrchk(hipMemcpy(xval_sortA, pointerval, nnz * sizeof(*xval_sortA), hipMemcpyHostToDevice)); 
 
          
     hipsparseHandle_t handle;    cusparseSafeCall(hipsparseCreate(&handle));

	hipsparseMatDescr_t descrA;      cusparseSafeCall(hipsparseCreateMatDescr(&descrA));
	hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE);        
    
  
       double *h_A_dense1;
       h_A_dense1 = (double *)mxGetDoubles(INPUTDENSEA);
 
              mxDestroyArray(row_sort);
			  
			  
   double  *VALOUT=0;
   mxGPUArray *VAL;
if (numAColumns == 1) {	
    
  size_t   pivot_dimensionsvalueV[1] = {numARows};
  VAL = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionsvalueV, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    VALOUT = (double *)mxGPUGetData(VAL);
       gpuErrchk(hipMemcpy(VALOUT, h_A_dense1, sizeof(double) * numARows , hipMemcpyHostToDevice));
    }  

if (numARows == 1) {
	
   size_t  pivot_dimensionsvalueV[2] = {1,numBColumns};
   VAL = mxGPUCreateGPUArray(2, (mwSize*) pivot_dimensionsvalueV, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    VALOUT = (double *)mxGPUGetData(VAL);
       gpuErrchk(hipMemcpy(VALOUT, h_A_dense1, sizeof(double) * numBColumns , hipMemcpyHostToDevice));
   
}			  
			  
      
    
  cusparseSafeCall(hipsparseDaxpyi( handle,  nnz, 
               &alpha, 
               xval_sortA, xrow_sortA, 
               VALOUT, HIPSPARSE_INDEX_BASE_ONE));
               
 
        mxGPUDestroyGPUArray(row_sortA);
        mxGPUDestroyGPUArray(val_sortA);

               
  OUTPUTMATRIX = mxGPUCreateMxArrayOnGPU(VAL);             

       
        mxGPUDestroyGPUArray(VAL);
  
        hipsparseDestroyMatDescr(descrA);  
		hipsparseDestroy(handle);


    }
    else{
        mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Incorrect input arguments! %s\n");    
        }
    
 }
        //
    else{
        mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Incorrect input arguments! %s\n");    
        }

}
