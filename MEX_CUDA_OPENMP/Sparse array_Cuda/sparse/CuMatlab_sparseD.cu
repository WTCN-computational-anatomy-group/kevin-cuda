#include "hip/hip_runtime.h"

/*
 * This CUDA-Cusparse code can handle/work with  any type of the input mxArrays, 
 * GPUarray or standard matlab CPU array as input {prhs[0],prhs[1],prhs[2]  := mxGPUArray or CPU Array}[double or complex double]
 * Create sparse matrix  
 * Z=CuMatlab_sparse(X) 
 * Z=CuMatlab_sparse(X,Y)
 * Z=CuMatlab_sparse(X,Y,Z)
 * Z=CuMatlab_sparse(X,Y,Z,row,column) 
 * Z=CuMatlab_sparse(X,Y,Z,row,column,nz)
 * Developed at UCL, Institute of Neurology, 12 Queen Square, WC1N 3AR, London
 * Wellcome Trust Centre for Neuroimaging
 * Part of the project SPM(http://www.fil.ion.ucl.ac.uk/spm)
 * Copyright 2018
 * Kevin Bronik
 */

#include "matrix.h"
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <hipsparse.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <algorithm>
#include "SPARSEHELPER.h"
#include "ERRORCHK.h"
#include <omp.h>


// Input Arguments
#define	ROW      prhs[0]
#define	COLUMN   prhs[1]
#define	VALUE    prhs[2]
#define	NROWS    prhs[3]
#define	NCOLS    prhs[4]



// Output Arguments
#define	OUTPUTMATRIX   plhs[0]

 
   
    
extern "C" static void mexCuMatlab_sparseD(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
 int nDevices;
hipError_t errCode =hipGetDeviceCount(&nDevices); 
//int nDevices;
//hipGetDeviceCount(&nDevices);

if (errCode != hipSuccess){
printf("Error! No CUDA devices found! \n");
return;
}

    char const * const InputErrMsg = "Invalid input to MEX file, number of input arguments must be five.";
    char const * const OutputErrMsg = "Invalid output to MEX file, number of output arguments must be one.";
   if ((nrhs!=5)) {
        mexErrMsgIdAndTxt("MATLAB:mexatexit:invalidInput", InputErrMsg);
    }
   if ((nlhs!=1)) {
        mexErrMsgIdAndTxt("MATLAB:mexatexit:invalidInput", OutputErrMsg);
    }

 char *input_buf0;
 input_buf0 = mxArrayToString(ROW);

      if ((mxIsChar(ROW))){
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Input(FIRST ARGUMENT) must be array, or gpuArray object not  %s\n",input_buf0);
    }
    
     char *input_buf1;
 input_buf1 = mxArrayToString(COLUMN);

      if ((mxIsChar(COLUMN))){
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Input(SECOND ARGUMENT) must be array, or gpuArray object not  %s\n",input_buf1);
    }
    
     char *input_buf2;
 input_buf2 = mxArrayToString(VALUE);

      if ((mxIsChar(VALUE))){
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Input(THIRD ARGUMENT) must be array, or gpuArray object not  %s\n",input_buf2);
    }

if (mxIsGPUArray(ROW)  && mxIsGPUArray(COLUMN) && mxIsGPUArray(VALUE) ) {

     mxInitGPU();
    
   mxGPUArray const *ROWGPU;
   ROWGPU= mxGPUCreateFromMxArray(ROW);
   
    
   mxGPUArray const *COLUMNGPU;
   COLUMNGPU= mxGPUCreateFromMxArray(COLUMN);
    
   
   mxGPUArray const *VALUEGPU;
   VALUEGPU= mxGPUCreateFromMxArray(VALUE);
   
   if((mxGPUIsSparse(ROWGPU)==1)|| (mxGPUIsSparse(COLUMNGPU)==1) || (mxGPUIsSparse(VALUEGPU)==1)){
       //plhs[0] = mxGPUCreateMxArrayOnGPU(INPUTMATRIXGPUx);
       printf("Warning! Input(FIRST, SECOND and THIRD ARGUMENTS) must be non sparse! \n");  
      // mxGPUDestroyGPUArray(INPUTMATRIXGPUx);
      mxGPUDestroyGPUArray(ROWGPU);
      mxGPUDestroyGPUArray(COLUMNGPU);
      mxGPUDestroyGPUArray(VALUEGPU);
                return;
    
    }
   
   if ( (mxGPUGetClassID(ROWGPU) != mxDOUBLE_CLASS)  && (mxGPUGetClassID(COLUMNGPU) != mxDOUBLE_CLASS) && (mxGPUGetClassID(VALUEGPU) != mxDOUBLE_CLASS)) {
      mxGPUDestroyGPUArray(ROWGPU);
      mxGPUDestroyGPUArray(COLUMNGPU);
      mxGPUDestroyGPUArray(VALUEGPU);
       
       mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Invalid input to MEX file, input(FIRST, SECOND AND THIRD ARGUMENTS) must be  double precision.");
    }
    if ( (mxGPUGetComplexity(ROWGPU) != mxREAL)  || (mxGPUGetComplexity(COLUMNGPU) != mxREAL) || (mxGPUGetComplexity(VALUEGPU) != mxREAL)) {
      mxGPUDestroyGPUArray(ROWGPU);
      mxGPUDestroyGPUArray(COLUMNGPU);
      mxGPUDestroyGPUArray(VALUEGPU); 
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Invalid input to MEX file, input(FIRST, SECOND AND THIRD ARGUMENTS) must be real with no imaginary components.");
    }
    if ( !(mxIsScalar(NROWS)) || !(mxIsScalar(NCOLS))) {
      mxGPUDestroyGPUArray(ROWGPU);
      mxGPUDestroyGPUArray(COLUMNGPU);
      mxGPUDestroyGPUArray(VALUEGPU);
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Invalid input to MEX file, input (FOURTH and  FIFTH ARGUMENTS) must be scalar.");
             
    }
    int NrowsA= (int)mxGetScalar(NROWS);   
      
    int NcolsA= (int)mxGetScalar(NCOLS);
    
    int nnzR= static_cast<int> (mxGPUGetNumberOfElements(ROWGPU));
    int nnzC= static_cast<int> (mxGPUGetNumberOfElements(COLUMNGPU));
    int nnzV= static_cast<int> (mxGPUGetNumberOfElements(VALUEGPU));
    int NNZMAXA=nnzV;
    if ( (nnzR!= nnzC) || (nnzC != nnzV)|| (nnzR != nnzV) ) {
	  mxGPUDestroyGPUArray(ROWGPU);
      mxGPUDestroyGPUArray(COLUMNGPU);
      mxGPUDestroyGPUArray(VALUEGPU);
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Invalid input to MEX file, input vectors (FIRST, SECOND, THIRD ARGUMENTS) must be the same lengths.");
             
    }
   if ( nnzR>(NrowsA*NcolsA) ) {
	  mxGPUDestroyGPUArray(ROWGPU);
      mxGPUDestroyGPUArray(COLUMNGPU);
      mxGPUDestroyGPUArray(VALUEGPU);
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Invalid input to MEX file, index exceeds array bounds [number of non zero greater than matrix dimensions (row*column)].");
             
    }
    
    mxArray * RW=mxGPUCreateMxArrayOnCPU(ROWGPU);
	mxArray * CL =mxGPUCreateMxArrayOnCPU(COLUMNGPU);
    mxArray * VL=mxGPUCreateMxArrayOnCPU(VALUEGPU);
    
   // int *nrr=(int *)malloc(nnzR * sizeof(*nrr));
   // int *nrc=(int *)malloc(nnzR * sizeof(*nrc));
	    std::vector<MATRIX> vect;
        int j;
      #pragma omp for  schedule(static) nowait
         for ( j = 0; j < nnzR; ++j) {
           vect.push_back(MATRIX((static_cast<int> ((mxGetDoubles(RW))[j])),  (static_cast<int> ((mxGetDoubles(CL))[j])), (static_cast<double> ((mxGetDoubles(VL))[j])))); 
         //  nrr[i]=(static_cast<int> ((mxGetPr(RW))[i]));
         //  nrc[i]=(static_cast<int> ((mxGetPr(CL))[i]));
           }
            
  // int  Nr= *std::max_element(nrr, nrr + nnzR, max_elem);
  // int  Nc= *std::max_element(nrc, nrc + nnzR, max_elem);  
           
	std::sort(vect.begin(), vect.end());
	std::vector<MATRIX> vect_temp; 
    vect_temp= vect;
    	
    int i = 0;
	     
	std::vector<MATRIX>::iterator ity = vect.begin();
//	#pragma omp parallel for  shared(nnzR) private(i)
	for (std::vector<MATRIX>::iterator itx = vect.begin(); itx != vect.end(); itx++){
		
		
		ity = itx + 1;
		while (ity != vect.end())
		{


			//for (ity ; ity != vectx.end(); ity++){

			if (itx->row_C == ity->row_C && itx->column_C == ity->column_C){
				vect_temp[i].value_C = vect_temp[i].value_C + ity->value_C;

				vect_temp[std::distance(vect.begin(), ity)].checked = true;

			}
			ity++;
				//}
		}
		i++;
		
	}
    //    #pragma omp parallel for 
  for (auto it = vect_temp.begin(); it != vect_temp.end();) {
		if ((it->checked==true) ||(it->value_C==0)) {
			it = vect_temp.erase(it);
		}
		else {
			++it;
		}
	}
  nnzR=nnzC=nnzV= (int)vect_temp.size();
  
   mxArray * ROWx =mxCreateNumericMatrix(nnzR, 1, mxINT32_CLASS, mxREAL);
    int *h_A_RowIndices_coo = (int *)mxGetInt32s(ROWx);
    
           #pragma omp parallel for shared(nnzR) private(i)
         for (i = 0; i < nnzR; ++i) {
          // h_A_RowIndices_coo[i] = static_cast<int> ((mxGetPr(RW))[i]); 
           h_A_RowIndices_coo[i] =vect_temp[i].row_C;
            }
            
   mxArray * COLUMNx =mxCreateNumericMatrix(nnzC, 1, mxINT32_CLASS, mxREAL);
    int *h_A_ColIndices_coo = (int *)mxGetInt32s(COLUMNx);
    
           #pragma omp parallel for shared(nnzC) private(i)
         for (i = 0; i < nnzC; ++i) {
          // h_A_ColIndices_coo[i] = static_cast<int> ((mxGetPr(CL))[i]);
           h_A_ColIndices_coo[i] = vect_temp[i].column_C;
            }
    
    
   mxArray * VALUEx =mxCreateNumericMatrix(nnzV, 1, mxDOUBLE_CLASS, mxREAL);
    double *h_A1_coo = (double *)mxGetDoubles(VALUEx); 
    
         #pragma omp parallel for shared(nnzV) private(i)
         for (i = 0; i < nnzV; ++i) {
          // h_A1_coo[i] = static_cast<double> ((mxGetPr(VL))[i]); 
           h_A1_coo[i] = vect_temp[i].value_C;
            }

    int  Nr= *std::max_element(h_A_RowIndices_coo, h_A_RowIndices_coo + nnzR, max_elem);
   
    int  Nc= *std::max_element(h_A_ColIndices_coo, h_A_ColIndices_coo + nnzC, max_elem);          

   if ( (Nr>NrowsA) || (Nc>NcolsA) ) {
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Invalid input to MEX file, index exceeds array bounds: max(first vector)> fourth argument's value or max(second vector)> fifth argument's value .");
             
    }
	//NrowsA= h_A_RowIndices_coo[nnzR-1];    
       
    //NcolsA= Nc;
//////
  	hipsparseHandle_t handle;    cusparseSafeCall(hipsparseCreate(&handle));

     
	//double *d_A;            gpuErrchk(hipMalloc(&d_A, nnzV * sizeof(*d_A)));
	//int *d_A_RowIndices;    gpuErrchk(hipMalloc(&d_A_RowIndices, (NrowsA + 1) * sizeof(*d_A_RowIndices)));
	//int *d_A_ColIndices;    gpuErrchk(hipMalloc(&d_A_ColIndices, nnzV * sizeof(*d_A_ColIndices)));
	//int *d_cooRowIndA;       gpuErrchk(hipMalloc(&d_cooRowIndA, nnzV * sizeof(*d_cooRowIndA)));
	
   size_t pivot_dimensA[1] = {nnzV};
   size_t pivot_dimensROW_A[1] = {NrowsA + 1};
   size_t pivot_dimensCOL_A[1] = {nnzV};
   size_t pivot_dimensCOO_A[1] = {nnzV};
   
   mxGPUArray *A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensA, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    double  *d_A = (double *)mxGPUGetData(A);
   mxGPUArray * ROW_A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensROW_A, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_A_RowIndices = (int *)mxGPUGetData(ROW_A);
   mxGPUArray * COL_A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensCOL_A, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_A_ColIndices = (int *)mxGPUGetData(COL_A);
    mxGPUArray * COO_A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensCOO_A, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_cooRowIndA = (int *)mxGPUGetData(COO_A); 
	
	
	// --- Descriptor for sparse matrix A
	gpuErrchk(hipMemcpy(d_A, h_A1_coo, nnzV * sizeof(*d_A), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_A_ColIndices, h_A_ColIndices_coo, nnzV * sizeof(*d_A_ColIndices), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_cooRowIndA, h_A_RowIndices_coo, nnzV * sizeof(*d_cooRowIndA), hipMemcpyHostToDevice));

	hipsparseMatDescr_t descrA;      cusparseSafeCall(hipsparseCreateMatDescr(&descrA));
	hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE);

	int *Pa = NULL;
	void *pBuffera = NULL;
	size_t pBufferSizeInBytesa = 0;
	hipsparseXcoosort_bufferSizeExt(handle, NrowsA, NcolsA,
		nnzV,
		d_cooRowIndA,
		d_A_ColIndices, &pBufferSizeInBytesa);

	gpuErrchk(hipMalloc(&pBuffera, sizeof(char)*pBufferSizeInBytesa));
	gpuErrchk(hipMalloc(&Pa, sizeof(int)*nnzV));
	hipsparseCreateIdentityPermutation(handle, nnzV, Pa);
	cusparseSafeCall(hipsparseXcoosortByRow(handle, NrowsA, NcolsA,
		nnzV,
		d_cooRowIndA,
		d_A_ColIndices,
		Pa,
		pBuffera));

	cusparseSafeCall(hipsparseDgthr(handle, nnzV, d_A, d_A, Pa, HIPSPARSE_INDEX_BASE_ZERO));

	cusparseSafeCall(hipsparseXcoo2csr(handle,
		d_cooRowIndA,
		nnzV,
		NrowsA,
		d_A_RowIndices,
		HIPSPARSE_INDEX_BASE_ONE));


   size_t pivot_dimensionsrow[1] = {nnzR};
   size_t pivot_dimensionscolumn[1] = {NcolsA+1}; 
   size_t pivot_dimensionsvalue[1] = {nnzV};
   mxGPUArray * ROW_SORT1 = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionsrow, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *ROWSORT = (int *)mxGPUGetData(ROW_SORT1);
   mxGPUArray * COL_SORT1 = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionscolumn, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *COLSORT = (int *)mxGPUGetData(COL_SORT1);
    mxGPUArray *VAL_SORT1 = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionsvalue, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    double  *VALSORT = (double *)mxGPUGetData(VAL_SORT1);

	cusparseSafeCall(hipsparseDcsr2csc(handle, NrowsA, NcolsA, nnzV, d_A, d_A_RowIndices, d_A_ColIndices, VALSORT, ROWSORT, COLSORT, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ONE));

	//gpuErrchk(hipFree(d_A));
	//gpuErrchk(hipFree(d_A_RowIndices));
	//gpuErrchk(hipFree(d_A_ColIndices));
	//gpuErrchk(hipFree(d_cooRowIndA));
	
	mxGPUDestroyGPUArray(A);
    mxGPUDestroyGPUArray(ROW_A);
    mxGPUDestroyGPUArray(COL_A);
    mxGPUDestroyGPUArray(COO_A);
	
	
	
	gpuErrchk(hipFree(pBuffera));
	gpuErrchk(hipFree(Pa));
    
    
   mwSize nnzm=(mwSize)nnzV;
   OUTPUTMATRIX = mxCreateSparse(NrowsA,NcolsA,nnzm,mxREAL);
   
    
   mxArray *RS= mxGPUCreateMxArrayOnCPU(ROW_SORT1);
   int * rs= (int *)mxGetInt32s(RS);
   mxArray *CS= mxGPUCreateMxArrayOnCPU(COL_SORT1);
   int * cs= (int *)mxGetInt32s(CS);

    
      mwIndex *irs,*jcs;
  

        irs = static_cast<mwIndex *> (mxMalloc (nnzR * sizeof(mwIndex)));
       #pragma omp parallel for shared(nnzR) private(i)
         for ( i = 0; i < nnzR; ++i) {
           irs[i] = static_cast<mwIndex> (rs[i])-1; 
            }
      
      jcs = static_cast<mwIndex *> (mxMalloc ((NcolsA+1) * sizeof(mwIndex)));
        int nc1= NcolsA+1;
      #pragma omp parallel for shared(nc1) private(i)
            for (i = 0; i < nc1; ++i) {
           jcs[i] = static_cast<mwIndex> (cs[i])-1;
            }
             
           
        mxDouble* PRS = (mxDouble*) mxMalloc (nnzV * sizeof(mxDouble));
        gpuErrchk(hipMemcpy(PRS, VALSORT, nnzV * sizeof(mxDouble), hipMemcpyDeviceToHost));

           
    
        
        mxFree (mxGetJc (OUTPUTMATRIX)) ;
        mxFree (mxGetIr (OUTPUTMATRIX)) ;
        mxFree (mxGetDoubles (OUTPUTMATRIX)) ;
        mxSetNzmax(OUTPUTMATRIX, (static_cast<mwSize>(NNZMAXA)));
        mxSetIr(OUTPUTMATRIX, (mwIndex *)irs);
        mxSetJc(OUTPUTMATRIX, (mwIndex *)jcs);
        int s = mxSetDoubles(OUTPUTMATRIX, (mxDouble *)PRS);
        if ( s==0) {
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "the function is unsuccessful, either mxArray is not an unshared mxDOUBLE_CLASS array, or the data is not allocated with mxCalloc.");
             
         }
         mxDestroyArray(RS);
         mxDestroyArray(CS);

         mxDestroyArray(RW);
         mxDestroyArray(CL);
         mxDestroyArray(VL);
      mxDestroyArray(ROWx);
      mxDestroyArray(COLUMNx);
      mxDestroyArray(VALUEx);
      
      mxGPUDestroyGPUArray(ROW_SORT1);
      mxGPUDestroyGPUArray(COL_SORT1);
      mxGPUDestroyGPUArray(VAL_SORT1);
      mxGPUDestroyGPUArray(ROWGPU);
      mxGPUDestroyGPUArray(COLUMNGPU);
      mxGPUDestroyGPUArray(VALUEGPU);
      hipsparseDestroyMatDescr(descrA);	  
      hipsparseDestroy(handle);
   
   }
     
////////////////////////////////////////////////////////////////////////////////////  
    else if (!(mxIsGPUArray(ROW))  && !(mxIsGPUArray(COLUMN)) && !(mxIsGPUArray(VALUE))){

   if((mxIsSparse(ROW)) || (mxIsSparse(COLUMN))  || (mxIsSparse(VALUE))) {
    
   
       printf("Warning! Input(FIRST ARGUMENT) must be non sparse!\n");   
                return;
        
    } 
            
    if ( !(mxIsScalar(NROWS)) || !(mxIsScalar(NCOLS))) {
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Invalid input to MEX file, input (FOURTH and  FIFTH ARGUMENTS) must be scalar.");
             
    }       
            
    int NrowsA= (int)mxGetScalar(NROWS);   
      
    int NcolsA= (int)mxGetScalar(NCOLS);
    
    int nnzR= (int)mxGetNumberOfElements(ROW);
    int nnzC= (int)mxGetNumberOfElements(COLUMN);
    int nnzV= (int)mxGetNumberOfElements(VALUE);
    int NNZMAXA=nnzV;
   if ( (nnzR!= nnzC) || (nnzC != nnzV)|| (nnzR != nnzV) ) {
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Invalid input to MEX file, input vectors (FIRST, SECOND, THIRD ARGUMENTS) must be the same lengths.");
             
    }  
    if ( nnzR>(NrowsA*NcolsA) ) {
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Invalid input to MEX file, index exceeds array bounds [number of non zero greater than matrix dimensions (row*column)].");
             
    }
   if ( (mxGetClassID(ROW) != mxDOUBLE_CLASS) || (mxGetClassID(COLUMN) != mxDOUBLE_CLASS)||(mxGetClassID(VALUE) != mxDOUBLE_CLASS)) {
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Invalid input to MEX file, input(FIRST, SECOND AND THIRD ARGUMENTS) must be  double precision.");
             
    }
    if ( (mxIsComplex(ROW))  || (mxIsComplex(COLUMN)) || (mxIsComplex(VALUE))) {
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Invalid input to MEX file, input(FIRST, SECOND AND THIRD ARGUMENTS) must be real with no imaginary components.");
    } 
    
    mxInitGPU();

    
    //int *nrr=(int *)malloc(nnzR * sizeof(*nrr));
   // int *nrc=(int *)malloc(nnzR * sizeof(*nrc));
    
    std::vector<MATRIX> vect;
         int j;
	#pragma omp for schedule(static) nowait
         for ( j = 0; j < nnzR; ++j) {
           vect.push_back(MATRIX((static_cast<int> ((mxGetDoubles(ROW))[j])),  (static_cast<int> ((mxGetDoubles(COLUMN))[j])), (static_cast<double> ((mxGetDoubles(VALUE))[j])))); 
          // nrr[i]=(static_cast<int> ((mxGetPr(ROW))[i]));
         //  nrc[i]=(static_cast<int> ((mxGetPr(COLUMN))[i]));
            }
            
  // int  Nr= *std::max_element(nrr, nrr + nnzR, max_elem);
   
  // int  Nc= *std::max_element(nrc, nrc + nnzR, max_elem);  

   
            
	std::sort(vect.begin(), vect.end());
	std::vector<MATRIX> vect_temp; 
    vect_temp= vect;
    	
    int i = 0;
	     
	std::vector<MATRIX>::iterator ity = vect.begin();
	//#pragma omp parallel for default(none) shared(vect,vect_temp )
	for (std::vector<MATRIX>::iterator itx = vect.begin(); itx != vect.end(); itx++){
		
		
		ity = itx + 1;
		while (ity != vect.end())
		{


			//for (ity ; ity != vectx.end(); ity++){

			if (itx->row_C == ity->row_C && itx->column_C == ity->column_C){
				vect_temp[i].value_C = vect_temp[i].value_C + ity->value_C;

				vect_temp[std::distance(vect.begin(), ity)].checked = true;

			}
			ity++;
				//}
		}
		i++;
		
	}
    
   // #pragma omp parallel for default(none) shared(vect_temp)
   
  for (auto it = vect_temp.begin(); it != vect_temp.end();) {
		if ((it->checked==true) ||(it->value_C==0)) {
			it = vect_temp.erase(it);
		}
		else {
			++it;
		}
	}
  nnzR=nnzC=nnzV= (int) vect_temp.size();

    
   mxArray * ROWx =mxCreateNumericMatrix(nnzR, 1, mxINT32_CLASS, mxREAL);
    int *h_A_RowIndices_coo = (int *)mxGetInt32s(ROWx);
    
           #pragma omp parallel for shared(nnzR) private(i)
         for (i = 0; i < nnzR; ++i) {
           //h_A_RowIndices_coo[i] = static_cast<int> ((mxGetPr(ROW))[i]);
           h_A_RowIndices_coo[i] =vect_temp[i].row_C;
            }
   mxArray * COLUMNx =mxCreateNumericMatrix(nnzC, 1, mxINT32_CLASS, mxREAL);
    int *h_A_ColIndices_coo = (int *)mxGetInt32s(COLUMNx);
           #pragma omp parallel for shared(nnzC) private(i)
         for (i = 0; i < nnzC; ++i) {
           //h_A_ColIndices_coo[i] = static_cast<int> ((mxGetPr(COLUMN))[i]); 
         h_A_ColIndices_coo[i] = vect_temp[i].column_C; 
            }
    
    
   mxArray * VALUEx =mxCreateNumericMatrix(nnzV, 1, mxDOUBLE_CLASS, mxREAL);
    double *h_A1_coo = (double *)mxGetDoubles(VALUEx);    
         #pragma omp parallel for shared(nnzV) private(i)
         for ( i = 0; i < nnzV; ++i) {
           //h_A1_coo[i] =static_cast<double> ((mxGetPr(VALUE))[i]);
           h_A1_coo[i] = vect_temp[i].value_C;
            }
    
            
    int  Nr= *std::max_element(h_A_RowIndices_coo, h_A_RowIndices_coo + nnzR, max_elem);
   
    int  Nc= *std::max_element(h_A_ColIndices_coo, h_A_ColIndices_coo + nnzC, max_elem);          

   if ( (Nr>NrowsA) || (Nc>NcolsA) ) {
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Invalid input to MEX file, index exceeds array bounds: max(first vector)> fourth argument's value or max(second vector)> fifth argument's value .");
             
    }
    // NrowsA= h_A_RowIndices_coo[nnzR-1];    
       
   //  NcolsA= Nc; 
////////////
    hipsparseHandle_t handle;    cusparseSafeCall(hipsparseCreate(&handle));
	

	//double *d_A;            gpuErrchk(hipMalloc(&d_A, nnzV * sizeof(*d_A)));
	//int *d_A_RowIndices;    gpuErrchk(hipMalloc(&d_A_RowIndices, (NrowsA + 1) * sizeof(*d_A_RowIndices)));
	//int *d_A_ColIndices;    gpuErrchk(hipMalloc(&d_A_ColIndices, nnzV * sizeof(*d_A_ColIndices)));
	//int *d_cooRowIndA;       gpuErrchk(hipMalloc(&d_cooRowIndA, nnzV * sizeof(*d_cooRowIndA)));
	
   size_t pivot_dimensA[1] = {nnzV};
   size_t pivot_dimensROW_A[1] = {NrowsA + 1};
   size_t pivot_dimensCOL_A[1] = {nnzV};
   size_t pivot_dimensCOO_A[1] = {nnzV};
   
   mxGPUArray *A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensA, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    double  *d_A = (double *)mxGPUGetData(A);
   mxGPUArray * ROW_A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensROW_A, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_A_RowIndices = (int *)mxGPUGetData(ROW_A);
   mxGPUArray * COL_A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensCOL_A, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_A_ColIndices = (int *)mxGPUGetData(COL_A);
    mxGPUArray * COO_A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensCOO_A, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_cooRowIndA = (int *)mxGPUGetData(COO_A); 
	
	
	
	gpuErrchk(hipMemcpy(d_A, h_A1_coo, nnzV * sizeof(*d_A), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_A_ColIndices, h_A_ColIndices_coo, nnzV * sizeof(*d_A_ColIndices), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_cooRowIndA, h_A_RowIndices_coo, nnzV * sizeof(*d_cooRowIndA), hipMemcpyHostToDevice));


	hipsparseMatDescr_t descrA;      cusparseSafeCall(hipsparseCreateMatDescr(&descrA));
	hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE);

	int *Pa = NULL;
	void *pBuffera = NULL;
	size_t pBufferSizeInBytesa = 0;
	hipsparseXcoosort_bufferSizeExt(handle, NrowsA, NcolsA,
		nnzV,
		d_cooRowIndA,
		d_A_ColIndices, &pBufferSizeInBytesa);

	gpuErrchk(hipMalloc(&pBuffera, sizeof(char)*pBufferSizeInBytesa));
	gpuErrchk(hipMalloc(&Pa, sizeof(int)*nnzV));
	hipsparseCreateIdentityPermutation(handle, nnzV, Pa);
	cusparseSafeCall(hipsparseXcoosortByRow(handle, NrowsA, NcolsA,
		nnzV,
		d_cooRowIndA,
		d_A_ColIndices,
		Pa,
		pBuffera));

	cusparseSafeCall(hipsparseDgthr(handle, nnzV, d_A, d_A, Pa, HIPSPARSE_INDEX_BASE_ZERO));

	cusparseSafeCall(hipsparseXcoo2csr(handle,
		d_cooRowIndA,
		nnzV,
		NrowsA,
		d_A_RowIndices,
		HIPSPARSE_INDEX_BASE_ONE));

		
		
   size_t pivot_dimensionsrow[1] = {nnzR};
   size_t pivot_dimensionscolumn[1] = {NcolsA+1}; 
   size_t pivot_dimensionsvalue[1] = {nnzV};
   mxGPUArray * ROW_SORT1 = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionsrow, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *ROWSORT = (int *)mxGPUGetData(ROW_SORT1);
   mxGPUArray * COL_SORT1 = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionscolumn, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *COLSORT = (int *)mxGPUGetData(COL_SORT1);
    mxGPUArray *VAL_SORT1 = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionsvalue, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    double  *VALSORT = (double *)mxGPUGetData(VAL_SORT1);

	cusparseSafeCall(hipsparseDcsr2csc(handle, NrowsA, NcolsA, nnzV, d_A, d_A_RowIndices, d_A_ColIndices, VALSORT, ROWSORT, COLSORT, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ONE));

	//gpuErrchk(hipFree(d_A));
	//gpuErrchk(hipFree(d_A_RowIndices));
	//gpuErrchk(hipFree(d_A_ColIndices));
	//gpuErrchk(hipFree(d_cooRowIndA));
	
	mxGPUDestroyGPUArray(A);
    mxGPUDestroyGPUArray(ROW_A);
    mxGPUDestroyGPUArray(COL_A);
    mxGPUDestroyGPUArray(COO_A);
	
	
	
	gpuErrchk(hipFree(pBuffera));
	gpuErrchk(hipFree(Pa));
    
    
   mwSize nnzm=(mwSize)nnzV;
   OUTPUTMATRIX = mxCreateSparse(NrowsA,NcolsA,nnzm,mxREAL);
   
    
   mxArray *RS= mxGPUCreateMxArrayOnCPU(ROW_SORT1);
   int * rs= (int *)mxGetInt32s(RS);
   mxArray *CS= mxGPUCreateMxArrayOnCPU(COL_SORT1);
   int * cs= (int *)mxGetInt32s(CS);

    
      mwIndex *irs,*jcs;
  

        irs = static_cast<mwIndex *> (mxMalloc (nnzR * sizeof(mwIndex)));
       #pragma omp parallel for shared(nnzR) private(i)
         for ( i = 0; i < nnzR; ++i) {
           irs[i] = static_cast<mwIndex> (rs[i])-1; 
            }
      
      jcs = static_cast<mwIndex *> (mxMalloc ((NcolsA+1) * sizeof(mwIndex)));
        int nc1= NcolsA+1;
      #pragma omp parallel for shared(nc1) private(i)
            for (i = 0; i < nc1; ++i) {
           jcs[i] = static_cast<mwIndex> (cs[i])-1;
            }
             
        mxDouble* PRS = (mxDouble*) mxMalloc (nnzV * sizeof(mxDouble));
        gpuErrchk(hipMemcpy(PRS, VALSORT, nnzV * sizeof(mxDouble), hipMemcpyDeviceToHost));

           
    
        
        mxFree (mxGetJc (OUTPUTMATRIX)) ;
        mxFree (mxGetIr (OUTPUTMATRIX)) ;
        mxFree (mxGetDoubles (OUTPUTMATRIX)) ;
        mxSetNzmax(OUTPUTMATRIX, (static_cast<mwSize>(NNZMAXA)));
        mxSetIr(OUTPUTMATRIX, (mwIndex *)irs);
        mxSetJc(OUTPUTMATRIX, (mwIndex *)jcs);
        int s = mxSetDoubles(OUTPUTMATRIX, (mxDouble *)PRS);
        if ( s==0) {
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "the function is unsuccessful, either mxArray is not an unshared mxDOUBLE_CLASS array, or the data is not allocated with mxCalloc.");
             
         }
         mxDestroyArray(RS);
         mxDestroyArray(CS);

 
      mxDestroyArray(ROWx);
      mxDestroyArray(COLUMNx);
      mxDestroyArray(VALUEx);
      
      mxGPUDestroyGPUArray(ROW_SORT1);
      mxGPUDestroyGPUArray(COL_SORT1);
      mxGPUDestroyGPUArray(VAL_SORT1);
	  
      hipsparseDestroyMatDescr(descrA);
      hipsparseDestroy(handle);

               }
           
               
    else{
        mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Incorrect input arguments! %s\n");    
        }

}
