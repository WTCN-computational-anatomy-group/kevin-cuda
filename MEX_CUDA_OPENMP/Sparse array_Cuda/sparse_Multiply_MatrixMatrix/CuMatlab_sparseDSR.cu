
/*
 * This CUDA-Cusparse code can handle/work with  any type of the input mxArrays, 
 * GPUarray or standard matlab CPU array as input {prhs[0]/prhs[1] := mxGPUArray or CPU Array}[double/complex double]
 * Sparse/Dense matrix-sparse/dense matrix multiplication   Z=CuMatlab_multiply(Sparse/Dense(X),Sparse/Dense(Y)).
 * Developed at UCL, Institute of Neurology, 12 Queen Square, WC1N 3AR, London
 * Wellcome Trust Centre for Neuroimaging
 * Part of the project SPM(http://www.fil.ion.ucl.ac.uk/spm)
 * Copyright 2018
 * Kevin Bronik
 */

#include "matrix.h"
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <hipsparse.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "SPARSEHELPER.h"
#include "ERRORCHK.h"
#include <omp.h>
// Input Arguments
#define	INPUTDENSEA   prhs[0]
#define	INPUTSPARSEB   prhs[1]

// Output Arguments
#define	OUTPUTMATRIX  plhs[0]



  
    
extern "C" static void mexCuMatlab_sparseDSR(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
 int nDevices;
hipError_t errCode =hipGetDeviceCount(&nDevices); 
//int nDevices;
//hipGetDeviceCount(&nDevices);

if (errCode != hipSuccess){
printf("Error! No CUDA devices found! \n");
return;
}

    char const * const InputErrMsg = "Invalid input to MEX file, number of input arguments must be two.";
    char const * const OutputErrMsg = "Invalid output to MEX file, number of output arguments must be one.";
   if ((nrhs!=2)) {
        mexErrMsgIdAndTxt("MATLAB:mexatexit:invalidInput", InputErrMsg);
    }
   if ((nlhs!=1)) {
        mexErrMsgIdAndTxt("MATLAB:mexatexit:invalidInput", OutputErrMsg);
    }
 char *input_buf0;
 input_buf0 = mxArrayToString(INPUTDENSEA);

      if ((mxIsChar(INPUTDENSEA))){
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Input(FIRST ARGUMENT) must be array, or gpuArray object not  %s\n",input_buf0);
    }
    
 char *input_buf1;
 input_buf1 = mxArrayToString(INPUTSPARSEB);

      if ((mxIsChar(INPUTSPARSEB))){
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Input(SECOND ARGUMENT) must be array, or gpuArray object not  %s\n",input_buf1);
    } 



if (mxIsGPUArray(INPUTDENSEA) && mxIsGPUArray(INPUTSPARSEB)) {
    
    mxGPUArray const *INPUTDENSEGPUA;
    mxGPUArray const *INPUTSPARSEGPUB;
    
    
    /* Initialize the MathWorks GPU API. */
    mxInitGPU();
    INPUTDENSEGPUA = mxGPUCreateFromMxArray(INPUTDENSEA);
    INPUTSPARSEGPUB = mxGPUCreateFromMxArray(INPUTSPARSEB);
    
   
	
    if((!mxGPUIsSparse(INPUTDENSEGPUA))&& (mxGPUIsSparse(INPUTSPARSEGPUB)) ){
        
    const mwSize *dimsGPUSA;
    dimsGPUSA=mxGPUGetDimensions(INPUTDENSEGPUA);
    int numARows, numAColumns;
    numARows = (int)dimsGPUSA[0]; /* gets number of rows of A */
    numAColumns = (int)dimsGPUSA[1]; /* gets number of columns of A */
    
    const mwSize *dimsGPUSB;
    dimsGPUSB=mxGPUGetDimensions(INPUTSPARSEGPUB);
    int numBRows, numBColumns;
    numBRows = (int)dimsGPUSB[0]; /* gets number of rows of B */
    numBColumns = (int)dimsGPUSB[1]; /* gets number of columns of B */
    if ( numAColumns != numBRows) {
		
		mxGPUDestroyGPUArray(INPUTDENSEGPUA);
        mxGPUDestroyGPUArray(INPUTSPARSEGPUB);
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Invalid input to MEX file, column number of dense matrix(first argument) must match the row number of sparse matrix(second argument).");
             
    }
   
   double const *d_A_dense;
   d_A_dense = (double const *)(mxGPUGetDataReadOnly(INPUTDENSEGPUA));
    
    mwIndex nnz2;
    mxArray * VLSXY2 = mxGPUCreateMxArrayOnCPU(INPUTSPARSEGPUB);
    nnz2 = *(mxGetJc(VLSXY2) + numBColumns);
    int nnzB = (int)nnz2;
          
    mxArray *  ROW_SORTB = mxCreateNumericMatrix(nnzB, 1,mxINT32_CLASS, mxREAL);
    int *ROWSORTB  = (int *)mxGetInt32s(ROW_SORTB);
       SetIr_Data(VLSXY2, ROWSORTB);
    
   mxArray *  COL_SORTB = mxCreateNumericMatrix(nnzB, 1, mxINT32_CLASS, mxREAL);
    int  *COLSORTB = (int *)mxGetInt32s(COL_SORTB);
          SetJc_Int(VLSXY2, COLSORTB);
      
    double  *VALSORTB = (double *)mxGetDoubles(VLSXY2);
           
     hipsparseHandle_t handle;    cusparseSafeCall(hipsparseCreate(&handle));

	hipsparseMatDescr_t descrA;      cusparseSafeCall(hipsparseCreateMatDescr(&descrA));
	hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE);        
    
    
    int nnzA = 0;                            // --- Number of nonzero elements in dense matrix A
	const int lda = numARows;
	//int *d_nnzPerVectorA;   // gpuErrchk(hipMalloc(&d_nnzPerVectorA, numARows * sizeof(*d_nnzPerVectorA)));
	
	size_t pivot_pervect[1] = {numARows};
    mxGPUArray *PerVect = mxGPUCreateGPUArray(1, (mwSize*) pivot_pervect, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
	int *d_nnzPerVectorA = (int*)mxGPUGetData(PerVect);
	//double *d_A_dense;  gpuErrchk(hipMalloc(&d_A_dense, numARows * numAColumns * sizeof(*d_A_dense)));
	//gpuErrchk(hipMemcpy(d_A_dense, h_A_dense1, numARows * numAColumns * sizeof(*d_A_dense), hipMemcpyHostToDevice));
	cusparseSafeCall(hipsparseDnnz(handle, HIPSPARSE_DIRECTION_ROW, numARows, numAColumns, descrA, d_A_dense, lda, d_nnzPerVectorA, &nnzA));      
       // double *d_A;           // gpuErrchk(hipMalloc(&d_A, nnzA * sizeof(*d_A)));
		//int *d_A_RowIndices;    //gpuErrchk(hipMalloc(&d_A_RowIndices, (numARows + 1) * sizeof(*d_A_RowIndices)));
		//int *d_A_ColIndices;    //gpuErrchk(hipMalloc(&d_A_ColIndices, nnzA * sizeof(*d_A_ColIndices)));
		
		
   size_t pivot_dimensA[1] = {nnzA};
   size_t pivot_dimensROW_A[1] = {numARows+1};
   size_t pivot_dimensCOL_A[1] = {nnzA};
   
   
   mxGPUArray *A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensA, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    double  *d_A = (double *)mxGPUGetData(A);
   mxGPUArray * ROW_A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensROW_A, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_A_RowIndices = (int *)mxGPUGetData(ROW_A);
   mxGPUArray * COL_A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensCOL_A, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_A_ColIndices = (int *)mxGPUGetData(COL_A);

		cusparseSafeCall(hipsparseDdense2csr(handle, numARows, numAColumns, descrA, d_A_dense, lda, d_nnzPerVectorA, d_A, d_A_RowIndices, d_A_ColIndices));       
        //gpuErrchk(hipFree(d_A_dense));
        mxGPUDestroyGPUArray(PerVect);
         mxGPUDestroyGPUArray(INPUTDENSEGPUA);
         mxGPUDestroyGPUArray(INPUTSPARSEGPUB);
	//double *d_B;            //gpuErrchk(hipMalloc(&d_B, nnzB * sizeof(*d_B)));
	//int *d_B_RowIndices;    //gpuErrchk(hipMalloc(&d_B_RowIndices, (numBRows + 1) * sizeof(*d_B_RowIndices)));
	//int *d_B_ColIndices;   // gpuErrchk(hipMalloc(&d_B_ColIndices, nnzB * sizeof(*d_B_ColIndices)));
	//int *d_cooRowIndB;      // gpuErrchk(hipMalloc(&d_cooRowIndB, nnzB * sizeof(*d_cooRowIndB)));
	
   size_t pivot_dimensB[1] = {nnzB};
   size_t pivot_dimensROW_B[1] = {numBRows+1};
   size_t pivot_dimensCOL_B[1] = {nnzB};
   size_t pivot_dimensCOO_B[1] = {nnzB};
   
   mxGPUArray *B = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensB, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    double  *d_B = (double *)mxGPUGetData(B);
   mxGPUArray * ROW_B = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensROW_B, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_B_RowIndices = (int *)mxGPUGetData(ROW_B);
   mxGPUArray * COL_B = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensCOL_B, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_B_ColIndices = (int *)mxGPUGetData(COL_B);
    mxGPUArray * COO_B = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensCOO_B, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_cooRowIndB = (int *)mxGPUGetData(COO_B);
	
	
	
	
	
	
	// --- Descriptor for sparse matrix B
	gpuErrchk(hipMemcpy(d_B, VALSORTB, nnzB * sizeof(*d_B), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_B_ColIndices, COLSORTB, nnzB * sizeof(*d_B_ColIndices), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_cooRowIndB, ROWSORTB, nnzB * sizeof(*d_cooRowIndB), hipMemcpyHostToDevice));
    
    
	int *Pb = NULL;
	void *pBufferb = NULL;
	size_t pBufferSizeInBytesb = 0;
	hipsparseXcoosort_bufferSizeExt(handle, numBRows, numBColumns,
		nnzB,
		d_cooRowIndB,
		d_B_ColIndices, &pBufferSizeInBytesb);

	gpuErrchk(hipMalloc(&pBufferb, sizeof(char)*pBufferSizeInBytesb));
	gpuErrchk(hipMalloc(&Pb, sizeof(int)*nnzB));
	hipsparseCreateIdentityPermutation(handle, nnzB, Pb);
	cusparseSafeCall(hipsparseXcoosortByRow(handle, numBRows, numBColumns,
		nnzB,
		d_cooRowIndB,
		d_B_ColIndices,
		Pb,
		pBufferb));

	cusparseSafeCall(hipsparseDgthr(handle, nnzB, d_B, d_B, Pb, HIPSPARSE_INDEX_BASE_ZERO));

	cusparseSafeCall(hipsparseXcoo2csr(handle,
		d_cooRowIndB,
		nnzB,
		numBRows,
		d_B_RowIndices,
		HIPSPARSE_INDEX_BASE_ONE));

	cusparseSafeCall(hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST));

	int nnzC = 0;
	int baseC = 0;
// nnzTotalDevHostPtr points to host memory
    int *nnzTotalDevHostPtr = &nnzC;
	//int *d_C_RowIndices;    gpuErrchk(hipMalloc((void **)&d_C_RowIndices, sizeof(int)*(numARows + 1)));
	
   size_t pivot_dimensROW_C[1] = {numARows+1};
   
   mxGPUArray * ROW_C = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensROW_C, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_C_RowIndices = (int *)mxGPUGetData(ROW_C);
	// where op ( A ) , op ( B ) and C are m�k (numARows�numAColumns), k�n(numBRows�numBColumns), and m�n(numARows�numBColumns) sparse matrices
	cusparseSafeCall(hipsparseXcsrgemmNnz(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, numARows, numBColumns, numAColumns,
		descrA, nnzA, d_A_RowIndices, d_A_ColIndices,
		descrA, nnzB, d_B_RowIndices, d_B_ColIndices,
		descrA, d_C_RowIndices, nnzTotalDevHostPtr ));
       
        
        
     if (NULL != nnzTotalDevHostPtr){
    nnzC = *nnzTotalDevHostPtr;
    }else{
    hipMemcpy(&nnzC, d_C_RowIndices+numARows, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&baseC, d_C_RowIndices, sizeof(int), hipMemcpyDeviceToHost);
    nnzC -= baseC;
     }

	gpuErrchk(hipFree(pBufferb));
	gpuErrchk(hipFree(Pb));
	//gpuErrchk(hipFree(d_cooRowIndB));
	
    
   if (nnzC==0) {
          
         OUTPUTMATRIX = mxCreateSparse(numARows,numBColumns,0,mxREAL);
         
         return;
             
    } 
    
    
   size_t pivot_dimensionsrow[1] = {nnzC};
   size_t pivot_dimensionscolumn[1] = {numBColumns+1}; 
   size_t pivot_dimensionsvalue[1] = {nnzC};
   mxGPUArray * ROW_SORTC = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionsrow, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *ROWSORTC = (int *)mxGPUGetData(ROW_SORTC);
   mxGPUArray * COL_SORTC = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionscolumn, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *COLSORTC = (int *)mxGPUGetData(COL_SORTC);
    mxGPUArray *VAL_SORTC = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionsvalue, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    double  *VALSORTC = (double *)mxGPUGetData(VAL_SORTC);
   mwSize nnzm=(mwSize)nnzC;
   
   OUTPUTMATRIX = mxCreateSparse(numARows,numBColumns,nnzm,mxREAL);
    
    
   // double *d_C;          gpuErrchk(hipMalloc((void **)&d_C, sizeof(double)*(nnzC)));
   // int *d_C_ColIndices;   gpuErrchk(hipMalloc((void **)&d_C_ColIndices, sizeof(int)*(nnzC)));
	
	size_t pivot_dimensC[1] = {nnzC};
    size_t pivot_dimensCOL_C[1] = {nnzC};
   
   
   mxGPUArray *C = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensC, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    double  *d_C = (double *)mxGPUGetData(C);
   mxGPUArray * COL_C = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensCOL_C, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_C_ColIndices = (int *)mxGPUGetData(COL_C);
    
    
	
	
	    
   	cusparseSafeCall(hipsparseDcsrgemm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, numARows, numBColumns, numAColumns,
		descrA, nnzA,
		d_A, d_A_RowIndices, d_A_ColIndices,
		descrA, nnzB,
		d_B, d_B_RowIndices, d_B_ColIndices,
		descrA,
		d_C, d_C_RowIndices, d_C_ColIndices));
        
	//gpuErrchk(hipFree(d_A));
	//gpuErrchk(hipFree(d_A_RowIndices));
	//gpuErrchk(hipFree(d_A_ColIndices));
	
	//gpuErrchk(hipFree(d_B));
	//gpuErrchk(hipFree(d_B_RowIndices));
	//gpuErrchk(hipFree(d_B_ColIndices));
	
	mxGPUDestroyGPUArray(A);
    mxGPUDestroyGPUArray(ROW_A);
    mxGPUDestroyGPUArray(COL_A);
 
    mxGPUDestroyGPUArray(B);
    mxGPUDestroyGPUArray(ROW_B);
    mxGPUDestroyGPUArray(COL_B);
    mxGPUDestroyGPUArray(COO_B);
	//double *d_value_csc;  gpuErrchk(hipMalloc((void **)&d_value_csc, sizeof(double)*(nnzC)));
	//int *d_row_csc;       gpuErrchk(hipMalloc((void **)&d_row_csc, sizeof(int)*(nnzC)));
	//int *d_col_csc;       gpuErrchk(hipMalloc((void **)&d_col_csc, sizeof(int)*(numBColumns + 1)));

	cusparseSafeCall(hipsparseDcsr2csc(handle, numARows, numBColumns, nnzC, d_C, d_C_RowIndices, d_C_ColIndices, VALSORTC, ROWSORTC, COLSORTC, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ONE));
    /*

	int *Pc = NULL;
	void *pBufferc = NULL;
	size_t pBufferSizeInBytesc = 0;
	hipsparseXcscsort_bufferSizeExt(handle, numARows, numBColumns,
		nnzC,
		d_col_csc,
		d_row_csc, &pBufferSizeInBytesc);
   
	gpuErrchk(hipMalloc(&pBufferc, sizeof(char)*pBufferSizeInBytesc));
	gpuErrchk(hipMalloc(&Pc, sizeof(int)*nnzC));
	hipsparseCreateIdentityPermutation(handle, nnzC, Pc);
	cusparseSafeCall(hipsparseXcscsort(handle, numARows, numBColumns,
		nnzC,
		descrA,
		d_col_csc,
		d_row_csc,
		Pc,
		pBufferc));

	cusparseSafeCall(hipsparseDgthr(handle, nnzC, d_value_csc, d_value_csc, Pc, HIPSPARSE_INDEX_BASE_ZERO));
    */
  
	//gpuErrchk(hipMemcpy(VALSORTC, d_value_csc, sizeof(double)* nnzC, hipMemcpyDeviceToHost));
	//gpuErrchk(hipMemcpy(ROWSORTC, d_row_csc, sizeof(int)* (nnzC), hipMemcpyDeviceToHost));
	//gpuErrchk(hipMemcpy(COLSORTC, d_col_csc, sizeof(int)* (numBColumns + 1), hipMemcpyDeviceToHost));

    //gpuErrchk(hipFree(pBufferc));
	//gpuErrchk(hipFree(Pc));
  
    
   mxArray *RS= mxGPUCreateMxArrayOnCPU(ROW_SORTC);
   int * rs= (int *)mxGetInt32s(RS);
   mxArray *CS= mxGPUCreateMxArrayOnCPU(COL_SORTC);
   int * cs= (int *)mxGetInt32s(CS);

    
      mwIndex *irs,*jcs;
  
        irs = static_cast<mwIndex *> (mxMalloc (nnzC * sizeof(mwIndex)));
                           int i;
	   #pragma omp parallel for shared(nnzC) private(i)
         for (i = 0; i < nnzC; ++i) {
           irs[i] = static_cast<mwIndex> (rs[i])-1;  
            }
      
      jcs = static_cast<mwIndex *> (mxMalloc ((numBColumns+1) * sizeof(mwIndex)));
      int nc1= numBColumns+1;
       #pragma omp parallel for shared(nc1) private(i)
            for (i = 0; i < nc1; ++i) {
           jcs[i] = static_cast<mwIndex> (cs[i])-1;
            }
             
        mxDouble* PRS = (mxDouble*) mxMalloc (nnzC * sizeof(mxDouble));
        gpuErrchk(hipMemcpy(PRS, VALSORTC, nnzC * sizeof(mxDouble), hipMemcpyDeviceToHost));
		
           
    
   
        mxFree (mxGetJc (OUTPUTMATRIX)) ;
        mxFree (mxGetIr (OUTPUTMATRIX)) ;
        mxFree (mxGetDoubles(OUTPUTMATRIX)) ;
    
        mxSetIr(OUTPUTMATRIX, (mwIndex *)irs);
        mxSetJc(OUTPUTMATRIX, (mwIndex *)jcs);
        int s=mxSetDoubles(OUTPUTMATRIX, (mxDouble *)PRS);
         if ( s == 0) {
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "the function is unsuccessful, either mxArray is not an unshared mxDOUBLE_CLASS array, or the data is not allocated with mxCalloc.");
             
         }    
    
	//gpuErrchk(hipFree(d_C));
	//gpuErrchk(hipFree(d_C_RowIndices));
	//gpuErrchk(hipFree(d_C_ColIndices));

	//gpuErrchk(hipFree(d_value_csc));
	//gpuErrchk(hipFree(d_row_csc));
	//gpuErrchk(hipFree(d_col_csc));

      mxGPUDestroyGPUArray(C);
      mxGPUDestroyGPUArray(ROW_C);
      mxGPUDestroyGPUArray(COL_C);



         mxDestroyArray(VLSXY2);
         mxGPUDestroyGPUArray(VAL_SORTC);
         mxGPUDestroyGPUArray(ROW_SORTC);
         mxGPUDestroyGPUArray(COL_SORTC);
         mxDestroyArray(RS);
         mxDestroyArray(CS);

         
         mxDestroyArray(COL_SORTB);
         mxDestroyArray(ROW_SORTB);
        hipsparseDestroyMatDescr(descrA);  
		hipsparseDestroy(handle);
        
        }
    
        else{
        mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Incorrect input arguments! %s\n");    
        }
    
   }
     
////////////////////////////////////////////////////////////////////////////////////  
    else if (!(mxIsGPUArray(INPUTDENSEA)) && !(mxIsGPUArray(INPUTSPARSEB))){
   
     // if ((mxGetClassID(INPUTSPARSEA) != mxDOUBLE_CLASS) || (mxGetClassID(INPUTSPARSEB) != mxDOUBLE_CLASS)) {
       //  mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
           //     "Invalid input to MEX file, input(FIRST and SECOND  ARGUMENTS) must be  double precision.");
             
   // }
    if((!mxIsSparse(INPUTDENSEA))&& (mxIsSparse(INPUTSPARSEB)) ){
    
     mxInitGPU();
    const mwSize *dimsCPUA;
    dimsCPUA=mxGetDimensions(INPUTDENSEA);
    
    int  numARows = (int)dimsCPUA[0]; /* gets number of rows of A */
    int  numAColumns = (int)dimsCPUA[1]; /* gets number of columns of A */
   
    const mwSize *dimsCPUB;
    dimsCPUB=mxGetDimensions(INPUTSPARSEB);
    
    int  numBRows = (int)dimsCPUB[0]; /* gets number of rows of B */
    int  numBColumns = (int)dimsCPUB[1]; /* gets number of columns of B */
    if ( numAColumns != numBRows) {
	
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Invalid input to MEX file, column number of dense matrix(first argument) must match the row number of sparse matrix(second argument).");
             
    }
    
   double *h_A_dense1;
   h_A_dense1 = (double *)mxGetDoubles(INPUTDENSEA);
    
    mwIndex nnz2;
 
    nnz2 = *(mxGetJc(INPUTSPARSEB) + numBColumns);
    int nnzB = (int)nnz2;
    
          
    mxArray *  ROW_SORTB = mxCreateNumericMatrix(nnzB, 1,mxINT32_CLASS, mxREAL);
    int *ROWSORTB  = (int *)mxGetInt32s(ROW_SORTB);
       SetIr_Data(INPUTSPARSEB, ROWSORTB);

    
   mxArray *  COL_SORTB = mxCreateNumericMatrix(nnzB, 1, mxINT32_CLASS, mxREAL);
    int  *COLSORTB = (int *)mxGetInt32s(COL_SORTB);
          SetJc_Int(INPUTSPARSEB, COLSORTB);

      
    double  *VALSORTB = (double *)mxGetDoubles(INPUTSPARSEB); 
          
     hipsparseHandle_t handle;    cusparseSafeCall(hipsparseCreate(&handle));

	hipsparseMatDescr_t descrA;      cusparseSafeCall(hipsparseCreateMatDescr(&descrA));
	hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE);        
    
    
    int nnzA = 0;                            // --- Number of nonzero elements in dense matrix A
	const int lda = numARows;
	//int *d_nnzPerVectorA;    gpuErrchk(hipMalloc(&d_nnzPerVectorA, numARows * sizeof(*d_nnzPerVectorA)));
	size_t pivot_pervect[1] = {numARows};
    mxGPUArray *PerVect = mxGPUCreateGPUArray(1, (mwSize*) pivot_pervect, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
	int *d_nnzPerVectorA = (int*)mxGPUGetData(PerVect);
    
    //double *d_A_dense;  gpuErrchk(hipMalloc(&d_A_dense, numARows * numAColumns * sizeof(*d_A_dense)));
	
	  size_t pivot_dimensionsvalueDA[2] = {numARows, numAColumns};
      mxGPUArray *OUTMA = mxGPUCreateGPUArray(2, (mwSize*) pivot_dimensionsvalueDA, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
      double *d_A_dense = (double *)mxGPUGetData(OUTMA);
	
	
	
	gpuErrchk(hipMemcpy(d_A_dense, h_A_dense1, numARows * numAColumns * sizeof(*d_A_dense), hipMemcpyHostToDevice));
	cusparseSafeCall(hipsparseDnnz(handle, HIPSPARSE_DIRECTION_ROW, numARows, numAColumns, descrA, d_A_dense, lda, d_nnzPerVectorA, &nnzA));      
       // double *d_A;           // gpuErrchk(hipMalloc(&d_A, nnzA * sizeof(*d_A)));
		//int *d_A_RowIndices;   // gpuErrchk(hipMalloc(&d_A_RowIndices, (numARows + 1) * sizeof(*d_A_RowIndices)));
		//int *d_A_ColIndices;   // gpuErrchk(hipMalloc(&d_A_ColIndices, nnzA * sizeof(*d_A_ColIndices)));
   size_t pivot_dimensA[1] = {nnzA};
   size_t pivot_dimensROW_A[1] = {numARows+1};
   size_t pivot_dimensCOL_A[1] = {nnzA};
   
   
   mxGPUArray *A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensA, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    double  *d_A = (double *)mxGPUGetData(A);
   mxGPUArray * ROW_A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensROW_A, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_A_RowIndices = (int *)mxGPUGetData(ROW_A);
   mxGPUArray * COL_A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensCOL_A, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_A_ColIndices = (int *)mxGPUGetData(COL_A);
    
    
		cusparseSafeCall(hipsparseDdense2csr(handle, numARows, numAColumns, descrA, d_A_dense, lda, d_nnzPerVectorA, d_A, d_A_RowIndices, d_A_ColIndices));       
		mxGPUDestroyGPUArray(OUTMA);
        //gpuErrchk(hipFree(d_nnzPerVectorA));
          mxGPUDestroyGPUArray(PerVect);
	//double *d_B;           // gpuErrchk(hipMalloc(&d_B, nnzB * sizeof(*d_B)));
	//int *d_B_RowIndices;   // gpuErrchk(hipMalloc(&d_B_RowIndices, (numBRows + 1) * sizeof(*d_B_RowIndices)));
	//int *d_B_ColIndices;   // gpuErrchk(hipMalloc(&d_B_ColIndices, nnzB * sizeof(*d_B_ColIndices)));
	//int *d_cooRowIndB;     //  gpuErrchk(hipMalloc(&d_cooRowIndB, nnzB * sizeof(*d_cooRowIndB)));
	
   size_t pivot_dimensB[1] = {nnzB};
   size_t pivot_dimensROW_B[1] = {numBRows+1};
   size_t pivot_dimensCOL_B[1] = {nnzB};
   size_t pivot_dimensCOO_B[1] = {nnzB};
   
   mxGPUArray *B = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensB, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    double  *d_B = (double *)mxGPUGetData(B);
   mxGPUArray * ROW_B = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensROW_B, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_B_RowIndices = (int *)mxGPUGetData(ROW_B);
   mxGPUArray * COL_B = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensCOL_B, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_B_ColIndices = (int *)mxGPUGetData(COL_B);
    mxGPUArray * COO_B = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensCOO_B, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_cooRowIndB = (int *)mxGPUGetData(COO_B);
    
    
    
    // --- Descriptor for sparse matrix B
	gpuErrchk(hipMemcpy(d_B, VALSORTB, nnzB * sizeof(*d_B), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_B_ColIndices, COLSORTB, nnzB * sizeof(*d_B_ColIndices), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_cooRowIndB, ROWSORTB, nnzB * sizeof(*d_cooRowIndB), hipMemcpyHostToDevice));
    
    
	int *Pb = NULL;
	void *pBufferb = NULL;
	size_t pBufferSizeInBytesb = 0;
	hipsparseXcoosort_bufferSizeExt(handle, numBRows, numBColumns,
		nnzB,
		d_cooRowIndB,
		d_B_ColIndices, &pBufferSizeInBytesb);

	gpuErrchk(hipMalloc(&pBufferb, sizeof(char)*pBufferSizeInBytesb));
	gpuErrchk(hipMalloc(&Pb, sizeof(int)*nnzB));
	hipsparseCreateIdentityPermutation(handle, nnzB, Pb);
	cusparseSafeCall(hipsparseXcoosortByRow(handle, numBRows, numBColumns,
		nnzB,
		d_cooRowIndB,
		d_B_ColIndices,
		Pb,
		pBufferb));

	cusparseSafeCall(hipsparseDgthr(handle, nnzB, d_B, d_B, Pb, HIPSPARSE_INDEX_BASE_ZERO));

	cusparseSafeCall(hipsparseXcoo2csr(handle,
		d_cooRowIndB,
		nnzB,
		numBRows,
		d_B_RowIndices,
		HIPSPARSE_INDEX_BASE_ONE));

	cusparseSafeCall(hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST));

	int nnzC = 0;
	int baseC = 0;
// nnzTotalDevHostPtr points to host memory
    int *nnzTotalDevHostPtr = &nnzC;
	//int *d_C_RowIndices;   // gpuErrchk(hipMalloc((void **)&d_C_RowIndices, sizeof(int)*(numARows + 1)));
     size_t pivot_dimensROW_C[1] = {numARows+1};
   
    mxGPUArray * ROW_C = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensROW_C, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_C_RowIndices = (int *)mxGPUGetData(ROW_C);	
    
    // where op ( A ) , op ( B ) and C are m�k (numARows�numAColumns), k�n(numBRows�numBColumns), and m�n(numARows�numBColumns) sparse matrices
	cusparseSafeCall(hipsparseXcsrgemmNnz(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, numARows, numBColumns, numAColumns,
		descrA, nnzA, d_A_RowIndices, d_A_ColIndices,
		descrA, nnzB, d_B_RowIndices, d_B_ColIndices,
		descrA, d_C_RowIndices, nnzTotalDevHostPtr ));
       
        
        
     if (NULL != nnzTotalDevHostPtr){
    nnzC = *nnzTotalDevHostPtr;
    }else{
    hipMemcpy(&nnzC, d_C_RowIndices+numARows, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&baseC, d_C_RowIndices, sizeof(int), hipMemcpyDeviceToHost);
    nnzC -= baseC;
     }

	gpuErrchk(hipFree(pBufferb));
	gpuErrchk(hipFree(Pb));
	//gpuErrchk(hipFree(d_cooRowIndB));
             
   
   if (nnzC==0) {
          
         OUTPUTMATRIX = mxCreateSparse(numARows,numBColumns,0,mxREAL);
         
         return;
             
    } 
    
      
  size_t pivot_dimensionsrow[1] = {nnzC};
   size_t pivot_dimensionscolumn[1] = {numBColumns+1}; 
   size_t pivot_dimensionsvalue[1] = {nnzC};
   mxGPUArray * ROW_SORTC = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionsrow, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *ROWSORTC = (int *)mxGPUGetData(ROW_SORTC);
   mxGPUArray * COL_SORTC = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionscolumn, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *COLSORTC = (int *)mxGPUGetData(COL_SORTC);
    mxGPUArray *VAL_SORTC = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionsvalue, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    double  *VALSORTC = (double *)mxGPUGetData(VAL_SORTC);
   mwSize nnzm=(mwSize)nnzC;
   
   OUTPUTMATRIX = mxCreateSparse(numARows,numBColumns,nnzm,mxREAL);
    
   // double *d_C;         // gpuErrchk(hipMalloc((void **)&d_C, sizeof(double)*(nnzC)));
   // int *d_C_ColIndices; //  gpuErrchk(hipMalloc((void **)&d_C_ColIndices, sizeof(int)*(nnzC)));
	   
   	size_t pivot_dimensC[1] = {nnzC};
    size_t pivot_dimensCOL_C[1] = {nnzC};
   
   
   mxGPUArray *C = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensC, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    double  *d_C = (double *)mxGPUGetData(C);
   mxGPUArray * COL_C = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensCOL_C, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_C_ColIndices = (int *)mxGPUGetData(COL_C); 
    
   	cusparseSafeCall(hipsparseDcsrgemm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, numARows, numBColumns, numAColumns,
		descrA, nnzA,
		d_A, d_A_RowIndices, d_A_ColIndices,
		descrA, nnzB,
		d_B, d_B_RowIndices, d_B_ColIndices,
		descrA,
		d_C, d_C_RowIndices, d_C_ColIndices));
        
	//gpuErrchk(hipFree(d_A));
	//gpuErrchk(hipFree(d_A_RowIndices));
	//gpuErrchk(hipFree(d_A_ColIndices));
	
	//gpuErrchk(hipFree(d_B));
	//gpuErrchk(hipFree(d_B_RowIndices));
	//gpuErrchk(hipFree(d_B_ColIndices));
    
    mxGPUDestroyGPUArray(A);
    mxGPUDestroyGPUArray(ROW_A);
    mxGPUDestroyGPUArray(COL_A);
 
    mxGPUDestroyGPUArray(B);
    mxGPUDestroyGPUArray(ROW_B);
    mxGPUDestroyGPUArray(COL_B);
    mxGPUDestroyGPUArray(COO_B);
    
    
	cusparseSafeCall(hipsparseDcsr2csc(handle, numARows, numBColumns, nnzC, d_C, d_C_RowIndices, d_C_ColIndices, VALSORTC, ROWSORTC, COLSORTC, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ONE));
    /*

	int *Pc = NULL;
	void *pBufferc = NULL;
	size_t pBufferSizeInBytesc = 0;
	hipsparseXcscsort_bufferSizeExt(handle, numARows, numBColumns,
		nnzC,
		d_col_csc,
		d_row_csc, &pBufferSizeInBytesc);
   
	gpuErrchk(hipMalloc(&pBufferc, sizeof(char)*pBufferSizeInBytesc));
	gpuErrchk(hipMalloc(&Pc, sizeof(int)*nnzC));
	hipsparseCreateIdentityPermutation(handle, nnzC, Pc);
	cusparseSafeCall(hipsparseXcscsort(handle, numARows, numBColumns,
		nnzC,
		descrA,
		d_col_csc,
		d_row_csc,
		Pc,
		pBufferc));

	cusparseSafeCall(hipsparseDgthr(handle, nnzC, d_value_csc, d_value_csc, Pc, HIPSPARSE_INDEX_BASE_ZERO));
    */
  
	//gpuErrchk(hipMemcpy(VALSORTC, d_value_csc, sizeof(double)* nnzC, hipMemcpyDeviceToHost));
	//gpuErrchk(hipMemcpy(ROWSORTC, d_row_csc, sizeof(int)* (nnzC), hipMemcpyDeviceToHost));
	//gpuErrchk(hipMemcpy(COLSORTC, d_col_csc, sizeof(int)* (numBColumns + 1), hipMemcpyDeviceToHost));

    //gpuErrchk(hipFree(pBufferc));
	//gpuErrchk(hipFree(Pc));
  
    
   mxArray *RS= mxGPUCreateMxArrayOnCPU(ROW_SORTC);
   int * rs= (int *)mxGetInt32s(RS);
   mxArray *CS= mxGPUCreateMxArrayOnCPU(COL_SORTC);
   int * cs= (int *)mxGetInt32s(CS);

    
      mwIndex *irs,*jcs;
  
        irs = static_cast<mwIndex *> (mxMalloc (nnzC * sizeof(mwIndex)));
                           int i;
	   #pragma omp parallel for shared(nnzC) private(i)
         for (i = 0; i < nnzC; ++i) {
           irs[i] = static_cast<mwIndex> (rs[i])-1;  
            }
      
      jcs = static_cast<mwIndex *> (mxMalloc ((numBColumns+1) * sizeof(mwIndex)));
      int nc1= numBColumns+1;
       #pragma omp parallel for shared(nc1) private(i)
            for (i = 0; i < nc1; ++i) {
           jcs[i] = static_cast<mwIndex> (cs[i])-1;
            }
             
        mxDouble* PRS = (mxDouble*) mxMalloc (nnzC * sizeof(mxDouble));
        gpuErrchk(hipMemcpy(PRS, VALSORTC, nnzC * sizeof(mxDouble), hipMemcpyDeviceToHost));          

           
    
   
        mxFree (mxGetJc (OUTPUTMATRIX)) ;
        mxFree (mxGetIr (OUTPUTMATRIX)) ;
        mxFree (mxGetDoubles(OUTPUTMATRIX)) ;
    
        mxSetIr(OUTPUTMATRIX, (mwIndex *)irs);
        mxSetJc(OUTPUTMATRIX, (mwIndex *)jcs);
        int s=mxSetDoubles(OUTPUTMATRIX, (mxDouble *)PRS);
         if ( s == 0) {
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "the function is unsuccessful, either mxArray is not an unshared mxDOUBLE_CLASS array, or the data is not allocated with mxCalloc.");
             
         } 
        

	//gpuErrchk(hipFree(d_C));
	//gpuErrchk(hipFree(d_C_RowIndices));
	//gpuErrchk(hipFree(d_C_ColIndices));
      mxGPUDestroyGPUArray(C);
      mxGPUDestroyGPUArray(ROW_C);
      mxGPUDestroyGPUArray(COL_C);
      
	//gpuErrchk(hipFree(d_value_csc));
	//gpuErrchk(hipFree(d_row_csc));
	//gpuErrchk(hipFree(d_col_csc));

         mxGPUDestroyGPUArray(VAL_SORTC);
         mxGPUDestroyGPUArray(ROW_SORTC);
         mxGPUDestroyGPUArray(COL_SORTC);
         mxDestroyArray(RS);
         mxDestroyArray(CS);

         
         mxDestroyArray(COL_SORTB);
         mxDestroyArray(ROW_SORTB);
       hipsparseDestroyMatDescr(descrA);  

	   hipsparseDestroy(handle);
    }
    else{
        mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Incorrect input arguments! %s\n");    
        }
    
 }
        //
    else{
        mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Incorrect input arguments! %s\n");    
        }

}
