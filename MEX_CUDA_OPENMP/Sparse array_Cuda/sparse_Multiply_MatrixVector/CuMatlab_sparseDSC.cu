
/*
 * This CUDA-Cusparse code can handle/work with  any type of the input mxArrays, 
 * GPUarray or standard matlab CPU array as input {prhs[0]/prhs[1] := mxGPUArray or CPU Array}[double/complex double]
 * Sparse/Dense matrix-sparse/dense vector multiplication   Z=CuMatlab_multiplyV(Sparse/Dense(X),Sparse/Dense(Y), alpha).
 * Z= alpha*X*Y
 * Developed at UCL, Institute of Neurology, 12 Queen Square, WC1N 3AR, London
 * Wellcome Trust Centre for Neuroimaging
 * Part of the project SPM(http://www.fil.ion.ucl.ac.uk/spm)
 * Copyright 2018
 * Kevin Bronik
 */

#include "matrix.h"
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <hipsparse.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "SPARSEHELPER.h"
#include "ERRORCHK.h"
#include <omp.h>

// Input Arguments
#define	INPUTDENSEA   prhs[0]
#define	INPUTSPARSEB   prhs[1]
#define	ALPHA   prhs[2]
//#define	BETA    prhs[3]
// Output Arguments
#define	OUTPUTMATRIX  plhs[0]



  
    
extern "C" static void mexCuMatlab_sparseDSC(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
 int nDevices;
hipError_t errCode =hipGetDeviceCount(&nDevices); 
//int nDevices;
//hipGetDeviceCount(&nDevices);

if (errCode != hipSuccess){
printf("Error! No CUDA devices found! \n");
return;
}

    char const * const InputErrMsg = "Invalid input to MEX file, number of input arguments must be three.";
    char const * const OutputErrMsg = "Invalid output to MEX file, number of output arguments must be one.";
   if ((nrhs!=3)) {
        mexErrMsgIdAndTxt("MATLAB:mexatexit:invalidInput", InputErrMsg);
    }
   if ((nlhs!=1)) {
        mexErrMsgIdAndTxt("MATLAB:mexatexit:invalidInput", OutputErrMsg);
    }
 char *input_buf0;
 input_buf0 = mxArrayToString(INPUTDENSEA);

      if ((mxIsChar(INPUTDENSEA))){
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Input(FIRST ARGUMENT) must be array, or gpuArray object not  %s\n",input_buf0);
    }
    
 char *input_buf1;
 input_buf1 = mxArrayToString(INPUTSPARSEB);

      if ((mxIsChar(INPUTSPARSEB))){
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Input(SECOND ARGUMENT) must be array, or gpuArray object not  %s\n",input_buf1);
    } 

 char *input_buf2;
 input_buf2 = mxArrayToString(ALPHA);

      if ((mxIsChar(ALPHA))){
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Input(THIRD ARGUMENT) must be scalar not  %s\n",input_buf2);
    } 


if (mxIsGPUArray(INPUTDENSEA) && mxIsGPUArray(INPUTSPARSEB)) {
    
    mxGPUArray const *INPUTDENSEGPUA;
    mxGPUArray const *INPUTSPARSEGPUB;
    
    
    /* Initialize the MathWorks GPU API. */
    mxInitGPU();
    INPUTDENSEGPUA = mxGPUCreateFromMxArray(INPUTDENSEA);
    INPUTSPARSEGPUB = mxGPUCreateFromMxArray(INPUTSPARSEB);
    
   
	
    if((!mxGPUIsSparse(INPUTDENSEGPUA))&& (mxGPUIsSparse(INPUTSPARSEGPUB)) ){
        
    const mwSize *dimsGPUSA;
    dimsGPUSA=mxGPUGetDimensions(INPUTDENSEGPUA);
    int numARows, numAColumns;
    numARows = (int)dimsGPUSA[0]; /* gets number of rows of A */
    numAColumns = (int)dimsGPUSA[1]; /* gets number of columns of A */
    
    const mwSize *dimsGPUSB;
    dimsGPUSB=mxGPUGetDimensions(INPUTSPARSEGPUB);
    int numBRows, numBColumns;
    numBRows = (int)dimsGPUSB[0]; /* gets number of rows of B */
    numBColumns = (int)dimsGPUSB[1]; /* gets number of columns of B */
   if ( (numBColumns!= 1) ) {
              mxGPUDestroyGPUArray(INPUTDENSEGPUA);
              mxGPUDestroyGPUArray(INPUTSPARSEGPUB);   
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                        "Invalid input to MEX file, second argument must be a sparse column vector.");
             
    }
    
    if ( (numAColumns!= numBRows) ) {
              mxGPUDestroyGPUArray(INPUTDENSEGPUA);
              mxGPUDestroyGPUArray(INPUTSPARSEGPUB);    
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                        "Invalid input to MEX file, column number of dense matrix(first argument) must be equal to row numbers of sparse vector(second argument).");
             
    }
        mxComplexDouble*  al= (mxComplexDouble *)mxGetComplexDoubles(ALPHA);
       const hipDoubleComplex alpha = make_hipDoubleComplex(al[0].real, al[0].imag);
       //mxComplexDouble*  bl= (mxComplexDouble *)mxGetComplexDoubles(BETA);
       const hipDoubleComplex beta = make_hipDoubleComplex(0.0, 0.0);
   
    hipDoubleComplex const *d_A_dense;
   d_A_dense = (hipDoubleComplex const *)(mxGPUGetDataReadOnly(INPUTDENSEGPUA));
    
    mwIndex nnz2;
    mxArray * VLSXY2 = mxGPUCreateMxArrayOnCPU(INPUTSPARSEGPUB);
    nnz2 = *(mxGetJc(VLSXY2) + numBColumns);
    
    int nnzB= static_cast<int> (nnz2);    
                 
  
    mxArray *row_sortB =mxCreateNumericMatrix(nnzB, 1, mxINT32_CLASS, mxREAL);
    int *pointerrowB = (int *)mxGetInt32s(row_sortB);
   
    Ir_DataGetSetIXY(VLSXY2 , pointerrowB, nnzB);
  
 
   
    hipDoubleComplex  *pointervalB = (hipDoubleComplex *)mxGetComplexDoubles(VLSXY2);
            
   size_t pivot_dimensionsrowB[1] = {nnzB};
   
   size_t pivot_dimensionsvalueB[1] = {nnzB};    
      mxGPUArray *row_sortBB = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionsrowB, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);  
          
        int *xrow_sortB=(int *)mxGPUGetData(row_sortBB);
 gpuErrchk(hipMemcpy(xrow_sortB, pointerrowB, nnzB * sizeof(*xrow_sortB), hipMemcpyHostToDevice));
       
      mxGPUArray *val_sortBB = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionsvalueB, mxDOUBLE_CLASS, mxCOMPLEX, MX_GPU_DO_NOT_INITIALIZE);  
          
        hipDoubleComplex *xval_sortB=(hipDoubleComplex*)mxGPUGetData(val_sortBB);
 gpuErrchk(hipMemcpy(xval_sortB, pointervalB, nnzB * sizeof(*xval_sortB), hipMemcpyHostToDevice));    
    

         mxGPUDestroyGPUArray(INPUTSPARSEGPUB);
         mxDestroyArray(row_sortB);
         mxDestroyArray(VLSXY2);         
  
           
     hipsparseHandle_t handle;    cusparseSafeCall(hipsparseCreate(&handle));

	hipsparseMatDescr_t descrA;      cusparseSafeCall(hipsparseCreateMatDescr(&descrA));
	hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE);        
      
   
    size_t pivot_dimensionsvalueV[1] = {numBRows};

    mxGPUArray *dB_dense = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionsvalueV, mxDOUBLE_CLASS, mxCOMPLEX, MX_GPU_DO_NOT_INITIALIZE);
    hipDoubleComplex  *d_B_dense = (hipDoubleComplex *)mxGPUGetData(dB_dense);
 
     cusparseSafeCall(hipsparseZsctr(handle, nnzB, 
              xval_sortB, 
              xrow_sortB, d_B_dense, 
              HIPSPARSE_INDEX_BASE_ONE));
			  
          mxGPUDestroyGPUArray(row_sortBB);
          mxGPUDestroyGPUArray(val_sortBB);			  
    
    int nnzA = 0;                            // --- Number of nonzero elements in dense matrix A
	const int lda = numARows;
	//int *d_nnzPerVectorA;   // gpuErrchk(hipMalloc(&d_nnzPerVectorA, numARows * sizeof(*d_nnzPerVectorA)));
	
	size_t pivot_pervect[1] = {numARows};
    mxGPUArray *PerVect = mxGPUCreateGPUArray(1, (mwSize*) pivot_pervect, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
	int *d_nnzPerVectorA = (int*)mxGPUGetData(PerVect);
	//hipDoubleComplex *d_A_dense;  gpuErrchk(hipMalloc(&d_A_dense, numARows * numAColumns * sizeof(*d_A_dense)));
	//gpuErrchk(hipMemcpy(d_A_dense, h_A_dense1, numARows * numAColumns * sizeof(*d_A_dense), hipMemcpyHostToDevice));
	cusparseSafeCall(hipsparseZnnz(handle, HIPSPARSE_DIRECTION_ROW, numARows, numAColumns, descrA, d_A_dense, lda, d_nnzPerVectorA, &nnzA));      
       // hipDoubleComplex *d_A;           // gpuErrchk(hipMalloc(&d_A, nnzA * sizeof(*d_A)));
		//int *d_A_RowIndices;    //gpuErrchk(hipMalloc(&d_A_RowIndices, (numARows + 1) * sizeof(*d_A_RowIndices)));
		//int *d_A_ColIndices;    //gpuErrchk(hipMalloc(&d_A_ColIndices, nnzA * sizeof(*d_A_ColIndices)));
		
		
   size_t pivot_dimensA[1] = {nnzA};
   size_t pivot_dimensROW_A[1] = {numARows+1};
   size_t pivot_dimensCOL_A[1] = {nnzA};
   
   
   mxGPUArray *A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensA, mxDOUBLE_CLASS, mxCOMPLEX, MX_GPU_DO_NOT_INITIALIZE);
    hipDoubleComplex  *d_A = (hipDoubleComplex *)mxGPUGetData(A);
   mxGPUArray * ROW_A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensROW_A, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_A_RowIndices = (int *)mxGPUGetData(ROW_A);
   mxGPUArray * COL_A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensCOL_A, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_A_ColIndices = (int *)mxGPUGetData(COL_A);

		cusparseSafeCall(hipsparseZdense2csr(handle, numARows, numAColumns, descrA, d_A_dense, lda, d_nnzPerVectorA, d_A, d_A_RowIndices, d_A_ColIndices));       
        //gpuErrchk(hipFree(d_A_dense));
         mxGPUDestroyGPUArray(PerVect);
         mxGPUDestroyGPUArray(INPUTDENSEGPUA);

         
   
    
    size_t pivot_dimensionsvalue[1] = {numARows};

    mxGPUArray *VAL = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionsvalue, mxDOUBLE_CLASS, mxCOMPLEX, MX_GPU_INITIALIZE_VALUES);
    hipDoubleComplex  *VALOUT = (hipDoubleComplex *)mxGPUGetData(VAL);

 cusparseSafeCall(hipsparseZcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
               numARows, numAColumns, nnzA, &alpha, 
               descrA, 
               d_A, 
               d_A_RowIndices, d_A_ColIndices,
               d_B_dense, &beta, 
               VALOUT));
            
          mxGPUDestroyGPUArray(A);  
          mxGPUDestroyGPUArray(ROW_A); 
          mxGPUDestroyGPUArray(COL_A);            
          mxGPUDestroyGPUArray(dB_dense);
          
  OUTPUTMATRIX = mxGPUCreateMxArrayOnGPU(VAL);             

        mxGPUDestroyGPUArray(VAL);

        hipsparseDestroyMatDescr(descrA);   
		hipsparseDestroy(handle);        

        
        }
    
        else{
        mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Incorrect input arguments! %s\n");    
        }
    
   }
     
////////////////////////////////////////////////////////////////////////////////////  
    else if (!(mxIsGPUArray(INPUTDENSEA)) && !(mxIsGPUArray(INPUTSPARSEB))){
   
     // if ((mxGetClassID(INPUTSPARSEA) != mxDOUBLE_CLASS) || (mxGetClassID(INPUTSPARSEB) != mxDOUBLE_CLASS)) {
       //  mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
           //     "Invalid input to MEX file, input(FIRST and SECOND  ARGUMENTS) must be  hipDoubleComplex precision.");
             
   // }
    if((!mxIsSparse(INPUTDENSEA))&& (mxIsSparse(INPUTSPARSEB)) ){
    
     mxInitGPU();
    const mwSize *dimsCPUA;
    dimsCPUA=mxGetDimensions(INPUTDENSEA);
    
    int  numARows = (int)dimsCPUA[0]; /* gets number of rows of A */
    int  numAColumns = (int)dimsCPUA[1]; /* gets number of columns of A */
   
    const mwSize *dimsCPUB;
    dimsCPUB=mxGetDimensions(INPUTSPARSEB);
    
    int  numBRows = (int)dimsCPUB[0]; /* gets number of rows of B */
    int  numBColumns = (int)dimsCPUB[1]; /* gets number of columns of B */
   if ( (numBColumns!= 1) ) {
   
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                        "Invalid input to MEX file, second argument must be a sparse column vector.");
             
    }
    
    if ( (numAColumns!= numBRows) ) {
    
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                        "Invalid input to MEX file, column number of dense matrix(first argument) must be equal to row numbers of sparse vector(second argument).");
             
    }
    
        mxComplexDouble*  al= (mxComplexDouble *)mxGetComplexDoubles(ALPHA);
       const hipDoubleComplex alpha = make_hipDoubleComplex(al[0].real, al[0].imag);
       //mxComplexDouble*  bl= (mxComplexDouble *)mxGetComplexDoubles(BETA);
       const hipDoubleComplex beta = make_hipDoubleComplex(0.0, 0.0);
    
   hipDoubleComplex *h_A_dense1;
   h_A_dense1 = (hipDoubleComplex *)mxGetComplexDoubles(INPUTDENSEA);
    
    mwIndex nnz2;
 
    nnz2 = *(mxGetJc(INPUTSPARSEB) + numBColumns);

    int nnzB= static_cast<int> (nnz2);
    

    mxArray *row_sortB =mxCreateNumericMatrix(nnzB, 1, mxINT32_CLASS, mxREAL);
    int *pointerrowB = (int *)mxGetInt32s(row_sortB);
   
    Ir_DataGetSetIXY(INPUTSPARSEB , pointerrowB, nnzB);
    

   
    hipDoubleComplex  *pointervalB = (hipDoubleComplex *)mxGetComplexDoubles(INPUTSPARSEB);
            
   size_t pivot_dimensionsrowB[1] = {nnzB};
   
   size_t pivot_dimensionsvalueB[1] = {nnzB};    
      mxGPUArray *row_sortBB = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionsrowB, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);  
          
        int *xrow_sortB=(int *)mxGPUGetData(row_sortBB);
 gpuErrchk(hipMemcpy(xrow_sortB, pointerrowB, nnzB * sizeof(*xrow_sortB), hipMemcpyHostToDevice));
       
      mxGPUArray *val_sortBB = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionsvalueB, mxDOUBLE_CLASS, mxCOMPLEX, MX_GPU_DO_NOT_INITIALIZE);  
          
        hipDoubleComplex *xval_sortB=(hipDoubleComplex*)mxGPUGetData(val_sortBB);
 gpuErrchk(hipMemcpy(xval_sortB, pointervalB, nnzB * sizeof(*xval_sortB), hipMemcpyHostToDevice));    
   
         
         mxDestroyArray(row_sortB);  
          
	  
		  
     hipsparseHandle_t handle;    cusparseSafeCall(hipsparseCreate(&handle));

	hipsparseMatDescr_t descrA;      cusparseSafeCall(hipsparseCreateMatDescr(&descrA));
	hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE);        
    
   	
    size_t pivot_dimensionsvalueV[1] = {numBRows};

    mxGPUArray *DB_dense = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionsvalueV, mxDOUBLE_CLASS, mxCOMPLEX, MX_GPU_DO_NOT_INITIALIZE);
    hipDoubleComplex  *d_B_dense = (hipDoubleComplex *)mxGPUGetData(DB_dense);
     cusparseSafeCall(hipsparseZsctr(handle, nnzB, 
              xval_sortB, 
              xrow_sortB, d_B_dense, 
              HIPSPARSE_INDEX_BASE_ONE)); 
	
	      mxGPUDestroyGPUArray(row_sortBB);
          mxGPUDestroyGPUArray(val_sortBB);
		  
		  
    int nnzA = 0;                            // --- Number of nonzero elements in dense matrix A
	const int lda = numARows;
	//int *d_nnzPerVectorA;    gpuErrchk(hipMalloc(&d_nnzPerVectorA, numARows * sizeof(*d_nnzPerVectorA)));
	size_t pivot_pervect[1] = {numARows};
    mxGPUArray *PerVect = mxGPUCreateGPUArray(1, (mwSize*) pivot_pervect, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
	int *d_nnzPerVectorA = (int*)mxGPUGetData(PerVect);
	
	//hipDoubleComplex *d_A_dense;  gpuErrchk(hipMalloc(&d_A_dense, numARows * numAColumns * sizeof(*d_A_dense)));
	
	  size_t pivot_dimensionsvalueDA[2] = {numARows, numAColumns};
      mxGPUArray *OUTMA = mxGPUCreateGPUArray(2, (mwSize*) pivot_dimensionsvalueDA, mxDOUBLE_CLASS, mxCOMPLEX, MX_GPU_DO_NOT_INITIALIZE);
      hipDoubleComplex *d_A_dense = (hipDoubleComplex *)mxGPUGetData(OUTMA);
	
	gpuErrchk(hipMemcpy(d_A_dense, h_A_dense1, numARows * numAColumns * sizeof(*d_A_dense), hipMemcpyHostToDevice));
	cusparseSafeCall(hipsparseZnnz(handle, HIPSPARSE_DIRECTION_ROW, numARows, numAColumns, descrA, d_A_dense, lda, d_nnzPerVectorA, &nnzA));      
       // hipDoubleComplex *d_A;           // gpuErrchk(hipMalloc(&d_A, nnzA * sizeof(*d_A)));
		//int *d_A_RowIndices;   // gpuErrchk(hipMalloc(&d_A_RowIndices, (numARows + 1) * sizeof(*d_A_RowIndices)));
		//int *d_A_ColIndices;   // gpuErrchk(hipMalloc(&d_A_ColIndices, nnzA * sizeof(*d_A_ColIndices)));
   size_t pivot_dimensA[1] = {nnzA};
   size_t pivot_dimensROW_A[1] = {numARows+1};
   size_t pivot_dimensCOL_A[1] = {nnzA};
   
   
   mxGPUArray *A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensA, mxDOUBLE_CLASS, mxCOMPLEX, MX_GPU_DO_NOT_INITIALIZE);
    hipDoubleComplex  *d_A = (hipDoubleComplex *)mxGPUGetData(A);
   mxGPUArray * ROW_A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensROW_A, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_A_RowIndices = (int *)mxGPUGetData(ROW_A);
   mxGPUArray * COL_A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensCOL_A, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_A_ColIndices = (int *)mxGPUGetData(COL_A);
	

		cusparseSafeCall(hipsparseZdense2csr(handle, numARows, numAColumns, descrA, d_A_dense, lda, d_nnzPerVectorA, d_A, d_A_RowIndices, d_A_ColIndices));       
		  mxGPUDestroyGPUArray(OUTMA);
        //gpuErrchk(hipFree(d_nnzPerVectorA));
          mxGPUDestroyGPUArray(PerVect);
          
   
    
    size_t pivot_dimensionsvalue[1] = {numARows};

    mxGPUArray *VAL = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionsvalue, mxDOUBLE_CLASS, mxCOMPLEX, MX_GPU_INITIALIZE_VALUES);
    hipDoubleComplex  *VALOUT = (hipDoubleComplex *)mxGPUGetData(VAL);

 cusparseSafeCall(hipsparseZcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
               numARows, numAColumns, nnzA, &alpha, 
               descrA, 
               d_A, 
               d_A_RowIndices, d_A_ColIndices,
               d_B_dense, &beta, 
               VALOUT));
            
          mxGPUDestroyGPUArray(A);  
          mxGPUDestroyGPUArray(ROW_A); 
          mxGPUDestroyGPUArray(COL_A);            
          mxGPUDestroyGPUArray(DB_dense);
          
  OUTPUTMATRIX = mxGPUCreateMxArrayOnGPU(VAL);             

        mxGPUDestroyGPUArray(VAL);

        hipsparseDestroyMatDescr(descrA);   
		hipsparseDestroy(handle); 
		
    }
    else{
        mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Incorrect input arguments! %s\n");    
        }
    
 }
        //
    else{
        mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Incorrect input arguments! %s\n");    
        }

}
