
/*
 * This CUDA-Cusparse code can handle/work with  any type of the input mxArrays, 
 * GPUarray or standard matlab CPU array as input {prhs[0]/prhs[1] := mxGPUArray or CPU Array}[double/complex double]
 * Sparse/Dense matrix-sparse/dense matrix addition   Z=CuMatlab_add(Sparse/Dense(X),Sparse/Dense(Y), alpha, beta).
 * Z= alpha*X+beta*Y
 * Developed at UCL, Institute of Neurology, 12 Queen Square, WC1N 3AR, London
 * Wellcome Trust Centre for Neuroimaging
 * Part of the project SPM(http://www.fil.ion.ucl.ac.uk/spm)
 * Copyright 2018
 * Kevin Bronik
 */

#include "matrix.h"
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <hipsparse.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "SPARSEHELPER.h"
#include "ERRORCHK.h"
#include <omp.h>

// Input Arguments
#define	INPUTSPARSEA   prhs[0]
#define	INPUTDENSEB   prhs[1]
#define	ALPHA   prhs[2]
#define	BETA    prhs[3]
// Output Arguments
#define	OUTPUTMATRIX  plhs[0]



  
    
extern "C" static void mexCuMatlab_sparseSDC(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
 int nDevices;
hipError_t errCode =hipGetDeviceCount(&nDevices); 
//int nDevices;
//hipGetDeviceCount(&nDevices);

if (errCode != hipSuccess){
printf("Error! No CUDA devices found! \n");
return;
}

    char const * const InputErrMsg = "Invalid input to MEX file, number of input arguments must be four.";
    char const * const OutputErrMsg = "Invalid output to MEX file, number of output arguments must be one.";
   if ((nrhs!=4)) {
        mexErrMsgIdAndTxt("MATLAB:mexatexit:invalidInput", InputErrMsg);
    }
   if ((nlhs!=1)) {
        mexErrMsgIdAndTxt("MATLAB:mexatexit:invalidInput", OutputErrMsg);
    }
 char *input_buf0;
 input_buf0 = mxArrayToString(INPUTSPARSEA);

      if ((mxIsChar(INPUTSPARSEA))){
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Input(FIRST ARGUMENT) must be array, or gpuArray object not  %s\n",input_buf0);
    }
    
 char *input_buf1;
 input_buf1 = mxArrayToString(INPUTDENSEB);

      if ((mxIsChar(INPUTDENSEB))){
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Input(SECOND ARGUMENT) must be array, or gpuArray object not  %s\n",input_buf1);
    } 
 char *input_buf2;
 input_buf2 = mxArrayToString(ALPHA);

      if ((mxIsChar(ALPHA))){
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Input(THIRD ARGUMENT) must be scalar not  %s\n",input_buf2);
    } 
 char *input_buf3;
 input_buf3 = mxArrayToString(BETA);

      if ((mxIsChar(BETA))){
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Input(FOURTH ARGUMENT) must be scalar not  %s\n",input_buf3);
    }


if (mxIsGPUArray(INPUTSPARSEA) && mxIsGPUArray(INPUTDENSEB)) {
    
    mxGPUArray const *INPUTSPARSEGPUA;
    mxGPUArray const *INPUTDENSEGPUB;
    
    
    /* Initialize the MathWorks GPU API. */
    mxInitGPU();
    INPUTSPARSEGPUA = mxGPUCreateFromMxArray(INPUTSPARSEA);
    INPUTDENSEGPUB = mxGPUCreateFromMxArray(INPUTDENSEB);
    
   
	
    if((mxGPUIsSparse(INPUTSPARSEGPUA))&& (!mxGPUIsSparse(INPUTDENSEGPUB)) ){
        
    const mwSize *dimsGPUSA;
    dimsGPUSA=mxGPUGetDimensions(INPUTSPARSEGPUA);
    int numARows, numAColumns;
    numARows = (int)dimsGPUSA[0]; /* gets number of rows of A */
    numAColumns = (int)dimsGPUSA[1]; /* gets number of columns of A */
    
    const mwSize *dimsGPUSB;
    dimsGPUSB=mxGPUGetDimensions(INPUTDENSEGPUB);
    int numBRows, numBColumns;
    numBRows = (int)dimsGPUSB[0]; /* gets number of rows of B */
    numBColumns = (int)dimsGPUSB[1]; /* gets number of columns of B */
    if ( (numARows!= numBRows) && (numAColumns != numBColumns) ) {
		 mxGPUDestroyGPUArray(INPUTSPARSEGPUA);
         mxGPUDestroyGPUArray(INPUTDENSEGPUB);
		
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Invalid input to MEX file, row/column numbers of sparse matrix(first argument) must be equal to row/column numbers of dense matrix(second argument).");
             
    }
        mxComplexDouble*  al= (mxComplexDouble *)mxGetComplexDoubles(ALPHA);
       const hipDoubleComplex alpha = make_hipDoubleComplex(al[0].real, al[0].imag);
       mxComplexDouble*  bl= (mxComplexDouble *)mxGetComplexDoubles(BETA);
       const hipDoubleComplex beta = make_hipDoubleComplex(bl[0].real, bl[0].imag);
    
    hipDoubleComplex const *d_B_dense;
   d_B_dense = (hipDoubleComplex const *)(mxGPUGetDataReadOnly(INPUTDENSEGPUB));
    
    

    mwIndex nnz1;
     mxArray * VLSXY1 = mxGPUCreateMxArrayOnCPU(INPUTSPARSEGPUA);
    nnz1 = *(mxGetJc(VLSXY1) + numAColumns);
    int nnzA = (int)nnz1;
    
   
   mxArray *  ROW_SORTA = mxCreateNumericMatrix(nnzA, 1,mxINT32_CLASS, mxREAL);
    int *ROWSORTA  = (int *)mxGetInt32s(ROW_SORTA);
       SetIr_Data(VLSXY1, ROWSORTA);
    
   mxArray *  COL_SORTA = mxCreateNumericMatrix(nnzA, 1, mxINT32_CLASS, mxREAL);
    int  *COLSORTA = (int *)mxGetInt32s(COL_SORTA);
          SetJc_Int(VLSXY1, COLSORTA);
      
   
    hipDoubleComplex  *VALSORTA = (hipDoubleComplex *)mxGetComplexDoubles(VLSXY1);
           
    hipsparseHandle_t handle;    cusparseSafeCall(hipsparseCreate(&handle));

	hipsparseMatDescr_t descrA;      cusparseSafeCall(hipsparseCreateMatDescr(&descrA));
	hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE);


    int nnzB = 0;                            // --- Number of nonzero elements in dense matrix B
	const int ldb = numBRows;
		//int *d_nnzPerVectorB;    gpuErrchk(hipMalloc(&d_nnzPerVectorB, numBRows * sizeof(*d_nnzPerVectorB)));
	size_t pivot_pervect[1] = {numBRows};
    mxGPUArray *PerVect = mxGPUCreateGPUArray(1, (mwSize*) pivot_pervect, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
	int *d_nnzPerVectorB = (int*)mxGPUGetData(PerVect);
	//hipDoubleComplex *d_B_dense;  gpuErrchk(hipMalloc(&d_B_dense, numBRows * numBColumns * sizeof(*d_B_dense)));
	//gpuErrchk(hipMemcpy(d_B_dense, h_B_dense1, numBRows * numBColumns * sizeof(*d_B_dense), hipMemcpyHostToDevice));
	cusparseSafeCall(hipsparseZnnz(handle, HIPSPARSE_DIRECTION_ROW, numBRows, numBColumns, descrA, d_B_dense, ldb, d_nnzPerVectorB, &nnzB));
    
           //hipDoubleComplex *d_B;            gpuErrchk(hipMalloc(&d_B, nnzB * sizeof(*d_B)));
		//int *d_B_RowIndices;    gpuErrchk(hipMalloc(&d_B_RowIndices, (numBRows + 1) * sizeof(*d_B_RowIndices)));
		//int *d_B_ColIndices;    gpuErrchk(hipMalloc(&d_B_ColIndices, nnzB * sizeof(*d_B_ColIndices)));
		
   size_t pivot_dimensB[1] = {nnzB};
   size_t pivot_dimensROW_B[1] = {numBRows+1};
   size_t pivot_dimensCOL_B[1] = {nnzB};
   
   
   mxGPUArray *B = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensB, mxDOUBLE_CLASS, mxCOMPLEX, MX_GPU_DO_NOT_INITIALIZE);
    hipDoubleComplex  *d_B = (hipDoubleComplex *)mxGPUGetData(B);
   mxGPUArray * ROW_B = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensROW_B, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_B_RowIndices = (int *)mxGPUGetData(ROW_B);
   mxGPUArray * COL_B = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensCOL_B, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_B_ColIndices = (int *)mxGPUGetData(COL_B);

		cusparseSafeCall(hipsparseZdense2csr(handle, numBRows, numBColumns, descrA, d_B_dense, ldb, d_nnzPerVectorB, d_B, d_B_RowIndices, d_B_ColIndices));
        //gpuErrchk(hipFree(d_B_dense));       
        mxGPUDestroyGPUArray(PerVect);
		mxGPUDestroyGPUArray(INPUTSPARSEGPUA);
        mxGPUDestroyGPUArray(INPUTDENSEGPUB);
		
		
	//hipDoubleComplex *d_A;            gpuErrchk(hipMalloc(&d_A, nnzA * sizeof(*d_A)));
	//int *d_A_RowIndices;    gpuErrchk(hipMalloc(&d_A_RowIndices, (numARows + 1) * sizeof(*d_A_RowIndices)));
	//int *d_A_ColIndices;    gpuErrchk(hipMalloc(&d_A_ColIndices, nnzA * sizeof(*d_A_ColIndices)));
	//int *d_cooRowIndA;       gpuErrchk(hipMalloc(&d_cooRowIndA, nnzA * sizeof(*d_cooRowIndA)));
	
	
   size_t pivot_dimensA[1] = {nnzA};
   size_t pivot_dimensROW_A[1] = {numARows+1};
   size_t pivot_dimensCOL_A[1] = {nnzA};
   size_t pivot_dimensCOO_A[1] = {nnzA};
   
   mxGPUArray *A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensA, mxDOUBLE_CLASS, mxCOMPLEX, MX_GPU_DO_NOT_INITIALIZE);
    hipDoubleComplex  *d_A = (hipDoubleComplex *)mxGPUGetData(A);
   mxGPUArray * ROW_A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensROW_A, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_A_RowIndices = (int *)mxGPUGetData(ROW_A);
   mxGPUArray * COL_A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensCOL_A, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_A_ColIndices = (int *)mxGPUGetData(COL_A);
    mxGPUArray * COO_A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensCOO_A, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_cooRowIndA = (int *)mxGPUGetData(COO_A); 
	
	
	// --- Descriptor for sparse matrix A
	gpuErrchk(hipMemcpy(d_A, VALSORTA, nnzA * sizeof(*d_A), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_A_ColIndices, COLSORTA, nnzA * sizeof(*d_A_ColIndices), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_cooRowIndA, ROWSORTA, nnzA * sizeof(*d_cooRowIndA), hipMemcpyHostToDevice));


	int *Pa = NULL;
	void *pBuffera = NULL;
	size_t pBufferSizeInBytesa = 0;
	hipsparseXcoosort_bufferSizeExt(handle, numARows, numAColumns,
		nnzA,
		d_cooRowIndA,
		d_A_ColIndices, &pBufferSizeInBytesa);

	gpuErrchk(hipMalloc(&pBuffera, sizeof(char)*pBufferSizeInBytesa));
	gpuErrchk(hipMalloc(&Pa, sizeof(int)*nnzA));
	hipsparseCreateIdentityPermutation(handle, nnzA, Pa);
	cusparseSafeCall(hipsparseXcoosortByRow(handle, numARows, numAColumns,
		nnzA,
		d_cooRowIndA,
		d_A_ColIndices,
		Pa,
		pBuffera));

	cusparseSafeCall(hipsparseZgthr(handle, nnzA, d_A, d_A, Pa, HIPSPARSE_INDEX_BASE_ZERO));

	cusparseSafeCall(hipsparseXcoo2csr(handle,
		d_cooRowIndA,
		nnzA,
		numARows,
		d_A_RowIndices,
		HIPSPARSE_INDEX_BASE_ONE));

        
        
	cusparseSafeCall(hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST));

	int nnzC = 0;
	int baseC = 0;
// nnzTotalDevHostPtr points to host memory
    int *nnzTotalDevHostPtr = &nnzC;
	//int *d_C_RowIndices;    gpuErrchk(hipMalloc((void **)&d_C_RowIndices, sizeof(int)*(numARows + 1)));
   size_t pivot_dimensROW_C[1] = {numARows+1};
   
   mxGPUArray * ROW_C = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensROW_C, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_C_RowIndices = (int *)mxGPUGetData(ROW_C);
	// where op ( A ) , op ( B ) and C are m�k (numARows�numAColumns), l�n(numBRows�numBColumns), and m�n(numARows�numBColumns) sparse matrices m=l, k=n
	cusparseSafeCall(hipsparseXcsrgeamNnz(handle, numARows, numBColumns,
		descrA, nnzA, d_A_RowIndices, d_A_ColIndices,
		descrA, nnzB, d_B_RowIndices, d_B_ColIndices,
		descrA, d_C_RowIndices, nnzTotalDevHostPtr ));
       
        
        
     if (NULL != nnzTotalDevHostPtr){
    nnzC = *nnzTotalDevHostPtr;
    }else{
    hipMemcpy(&nnzC, d_C_RowIndices+numARows, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&baseC, d_C_RowIndices, sizeof(int), hipMemcpyDeviceToHost);
    nnzC -= baseC;
     }
	gpuErrchk(hipFree(pBuffera));
	gpuErrchk(hipFree(Pa));
	//gpuErrchk(hipFree(d_cooRowIndA));
    
   if (nnzC==0) {
          
         OUTPUTMATRIX = mxCreateSparse(numARows,numBColumns,0,mxCOMPLEX);
         
         return;
             
    } 
    
    
   size_t pivot_dimensionsrow[1] = {nnzC};
   size_t pivot_dimensionscolumn[1] = {numBColumns+1}; 
   size_t pivot_dimensionsvalue[1] = {nnzC};
   mxGPUArray * ROW_SORTC = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionsrow, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *ROWSORTC = (int *)mxGPUGetData(ROW_SORTC);
   mxGPUArray * COL_SORTC = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionscolumn, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *COLSORTC = (int *)mxGPUGetData(COL_SORTC);
    mxGPUArray *VAL_SORTC = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionsvalue, mxDOUBLE_CLASS, mxCOMPLEX, MX_GPU_DO_NOT_INITIALIZE);
    hipDoubleComplex  *VALSORTC = (hipDoubleComplex *)mxGPUGetData(VAL_SORTC);
   mwSize nnzm=(mwSize)nnzC;
   
   OUTPUTMATRIX = mxCreateSparse(numARows,numBColumns,nnzm,mxCOMPLEX);
    
   // hipDoubleComplex *d_C;          gpuErrchk(hipMalloc((void **)&d_C, sizeof(hipDoubleComplex)*(nnzC)));
   // int *d_C_ColIndices;   gpuErrchk(hipMalloc((void **)&d_C_ColIndices, sizeof(int)*(nnzC)));
  	
	size_t pivot_dimensC[1] = {nnzC};
    size_t pivot_dimensCOL_C[1] = {nnzC};
   
   
   mxGPUArray *C = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensC, mxDOUBLE_CLASS, mxCOMPLEX, MX_GPU_DO_NOT_INITIALIZE);
    hipDoubleComplex  *d_C = (hipDoubleComplex *)mxGPUGetData(C);
   mxGPUArray * COL_C = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensCOL_C, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_C_ColIndices = (int *)mxGPUGetData(COL_C); 

	
   	cusparseSafeCall(hipsparseZcsrgeam(handle, numARows, numBColumns, &alpha,
		descrA, nnzA,
		d_A, d_A_RowIndices, d_A_ColIndices, &beta,
		descrA, nnzB,
		d_B, d_B_RowIndices, d_B_ColIndices,
		descrA,
		d_C, d_C_RowIndices, d_C_ColIndices));
        
	//gpuErrchk(hipFree(d_A));
	//gpuErrchk(hipFree(d_A_RowIndices));
	//gpuErrchk(hipFree(d_A_ColIndices));
	
	//gpuErrchk(hipFree(d_B));
	//gpuErrchk(hipFree(d_B_RowIndices));
	//gpuErrchk(hipFree(d_B_ColIndices));
	
	mxGPUDestroyGPUArray(A);
    mxGPUDestroyGPUArray(ROW_A);
    mxGPUDestroyGPUArray(COL_A);
    mxGPUDestroyGPUArray(COO_A);
    mxGPUDestroyGPUArray(B);
    mxGPUDestroyGPUArray(ROW_B);
    mxGPUDestroyGPUArray(COL_B);
    
	//hipDoubleComplex *d_value_csc;  gpuErrchk(hipMalloc((void **)&d_value_csc, sizeof(hipDoubleComplex)*(nnzC)));
	//int *d_row_csc;       gpuErrchk(hipMalloc((void **)&d_row_csc, sizeof(int)*(nnzC)));
	//int *d_col_csc;       gpuErrchk(hipMalloc((void **)&d_col_csc, sizeof(int)*(numBColumns + 1)));

	cusparseSafeCall(hipsparseZcsr2csc(handle, numARows, numBColumns, nnzC, d_C, d_C_RowIndices, d_C_ColIndices, VALSORTC, ROWSORTC, COLSORTC, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ONE));
    /*

	int *Pc = NULL;
	void *pBufferc = NULL;
	size_t pBufferSizeInBytesc = 0;
	hipsparseXcscsort_bufferSizeExt(handle, numARows, numBColumns,
		nnzC,
		d_col_csc,
		d_row_csc, &pBufferSizeInBytesc);
   
	gpuErrchk(hipMalloc(&pBufferc, sizeof(char)*pBufferSizeInBytesc));
	gpuErrchk(hipMalloc(&Pc, sizeof(int)*nnzC));
	hipsparseCreateIdentityPermutation(handle, nnzC, Pc);
	cusparseSafeCall(hipsparseXcscsort(handle, numARows, numBColumns,
		nnzC,
		descrA,
		d_col_csc,
		d_row_csc,
		Pc,
		pBufferc));

	cusparseSafeCall(hipsparseZgthr(handle, nnzC, d_value_csc, d_value_csc, Pc, HIPSPARSE_INDEX_BASE_ZERO));
    */
  
	//gpuErrchk(hipMemcpy(VALSORTC, d_value_csc, sizeof(hipDoubleComplex)* nnzC, hipMemcpyDeviceToHost));
	//gpuErrchk(hipMemcpy(ROWSORTC, d_row_csc, sizeof(int)* (nnzC), hipMemcpyDeviceToHost));
	//gpuErrchk(hipMemcpy(COLSORTC, d_col_csc, sizeof(int)* (numBColumns + 1), hipMemcpyDeviceToHost));

    //gpuErrchk(hipFree(pBufferc));
	//gpuErrchk(hipFree(Pc));
  
    
   mxArray *RS= mxGPUCreateMxArrayOnCPU(ROW_SORTC);
   int * rs= (int *)mxGetInt32s(RS);
   mxArray *CS= mxGPUCreateMxArrayOnCPU(COL_SORTC);
   int * cs= (int *)mxGetInt32s(CS);

    
      mwIndex *irs,*jcs;
  
        irs = static_cast<mwIndex *> (mxMalloc (nnzC * sizeof(mwIndex)));
       int i;
	   #pragma omp parallel for shared(nnzC) private(i)
         for ( i = 0; i < nnzC; ++i) {
           irs[i] = static_cast<mwIndex> (rs[i])-1;  
            }
      
      jcs = static_cast<mwIndex *> (mxMalloc ((numBColumns+1) * sizeof(mwIndex)));
      int nc1= numBColumns+1;
       #pragma omp parallel for shared(nc1) private(i)
            for (i = 0; i < nc1; ++i) {
           jcs[i] = static_cast<mwIndex> (cs[i])-1;
            }
             
        mxComplexDouble* PRS = (mxComplexDouble*) mxMalloc (nnzC * sizeof(mxComplexDouble));
        gpuErrchk(hipMemcpy(PRS, VALSORTC, nnzC * sizeof(mxComplexDouble), hipMemcpyDeviceToHost));          
             
         
   
   
        mxFree (mxGetJc (OUTPUTMATRIX)) ;
        mxFree (mxGetIr (OUTPUTMATRIX)) ;
        mxFree (mxGetComplexDoubles (OUTPUTMATRIX)) ;
    
        mxSetIr(OUTPUTMATRIX, (mwIndex *)irs);
        mxSetJc(OUTPUTMATRIX, (mwIndex *)jcs);
        int m= mxSetComplexDoubles(OUTPUTMATRIX, (mxComplexDouble*)PRS);
        if ( m==0) {
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "the function is unsuccessful, either mxArray is not an unshared mxDOUBLE_CLASS array, or the data is not allocated with mxCalloc.");
             
         }
    
    
	//gpuErrchk(hipFree(d_C));
	//gpuErrchk(hipFree(d_C_RowIndices));
	//gpuErrchk(hipFree(d_C_ColIndices));

	//gpuErrchk(hipFree(d_value_csc));
	//gpuErrchk(hipFree(d_row_csc));
	//gpuErrchk(hipFree(d_col_csc));

      mxGPUDestroyGPUArray(C);
      mxGPUDestroyGPUArray(ROW_C);
      mxGPUDestroyGPUArray(COL_C);


         mxDestroyArray(VLSXY1);
         mxGPUDestroyGPUArray(VAL_SORTC);
         mxGPUDestroyGPUArray(ROW_SORTC);
         mxGPUDestroyGPUArray(COL_SORTC);
         mxDestroyArray(RS);
         mxDestroyArray(CS);

         
         mxDestroyArray(COL_SORTA);
         mxDestroyArray(ROW_SORTA);

        hipsparseDestroyMatDescr(descrA);   
		hipsparseDestroy(handle);
        
        }
    
        else{
        mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Incorrect input arguments! %s\n");    
        }
    
   }
     
////////////////////////////////////////////////////////////////////////////////////  
    else if (!(mxIsGPUArray(INPUTSPARSEA)) && !(mxIsGPUArray(INPUTDENSEB))){
   
     // if ((mxGetClassID(INPUTSPARSEA) != mxDOUBLE_CLASS) || (mxGetClassID(INPUTSPARSEB) != mxDOUBLE_CLASS)) {
       //  mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
           //     "Invalid input to MEX file, input(FIRST and SECOND  ARGUMENTS) must be  hipDoubleComplex precision.");
             
   // }
    if((mxIsSparse(INPUTSPARSEA))&& (!mxIsSparse(INPUTDENSEB)) ){
    
     mxInitGPU();
    const mwSize *dimsCPUA;
    dimsCPUA=mxGetDimensions(INPUTSPARSEA);
    
    int  numARows = (int)dimsCPUA[0]; /* gets number of rows of A */
    int  numAColumns = (int)dimsCPUA[1]; /* gets number of columns of A */
   
    const mwSize *dimsCPUB;
    dimsCPUB=mxGetDimensions(INPUTDENSEB);
    
    int  numBRows = (int)dimsCPUB[0]; /* gets number of rows of B */
    int  numBColumns = (int)dimsCPUB[1]; /* gets number of columns of B */
    if ( (numARows!= numBRows) && (numAColumns != numBColumns) ) {
		
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Invalid input to MEX file, row/column numbers of sparse matrix(first argument) must be equal to row/column numbers of dense matrix(second argument).");
             
    }
        mxComplexDouble*  al= (mxComplexDouble *)mxGetComplexDoubles(ALPHA);
       const hipDoubleComplex alpha = make_hipDoubleComplex(al[0].real, al[0].imag);
       mxComplexDouble*  bl= (mxComplexDouble *)mxGetComplexDoubles(BETA);
       const hipDoubleComplex beta = make_hipDoubleComplex(bl[0].real, bl[0].imag);
    
    mwIndex nnz1;
 
    nnz1 = *(mxGetJc(INPUTSPARSEA) + numAColumns);
    int nnzA = (int)nnz1;
    
    hipDoubleComplex *h_B_dense1;
   h_B_dense1 = (hipDoubleComplex *)mxGetComplexDoubles(INPUTDENSEB);
   
   mxArray *  ROW_SORTA = mxCreateNumericMatrix(nnzA, 1,mxINT32_CLASS, mxREAL);
    int *ROWSORTA  = (int *)mxGetInt32s(ROW_SORTA);
       
     SetIr_Data(INPUTSPARSEA, ROWSORTA);
    
   mxArray *  COL_SORTA = mxCreateNumericMatrix(nnzA, 1, mxINT32_CLASS, mxREAL);
    int  *COLSORTA = (int *)mxGetInt32s(COL_SORTA);
          
      SetJc_Int(INPUTSPARSEA, COLSORTA);
      
   
    hipDoubleComplex  *VALSORTA = (hipDoubleComplex *)mxGetComplexDoubles(INPUTSPARSEA);
           
    hipsparseHandle_t handle;    cusparseSafeCall(hipsparseCreate(&handle));

	hipsparseMatDescr_t descrA;      cusparseSafeCall(hipsparseCreateMatDescr(&descrA));
	hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE);


    int nnzB = 0;                            // --- Number of nonzero elements in dense matrix B
	const int ldb = numBRows;
	//int *d_nnzPerVectorB;    gpuErrchk(hipMalloc(&d_nnzPerVectorB, numBRows * sizeof(*d_nnzPerVectorB)));
	
	size_t pivot_pervect[1] = {numBRows};
    mxGPUArray *PerVect = mxGPUCreateGPUArray(1, (mwSize*) pivot_pervect, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
	int *d_nnzPerVectorB = (int*)mxGPUGetData(PerVect);
	
	//hipDoubleComplex *d_B_dense;  gpuErrchk(hipMalloc(&d_B_dense, numBRows * numBColumns * sizeof(*d_B_dense)));
	
	  size_t pivot_dimensionsvalueDB[2] = {numBRows, numBColumns};
      mxGPUArray *OUTMB = mxGPUCreateGPUArray(2, (mwSize*) pivot_dimensionsvalueDB, mxDOUBLE_CLASS, mxCOMPLEX, MX_GPU_DO_NOT_INITIALIZE);
      hipDoubleComplex  *d_B_dense = (hipDoubleComplex *)mxGPUGetData(OUTMB);	
	
	gpuErrchk(hipMemcpy(d_B_dense, h_B_dense1, numBRows * numBColumns * sizeof(*d_B_dense), hipMemcpyHostToDevice));
	cusparseSafeCall(hipsparseZnnz(handle, HIPSPARSE_DIRECTION_ROW, numBRows, numBColumns, descrA, d_B_dense, ldb, d_nnzPerVectorB, &nnzB));
    
    
        //hipDoubleComplex *d_B;            gpuErrchk(hipMalloc(&d_B, nnzB * sizeof(*d_B)));
		//int *d_B_RowIndices;    gpuErrchk(hipMalloc(&d_B_RowIndices, (numBRows + 1) * sizeof(*d_B_RowIndices)));
		//int *d_B_ColIndices;    gpuErrchk(hipMalloc(&d_B_ColIndices, nnzB * sizeof(*d_B_ColIndices)));
		
   size_t pivot_dimensB[1] = {nnzB};
   size_t pivot_dimensROW_B[1] = {numBRows+1};
   size_t pivot_dimensCOL_B[1] = {nnzB};
   
   
   mxGPUArray *B = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensB, mxDOUBLE_CLASS, mxCOMPLEX, MX_GPU_DO_NOT_INITIALIZE);
    hipDoubleComplex  *d_B = (hipDoubleComplex *)mxGPUGetData(B);
   mxGPUArray * ROW_B = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensROW_B, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_B_RowIndices = (int *)mxGPUGetData(ROW_B);
   mxGPUArray * COL_B = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensCOL_B, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_B_ColIndices = (int *)mxGPUGetData(COL_B);		
	

		cusparseSafeCall(hipsparseZdense2csr(handle, numBRows, numBColumns, descrA, d_B_dense, ldb, d_nnzPerVectorB, d_B, d_B_RowIndices, d_B_ColIndices));
        mxGPUDestroyGPUArray(OUTMB);      
        mxGPUDestroyGPUArray(PerVect);
	//hipDoubleComplex *d_A;            gpuErrchk(hipMalloc(&d_A, nnzA * sizeof(*d_A)));
	//int *d_A_RowIndices;    gpuErrchk(hipMalloc(&d_A_RowIndices, (numARows + 1) * sizeof(*d_A_RowIndices)));
	//int *d_A_ColIndices;    gpuErrchk(hipMalloc(&d_A_ColIndices, nnzA * sizeof(*d_A_ColIndices)));
	//int *d_cooRowIndA;       gpuErrchk(hipMalloc(&d_cooRowIndA, nnzA * sizeof(*d_cooRowIndA)));
	
   size_t pivot_dimensA[1] = {nnzA};
   size_t pivot_dimensROW_A[1] = {numARows+1};
   size_t pivot_dimensCOL_A[1] = {nnzA};
   size_t pivot_dimensCOO_A[1] = {nnzA};
   
   mxGPUArray *A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensA, mxDOUBLE_CLASS, mxCOMPLEX, MX_GPU_DO_NOT_INITIALIZE);
    hipDoubleComplex  *d_A = (hipDoubleComplex *)mxGPUGetData(A);
   mxGPUArray * ROW_A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensROW_A, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_A_RowIndices = (int *)mxGPUGetData(ROW_A);
   mxGPUArray * COL_A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensCOL_A, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_A_ColIndices = (int *)mxGPUGetData(COL_A);
    mxGPUArray * COO_A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensCOO_A, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_cooRowIndA = (int *)mxGPUGetData(COO_A); 
	// --- Descriptor for sparse matrix A
	gpuErrchk(hipMemcpy(d_A, VALSORTA, nnzA * sizeof(*d_A), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_A_ColIndices, COLSORTA, nnzA * sizeof(*d_A_ColIndices), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_cooRowIndA, ROWSORTA, nnzA * sizeof(*d_cooRowIndA), hipMemcpyHostToDevice));


	int *Pa = NULL;
	void *pBuffera = NULL;
	size_t pBufferSizeInBytesa = 0;
	hipsparseXcoosort_bufferSizeExt(handle, numARows, numAColumns,
		nnzA,
		d_cooRowIndA,
		d_A_ColIndices, &pBufferSizeInBytesa);

	gpuErrchk(hipMalloc(&pBuffera, sizeof(char)*pBufferSizeInBytesa));
	gpuErrchk(hipMalloc(&Pa, sizeof(int)*nnzA));
	hipsparseCreateIdentityPermutation(handle, nnzA, Pa);
	cusparseSafeCall(hipsparseXcoosortByRow(handle, numARows, numAColumns,
		nnzA,
		d_cooRowIndA,
		d_A_ColIndices,
		Pa,
		pBuffera));

	cusparseSafeCall(hipsparseZgthr(handle, nnzA, d_A, d_A, Pa, HIPSPARSE_INDEX_BASE_ZERO));

	cusparseSafeCall(hipsparseXcoo2csr(handle,
		d_cooRowIndA,
		nnzA,
		numARows,
		d_A_RowIndices,
		HIPSPARSE_INDEX_BASE_ONE));

        
        
	cusparseSafeCall(hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST));

	int nnzC = 0;
	int baseC = 0;
// nnzTotalDevHostPtr points to host memory
    int *nnzTotalDevHostPtr = &nnzC;
	//int *d_C_RowIndices;    gpuErrchk(hipMalloc((void **)&d_C_RowIndices, sizeof(int)*(numARows + 1)));
	
   size_t pivot_dimensROW_C[1] = {numARows+1};
   
   mxGPUArray * ROW_C = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensROW_C, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_C_RowIndices = (int *)mxGPUGetData(ROW_C);
	// where op ( A ) , op ( B ) and C are m�k (numARows�numAColumns), l�n(numBRows�numBColumns), and m�n(numARows�numBColumns) sparse matrices m=l, k=n
	cusparseSafeCall(hipsparseXcsrgeamNnz(handle, numARows, numBColumns,
		descrA, nnzA, d_A_RowIndices, d_A_ColIndices,
		descrA, nnzB, d_B_RowIndices, d_B_ColIndices,
		descrA, d_C_RowIndices, nnzTotalDevHostPtr ));
       
        
        
     if (NULL != nnzTotalDevHostPtr){
    nnzC = *nnzTotalDevHostPtr;
    }else{
    hipMemcpy(&nnzC, d_C_RowIndices+numARows, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&baseC, d_C_RowIndices, sizeof(int), hipMemcpyDeviceToHost);
    nnzC -= baseC;
     }
	gpuErrchk(hipFree(pBuffera));
	gpuErrchk(hipFree(Pa));
	//gpuErrchk(hipFree(d_cooRowIndA));    
   
   if (nnzC==0) {
          
         OUTPUTMATRIX = mxCreateSparse(numARows,numBColumns,0,mxCOMPLEX);
         
         return;
             
    } 
    
      
   
   size_t pivot_dimensionsrow[1] = {nnzC};
   size_t pivot_dimensionscolumn[1] = {numBColumns+1}; 
   size_t pivot_dimensionsvalue[1] = {nnzC};
   mxGPUArray * ROW_SORTC = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionsrow, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *ROWSORTC = (int *)mxGPUGetData(ROW_SORTC);
   mxGPUArray * COL_SORTC = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionscolumn, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *COLSORTC = (int *)mxGPUGetData(COL_SORTC);
    mxGPUArray *VAL_SORTC = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionsvalue, mxDOUBLE_CLASS, mxCOMPLEX, MX_GPU_DO_NOT_INITIALIZE);
    hipDoubleComplex  *VALSORTC = (hipDoubleComplex *)mxGPUGetData(VAL_SORTC);
   mwSize nnzm=(mwSize)nnzC;
   
   OUTPUTMATRIX = mxCreateSparse(numARows,numBColumns,nnzm,mxCOMPLEX);
    
   // hipDoubleComplex *d_C;          gpuErrchk(hipMalloc((void **)&d_C, sizeof(hipDoubleComplex)*(nnzC)));
   // int *d_C_ColIndices;   gpuErrchk(hipMalloc((void **)&d_C_ColIndices, sizeof(int)*(nnzC)));
  	
	
	size_t pivot_dimensC[1] = {nnzC};
    size_t pivot_dimensCOL_C[1] = {nnzC};
   
   
   mxGPUArray *C = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensC, mxDOUBLE_CLASS, mxCOMPLEX, MX_GPU_DO_NOT_INITIALIZE);
    hipDoubleComplex  *d_C = (hipDoubleComplex *)mxGPUGetData(C);
   mxGPUArray * COL_C = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensCOL_C, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_C_ColIndices = (int *)mxGPUGetData(COL_C);	
	
	    
   	cusparseSafeCall(hipsparseZcsrgeam(handle, numARows, numBColumns, &alpha,
		descrA, nnzA,
		d_A, d_A_RowIndices, d_A_ColIndices, &beta,
		descrA, nnzB,
		d_B, d_B_RowIndices, d_B_ColIndices,
		descrA,
		d_C, d_C_RowIndices, d_C_ColIndices));
	//gpuErrchk(hipFree(d_A));
	//gpuErrchk(hipFree(d_A_RowIndices));
	//gpuErrchk(hipFree(d_A_ColIndices));
	
	//gpuErrchk(hipFree(d_B));
	//gpuErrchk(hipFree(d_B_RowIndices));
	//gpuErrchk(hipFree(d_B_ColIndices));
	
	mxGPUDestroyGPUArray(A);
    mxGPUDestroyGPUArray(ROW_A);
    mxGPUDestroyGPUArray(COL_A);
    mxGPUDestroyGPUArray(COO_A);
    mxGPUDestroyGPUArray(B);
    mxGPUDestroyGPUArray(ROW_B);
    mxGPUDestroyGPUArray(COL_B);
    
	//hipDoubleComplex *d_value_csc;  gpuErrchk(hipMalloc((void **)&d_value_csc, sizeof(hipDoubleComplex)*(nnzC)));
	//int *d_row_csc;       gpuErrchk(hipMalloc((void **)&d_row_csc, sizeof(int)*(nnzC)));
	//int *d_col_csc;       gpuErrchk(hipMalloc((void **)&d_col_csc, sizeof(int)*(numBColumns + 1)));

	cusparseSafeCall(hipsparseZcsr2csc(handle, numARows, numBColumns, nnzC, d_C, d_C_RowIndices, d_C_ColIndices, VALSORTC, ROWSORTC, COLSORTC, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ONE));
    /*

	int *Pc = NULL;
	void *pBufferc = NULL;
	size_t pBufferSizeInBytesc = 0;
	hipsparseXcscsort_bufferSizeExt(handle, numARows, numBColumns,
		nnzC,
		d_col_csc,
		d_row_csc, &pBufferSizeInBytesc);
   
	gpuErrchk(hipMalloc(&pBufferc, sizeof(char)*pBufferSizeInBytesc));
	gpuErrchk(hipMalloc(&Pc, sizeof(int)*nnzC));
	hipsparseCreateIdentityPermutation(handle, nnzC, Pc);
	cusparseSafeCall(hipsparseXcscsort(handle, numARows, numBColumns,
		nnzC,
		descrA,
		d_col_csc,
		d_row_csc,
		Pc,
		pBufferc));

	cusparseSafeCall(hipsparseZgthr(handle, nnzC, d_value_csc, d_value_csc, Pc, HIPSPARSE_INDEX_BASE_ZERO));
    */
  
	//gpuErrchk(hipMemcpy(VALSORTC, d_value_csc, sizeof(hipDoubleComplex)* nnzC, hipMemcpyDeviceToHost));
	//gpuErrchk(hipMemcpy(ROWSORTC, d_row_csc, sizeof(int)* (nnzC), hipMemcpyDeviceToHost));
	//gpuErrchk(hipMemcpy(COLSORTC, d_col_csc, sizeof(int)* (numBColumns + 1), hipMemcpyDeviceToHost));

    //gpuErrchk(hipFree(pBufferc));
	//gpuErrchk(hipFree(Pc));
  
    
   mxArray *RS= mxGPUCreateMxArrayOnCPU(ROW_SORTC);
   int * rs= (int *)mxGetInt32s(RS);
   mxArray *CS= mxGPUCreateMxArrayOnCPU(COL_SORTC);
   int * cs= (int *)mxGetInt32s(CS);

    
      mwIndex *irs,*jcs;
  
        irs = static_cast<mwIndex *> (mxMalloc (nnzC * sizeof(mwIndex)));
       int i;
	   #pragma omp parallel for shared(nnzC) private(i)
         for ( i = 0; i < nnzC; ++i) {
           irs[i] = static_cast<mwIndex> (rs[i])-1;  
            }
      
      jcs = static_cast<mwIndex *> (mxMalloc ((numBColumns+1) * sizeof(mwIndex)));
      int nc1= numBColumns+1;
       #pragma omp parallel for shared(nc1) private(i)
            for (i = 0; i < nc1; ++i) {
           jcs[i] = static_cast<mwIndex> (cs[i])-1;
            }
             
        mxComplexDouble* PRS = (mxComplexDouble*) mxMalloc (nnzC * sizeof(mxComplexDouble));
        gpuErrchk(hipMemcpy(PRS, VALSORTC, nnzC * sizeof(mxComplexDouble), hipMemcpyDeviceToHost));          
             
          
   
   
        mxFree (mxGetJc (OUTPUTMATRIX)) ;
        mxFree (mxGetIr (OUTPUTMATRIX)) ;
        mxFree (mxGetComplexDoubles (OUTPUTMATRIX)) ;
    
        mxSetIr(OUTPUTMATRIX, (mwIndex *)irs);
        mxSetJc(OUTPUTMATRIX, (mwIndex *)jcs);
        int m= mxSetComplexDoubles(OUTPUTMATRIX, (mxComplexDouble*)PRS);
        if ( m==0) {
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "the function is unsuccessful, either mxArray is not an unshared mxDOUBLE_CLASS array, or the data is not allocated with mxCalloc.");
             
         }
    
    
	//gpuErrchk(hipFree(d_C));
	//gpuErrchk(hipFree(d_C_RowIndices));
	//gpuErrchk(hipFree(d_C_ColIndices));

	//gpuErrchk(hipFree(d_value_csc));
	//gpuErrchk(hipFree(d_row_csc));
	//gpuErrchk(hipFree(d_col_csc));

      mxGPUDestroyGPUArray(C);
      mxGPUDestroyGPUArray(ROW_C);
      mxGPUDestroyGPUArray(COL_C);


         //mxDestroyArray(VLSXY1);
         mxGPUDestroyGPUArray(VAL_SORTC);
         mxGPUDestroyGPUArray(ROW_SORTC);
         mxGPUDestroyGPUArray(COL_SORTC);
         mxDestroyArray(RS);
         mxDestroyArray(CS);
 
         
         mxDestroyArray(COL_SORTA);
         mxDestroyArray(ROW_SORTA);

        hipsparseDestroyMatDescr(descrA);   
		hipsparseDestroy(handle);
    }
    else{
        mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Incorrect input arguments! %s\n");    
        }
    
 }
        //
    else{
        mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Incorrect input arguments! %s\n");    
        }

}
