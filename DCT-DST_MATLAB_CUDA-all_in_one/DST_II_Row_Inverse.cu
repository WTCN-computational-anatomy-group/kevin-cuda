#include "hip/hip_runtime.h"
/*
 * Inverse Discrete Sine Transform in row wise (DST two)
 * DST_II_Row_Inverse
 * This CUDA code can handle/work with  any type of the input mxArrays, 
 * GPUarray or standard matlab CPU array as input {prhs[0] := mxGPUArray or CPU Array}
 * gpuArray output, B=DST_II_Row_Inverse(A)=mexFunction(A).
 * Developed at UCL, Institute of Neurology, 12 Queen Square, WC1N 3AR, London
 * Wellcome Trust Centre for Neuroimaging
 * Part of the project SPM(http://www.fil.ion.ucl.ac.uk/spm)
 * Copyright 2018
 * Kevin Bronik
 */
#include "matrix.h"
#include "DST_II_Row_Inverse.cuh"
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}
#define TILE_DIM 16

#define DEFAULT_DIM 32                     // Tile dimension 
#define 	DELTA(i, j)   ((i==j)?1:0)

//const double  PI_d = 3.141592653589793238462643383279502884; //pi



__global__ void DSTII_Row__InverseKernel(double  *A, double  *C,
	int numARows, int numAColumns,
	int numCRows, int numCColumns)
{
	double CValue = 0.0;
	const double  PI_d = 3.141592653589793238462643383279502884; //pi
	int Row = blockIdx.y*TILE_DIM + threadIdx.y;
	int Col = blockIdx.x*TILE_DIM + threadIdx.x;

	__shared__ double As[TILE_DIM][TILE_DIM];
	__shared__ double Bs[TILE_DIM][TILE_DIM];

	for (int k = 0; k < (TILE_DIM + numAColumns - 1) / TILE_DIM; k++) {

		if (k*TILE_DIM + threadIdx.x < numAColumns && Row < numARows)	{ As[threadIdx.y][threadIdx.x] = A[Row*numAColumns + k*TILE_DIM + threadIdx.x]; }
		else													{ As[threadIdx.y][threadIdx.x] = 0.0; }
                                                                             //Bs[threadIdx.y][threadIdx.x] = cos(((2 * Col + 1) / (2.0 * numAColumns))*PI_d*(threadIdx.y + k*TILE_DIM))*sqrt(1.0 / (1 + DELTA(1, (threadIdx.y + k*TILE_DIM) + 1)))*sqrt(2.0 / numAColumns);
		if (k*TILE_DIM + threadIdx.y < numAColumns && Col < numAColumns)	{ Bs[threadIdx.y][threadIdx.x] = sin((((threadIdx.y + k*TILE_DIM) + 1)*PI_d*(Col + 0.5)) / (numAColumns))*sqrt(2.0 / (numAColumns))*sqrt(1.0 / (1 + DELTA(numAColumns, (threadIdx.y + k*TILE_DIM) + 1))); }
		//Bs[threadIdx.y][threadIdx.x] = B[(k*TILE_DIM + threadIdx.y)*BCols + Col];
		else													{ Bs[threadIdx.y][threadIdx.x] = 0.0; }

		__syncthreads();

		for (int n = 0; n < TILE_DIM; ++n) { CValue += As[threadIdx.y][n] * Bs[n][threadIdx.x]; }

		__syncthreads();
	}

	if (Row < numCRows && Col < numCColumns) { C[((blockIdx.y * blockDim.y + threadIdx.y)*numCColumns) + (blockIdx.x*blockDim.x) + threadIdx.x] = CValue; }

}


// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
extern "C" void  CalculateTransformDSTInverseRowTwo(double * A, double * C, int numARows,
	int numAColumns, int numCRows, int numCColumns)
{


	double * hostA = A; // The A matrix
	//double * hostB = B; // The B matrix
	double * hostC = C; // The output C matrix
	//float * hostComputedC;
	double * deviceA;
	//double * deviceB;
	double * deviceC;

	//hostA = (float *)malloc(sizeof(float)*numARows*numAColumns);
	//hostB = (float *)malloc(sizeof(float)*numBRows*numBColumns);
	
	


	// Setting numCRows and numCColumns
	numCRows = numARows;
	numCColumns = numAColumns;

	//hostC = (float *)malloc(sizeof(float)*numCRows*numCColumns);
	//hostComputedC = (float *)malloc(sizeof(float)*numCRows*numCColumns);

	// Allocating GPU memory
	gpuErrchk(hipMalloc((void **)&deviceA, sizeof(double)*numARows*numAColumns));
	//hipMalloc((void **)&deviceB, sizeof(double)*numBRows*numBColumns);
	gpuErrchk(hipMalloc((void **)&deviceC, sizeof(double)*numCRows*numCColumns));
	
	//thrust::device_ptr< double >dev_ptr_A(deviceA);
	//thrust::device_ptr< double >dev_ptr_C(deviceC);

	// Copy memory to the GPU
	gpuErrchk(hipMemcpy(deviceA, hostA, sizeof(double)*numARows*numAColumns, hipMemcpyHostToDevice));
	//hipMemcpy(deviceB, hostB, sizeof(double)*numBRows*numBColumns, hipMemcpyHostToDevice);

	/////////////////////////////////////////////////////////
	dim3 dimBlock(TILE_DIM, TILE_DIM, 1);
	dim3 dimGrid;

	dimGrid.x = (numCColumns + dimBlock.x - 1) / dimBlock.x;
	dimGrid.y = (numCRows + dimBlock.y - 1) / dimBlock.y;
	DSTII_Row__InverseKernel << <dimGrid, dimBlock >> >(deviceA, deviceC, numARows, numAColumns, numCRows, numCColumns);
	//matrixMultiplyShared << <dimGrid, dimBlock >> >(thrust::raw_pointer_cast(&dev_ptr_A[0]), thrust::raw_pointer_cast(&dev_ptr_C[0]), numARows, numAColumns, numCRows, numCColumns);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());

	// Copy the results in GPU memory back to the CPU
	gpuErrchk(hipMemcpy(hostC, deviceC, sizeof(double)*numCRows*numCColumns, hipMemcpyDeviceToHost));

	C = hostC;

	//thrust::device_free(dev_ptr_A);
	//thrust::device_free(dev_ptr_C);
	gpuErrchk(hipFree(deviceA));
	//hipFree(deviceB);
	gpuErrchk(hipFree(deviceC));
	return;
    
}
