#include "hip/hip_runtime.h"

/*
 * Discrete Cosine/Sine Transform(DCT/DST and IDCT/IDST one to four-all in one)
 * DCT/DST and IDCT/IDST I ---> IV
 * This CUDA code can handle/work with  any type of the input mxArrays, 
 * GPUarray or standard matlab CPU array as input {prhs[0] := mxGPUArray or CPU Array}
 * gpuArray output, B=Discrete_Transform(A, , type of Transform (sine or cosine), type of Transform(direct/inverse), type of DCT/DST or IDCT/IDST, dimensions).
 * Developed at UCL, Institute of Neurology, 12 Queen Square, WC1N 3AR, London
 * Wellcome Trust Centre for Neuroimaging
 * Part of the project SPM(http://www.fil.ion.ucl.ac.uk/spm)
 * Copyright 2018
 * Kevin Bronik
 */

#include "matrix.h"
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include "Discrete_Transform_kernel.cuh"
#include "DCT_I_Column.cuh"
#include "DCT_I_Row.cuh"
#include "DCT_I_Column_Inverse.cuh"
#include "DCT_I_Row_Inverse.cuh"
#include "DCT_II_Row.cuh"
#include "DCT_II_Row_Inverse.cuh"
#include "DCT_II_Column.cuh"
#include "DCT_II_Column_Inverse.cuh"
#include "DCT_III_Row.cuh"
#include "DCT_III_Row_Inverse.cuh"
#include "DCT_III_Column.cuh"
#include "DCT_III_Column_Inverse.cuh"
#include "DCT_IV_Row.cuh"
#include "DCT_IV_Row_Inverse.cuh"
#include "DCT_IV_Column.cuh"
#include "DCT_IV_Column_Inverse.cuh"
#include "DST_I_Column.cuh"
#include "DST_I_Row.cuh"
#include "DST_I_Column_Inverse.cuh"
#include "DST_I_Row_Inverse.cuh"
#include "DST_II_Row.cuh"
#include "DST_II_Row_Inverse.cuh"
#include "DST_II_Column.cuh"
#include "DST_II_Column_Inverse.cuh"
#include "DST_III_Row.cuh"
#include "DST_III_Row_Inverse.cuh"
#include "DST_III_Column.cuh"
#include "DST_III_Column_Inverse.cuh"
#include "DST_IV_Row.cuh"
#include "DST_IV_Row_Inverse.cuh"
#include "DST_IV_Column.cuh"
#include "DST_IV_Column_Inverse.cuh"
//#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#define DEFAULT_DIM 32   
#define 	DELTA(i, j)   ((i==j)?1:0)
//#define TILE_DIM 16
unsigned int TILE_DIM=16;
// DCT
extern "C" void  CalculateTransformDCTColumnOne(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDCTInverseColumnOne(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDCTRowOne(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDCTInverseRowOne(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);    
    
extern "C" void  CalculateTransformDCTRowTwo(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDCTInverseRowTwo(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDCTColumnTwo(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDCTInverseColumnTwo(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDCTColumnThree(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns); 
    
extern "C" void  CalculateTransformDCTInverseColumnThree(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDCTRowThree(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDCTInverseRowThree(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDCTColumnFour(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);  
    
extern "C" void  CalculateTransformDCTInverseColumnFour(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDCTRowFour(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDCTInverseRowFour(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns); 
    
    // DST
extern "C" void  CalculateTransformDSTColumnOne(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDSTInverseColumnOne(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDSTRowOne(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDSTInverseRowOne(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);    
    
extern "C" void  CalculateTransformDSTRowTwo(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDSTInverseRowTwo(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDSTColumnTwo(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDSTInverseColumnTwo(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDSTColumnThree(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns); 
    
extern "C" void  CalculateTransformDSTInverseColumnThree(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDSTRowThree(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDSTInverseRowThree(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDSTColumnFour(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);  
    
extern "C" void  CalculateTransformDSTInverseColumnFour(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDSTRowFour(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDSTInverseRowFour(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns); 
    
void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
 int nDevices;
hipError_t errCode =hipGetDeviceCount(&nDevices); 
//int nDevices;
//hipGetDeviceCount(&nDevices);

if (errCode != hipSuccess){
printf("Error! No CUDA devices found! \n");
return;
}

//Discrete_Transform(x, 'cosine', 'direct', 'one' , 'column')
char row[] = "row";
char column[] = "column";
char one[] = "one";
char two[] = "two";
char three[] = "three";
char four[] = "four";

char direct[] = "direct";
char inverse[] = "inverse";
char cosine[] = "cosine";
char sine[] = "sine";
    //char const * const errId = "parallel:gpu:mexGPUExample:InvalidInput";
    char const * const InputErrMsg = "Invalid input to MEX file, input(FIRST ARGUMENT) must be single precision (float), and the number of input arguments must be five.";
    
        if ((nrhs!=5)) {
        mexErrMsgIdAndTxt("MATLAB:mexatexit:invalidInput", InputErrMsg);
    }

char *input_buf0;
 input_buf0 = mxArrayToString(prhs[0]);
 char *input_buf1;
 input_buf1 = mxArrayToString(prhs[1]);
char *input_buf2;
 input_buf2 = mxArrayToString(prhs[2]);
  char *input_buf3;
 input_buf3 = mxArrayToString(prhs[3]);
char *input_buf4;
 input_buf4 = mxArrayToString(prhs[4]);
      if ((mxIsChar(prhs[0]))){
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Input(FIRST ARGUMENT) must be array, or gpuArray object not  %s\n",input_buf0);
    }
     if (!(mxIsChar(prhs[1]))){
        mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Input(SECOND ARGUMENT) must be of type string.\n.");
    }
      if (!(mxIsChar(prhs[2]))){
        mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Input(THIRD ARGUMENT) must be of type string.\n.");
    }
        if (!(mxIsChar(prhs[3]))){
        mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Input(FOURTH ARGUMENT) must be of type string.\n.");
    }
      if (!(mxIsChar(prhs[4]))){
        mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Input(FIFTH ARGUMENT) must be of type string.\n.");
    } 
    ///////// input_buf0=FIRST, SECOND, THIRD, FOURTH, FIFTH  
  if ((strcmp (cosine,input_buf1) != 0) &&(strcmp (sine,input_buf1) != 0) )
{
    mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Input(SECOND ARGUMENT) must be 'cosine' or 'sine'  not  %s\n",input_buf1);
   }
    
  if ((strcmp (direct,input_buf2) != 0)&& (strcmp (inverse,input_buf2) != 0) )
{
        mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Input(THIRD ARGUMENT) must be 'direct' or 'inverse' not  %s\n",input_buf2);
    }
    

   if ((strcmp (one,input_buf3) != 0)&& (strcmp (two,input_buf3) != 0) && (strcmp (three,input_buf3) != 0) && (strcmp (four,input_buf3) != 0))
{
                mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Input(FOURTH ARGUMENT) must be 'one' or 'two' or 'three' or 'four' not  %s\n",input_buf3);

    }
       if ((strcmp (column,input_buf4) != 0)&&(strcmp (row,input_buf4) != 0))
{  
            mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Input(FIFTH ARGUMENT) must be 'column' or 'row' not  %s\n",input_buf4);
    }
///////  
 //mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
             //   "Input(FIFTH ARGUMENT) must be 'column' or 'row'.\n.");
    
 if (strcmp (cosine,input_buf1) == 0)
{

 if (strcmp (direct,input_buf2) == 0)
{   
  if (strcmp (column,input_buf4) == 0)
{   
///  input standard GPUarray 
    if (mxIsGPUArray(prhs[0])) {
		//mexErrMsgIdAndTxt(errId, errMsg);
           /* Declare all variables.*/
    mxGPUArray const *A;
    mxGPUArray *B;
    float const *d_A;
    float *d_B;
    int numARows, numAColumns, numCRows,  numCColumns;
    


    /* Initialize the MathWorks GPU API. */
    mxInitGPU();
    hipError_t error;
    int devID = 0;
    // get number of SMs on this GPU
    error = hipGetDevice(&devID);
    hipDeviceProp_t deviceProp;
     error = hipGetDeviceProperties(&deviceProp, devID);
      if (error != hipSuccess)
      {
          printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
          exit(EXIT_FAILURE);
      }
    int TILEDIM = (deviceProp.major < 2) ? 16 : 32;

    A = mxGPUCreateFromMxArray(prhs[0]);
    const mwSize *dims;
   dims=mxGPUGetDimensions(A);
   numARows = (int)dims[0]; /* gets number of rows of A */
   numAColumns = (int)dims[1]; /* gets number of columns of A */
 
 		
		numCRows = numARows;

		numCColumns = numAColumns;
  if (numARows==1)
 {   
 printf("Attention, this is a row vector, please try Discrete Cosine Transform in row wise \n");
 return;
 }
 
    char const * const errId = "parallel:gpu:mexGPUExample:InvalidInput";
    char const * const errMsg = "Invalid input to MEX file, input(FIRST ARGUMENT) must be single precision (float).";

    if (mxGPUGetClassID(A) != mxSINGLE_CLASS) {
        mexErrMsgIdAndTxt(errId, errMsg);
    }


    d_A = (float const *)(mxGPUGetDataReadOnly(A));
    
    
    B = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(A),
                            mxGPUGetDimensions(A),
                            mxGPUGetClassID(A),
                            mxGPUGetComplexity(A),
                            MX_GPU_DO_NOT_INITIALIZE);
    d_B = (float *)(mxGPUGetData(B));

    
    dim3 dimBlock;
	dim3 dimGrid;
     switch (TILEDIM){
        
        case 16:
            
     TILE_DIM= TILEDIM;
	 dimBlock.x=TILE_DIM;
	 dimBlock.y=TILE_DIM;
     dimBlock.z=1;
	dimGrid.x = (numCColumns + dimBlock.x - 1) / dimBlock.x;
	dimGrid.y = (numCRows + dimBlock.y - 1) / dimBlock.y;
 if (strcmp (one,input_buf3) == 0)
{
     
   DCTI_Column_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
 }  
 if (strcmp (two,input_buf3) == 0)
{
     
   DCTII_Column_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
 }
 if (strcmp (three,input_buf3) == 0)
{
     
   DCTIII_Column_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
 }
 if (strcmp (four,input_buf3) == 0)
{
     
   DCTIV_Column_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
 }

  //	hipError_t err1 = hipPeekAtLastError();//To capture last error in function call

	//hipDeviceSynchronize();//To synchronize the device

      plhs[0] = mxGPUCreateMxArrayOnGPU(B);
      
    mxGPUDestroyGPUArray(A);
    
    mxGPUDestroyGPUArray(B);
    
               case 32:
            
     TILE_DIM= TILEDIM;
	 dimBlock.x=TILE_DIM;
	 dimBlock.y=TILE_DIM;
     dimBlock.z=1;
	dimGrid.x = (numCColumns + dimBlock.x - 1) / dimBlock.x;
	dimGrid.y = (numCRows + dimBlock.y - 1) / dimBlock.y;
    
     if (strcmp (one,input_buf3) == 0)
{
     
   DCTI_Column_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
 }  
 if (strcmp (two,input_buf3) == 0)
{
     
   DCTII_Column_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
 }
 if (strcmp (three,input_buf3) == 0)
{
     
   DCTIII_Column_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
 }
 if (strcmp (four,input_buf3) == 0)
{
     
   DCTIV_Column_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
 }

  //	hipError_t err1 = hipPeekAtLastError();//To capture last error in function call

	//hipDeviceSynchronize();//To synchronize the device

      plhs[0] = mxGPUCreateMxArrayOnGPU(B);
      
    mxGPUDestroyGPUArray(A);
    
    mxGPUDestroyGPUArray(B);
    
    
      } 
	}
///  input standard array 

    else if (!(mxIsGPUArray(prhs[0]))){
            
   
   if (mxGetClassID(prhs[0]) != mxSINGLE_CLASS) {
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Invalid input to MEX file, input(FIRST ARGUMENT) must be single precision (float).");
    }       
  	int numARows = (int)mxGetM(prhs[0]); 		// number of rows in the matrix A
	int numAColumns = (int)mxGetN(prhs[0]); 	// number of columns in the matrix A
	int numCRows;		// number of rows in the matrix C (you have to set this)
	int numCColumns;	// number of columns in the matrix C (you have to set this)
    	
		numCRows = numARows;

		numCColumns = numAColumns;
        
 if (numARows==1)
 {   
 printf("Attention, this is a row vector, please try Discrete Cosine Transform in row wise \n");
 return;
 }
    mxInitGPU();
    

	float  * hostA ; // The A matrix
	hostA = (float *)mxGetData(prhs[0]);
    
    plhs[0] = mxCreateNumericMatrix(numCRows, numCColumns, mxSINGLE_CLASS, mxREAL);
    float  *pointer =(float*) mxGetPr(plhs[0]);
    //CalculateTransform(float * A, float * C, int numARows, int numAColumns, int numCRows, int numCColumns)  
     if (strcmp (one,input_buf3) == 0)
    {
     
       CalculateTransformDCTColumnOne(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
	
    }
     if (strcmp (two,input_buf3) == 0)
    {
     
       CalculateTransformDCTColumnTwo(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
	
    }
     if (strcmp (three,input_buf3) == 0)
    {
     
       CalculateTransformDCTColumnThree(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
	
    }
     if (strcmp (four,input_buf3) == 0)
    {
     
       CalculateTransformDCTColumnFour(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
	
    }

	//free(hostB);
    } 
    } // Column
///////////////////////////    
     if (strcmp (row,input_buf4) == 0)
{
   ///  input standard GPUarray 
    if (mxIsGPUArray(prhs[0])) {
		
   mxGPUArray const *A;
    

    mxGPUArray *B;
    float const *d_A;
   
    float *d_B;
   
    int numARows, numAColumns, numCRows,  numCColumns;
    
  

    /* Initialize the MathWorks GPU API. */
    mxInitGPU();
    hipError_t error;
    int devID = 0;
    // get number of SMs on this GPU
    error = hipGetDevice(&devID);
    hipDeviceProp_t deviceProp;
     error = hipGetDeviceProperties(&deviceProp, devID);
      if (error != hipSuccess)
      {
          printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
          exit(EXIT_FAILURE);
      }
    int TILEDIM = (deviceProp.major < 2) ? 16 : 32;

    A = mxGPUCreateFromMxArray(prhs[0]);
   const mwSize *dims;
 dims=mxGPUGetDimensions(A);
 numARows = (int)dims[0]; /* gets number of rows of A */
 numAColumns = (int)dims[1]; /* gets number of columns of A */
 
 if (numAColumns==1)
 {   
 printf("Attention, this is a column vector, please try Discrete Cosine Transform in column wise \n");
 return;
 }
 
 
    numCRows = numARows;
	numCColumns = numAColumns;

   char const * const errId = "parallel:gpu:mexGPUExample:InvalidInput";
   char const * const errMsg = "Invalid input to MEX file, input(FIRST ARGUMENT) must be single precision (float).";
    if (mxGPUGetClassID(A) != mxSINGLE_CLASS) {
        mexErrMsgIdAndTxt(errId, errMsg);
    }


    d_A = (float const *)(mxGPUGetDataReadOnly(A));
    
    
    B = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(A),
                            mxGPUGetDimensions(A),
                            mxGPUGetClassID(A),
                            mxGPUGetComplexity(A),
                            MX_GPU_DO_NOT_INITIALIZE);
    d_B = (float *)(mxGPUGetData(B));

    dim3 dimBlock;
	dim3 dimGrid;
     switch (TILEDIM){
        
        case 16:
            
     TILE_DIM= TILEDIM;
	 dimBlock.x=TILE_DIM;
	 dimBlock.y=TILE_DIM;
     dimBlock.z=1;
	dimGrid.x = (numCColumns + dimBlock.x - 1) / dimBlock.x;
	dimGrid.y = (numCRows + dimBlock.y - 1) / dimBlock.y;
 if (strcmp (one,input_buf3) == 0)
{
    
   DCTI_Row_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
}
 if (strcmp (two,input_buf3) == 0)
{
    
   DCTII_Row_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
}
 if (strcmp (three,input_buf3) == 0)
{
    
   DCTIII_Row_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
}
 if (strcmp (four,input_buf3) == 0)
{
    
   DCTIV_Row_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
}
      plhs[0] = mxGPUCreateMxArrayOnGPU(B);
      
    mxGPUDestroyGPUArray(A);
   
    mxGPUDestroyGPUArray(B);
    
                   case 32:
            
     TILE_DIM= TILEDIM;
	 dimBlock.x=TILE_DIM;
	 dimBlock.y=TILE_DIM;
     dimBlock.z=1;
	dimGrid.x = (numCColumns + dimBlock.x - 1) / dimBlock.x;
	dimGrid.y = (numCRows + dimBlock.y - 1) / dimBlock.y;
    
     if (strcmp (one,input_buf3) == 0)
{
    
   DCTI_Row_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
}
 if (strcmp (two,input_buf3) == 0)
{
    
   DCTII_Row_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
}
 if (strcmp (three,input_buf3) == 0)
{
    
   DCTIII_Row_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
}
 if (strcmp (four,input_buf3) == 0)
{
    
   DCTIV_Row_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
}
      plhs[0] = mxGPUCreateMxArrayOnGPU(B);
      
    mxGPUDestroyGPUArray(A);
   
    mxGPUDestroyGPUArray(B);
    
    
    }
     
	}
///  input standard array 

    else if (!(mxIsGPUArray(prhs[0]))){
            
   if (mxGetClassID(prhs[0]) != mxSINGLE_CLASS) {
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Invalid input to MEX file, input(FIRST ARGUMENT) must be single precision (float).");
    }        
  	int numARows = (int)mxGetM(prhs[0]); 		// number of rows in the matrix A
	int numAColumns = (int)mxGetN(prhs[0]); 	// number of columns in the matrix A
	
	int numCRows;		// number of rows in the matrix C (you have to set this)
	int numCColumns;	// number of columns in the matrix C (you have to set this)
	
	numCRows = numARows;

	numCColumns = numAColumns;
	

	float  * hostA ; // The A matrix
	
	
 if (numAColumns==1)
 {   
 printf("Attention, this is a column vector, please try Discrete Cosine Transform in column wise \n");
 return;
 }

mxInitGPU();
    

	hostA = (float *)mxGetData(prhs[0]);
   
    plhs[0] = mxCreateNumericMatrix(numCRows, numCColumns, mxSINGLE_CLASS, mxREAL);
    float  *pointer = (float*)mxGetPr(plhs[0]);
     if (strcmp (one,input_buf3) == 0)
{  
      CalculateTransformDCTRowOne(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
      
      } 
       if (strcmp (two,input_buf3) == 0)
{  
      CalculateTransformDCTRowTwo(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
      
      }
       if (strcmp (three,input_buf3) == 0)
{  
      CalculateTransformDCTRowThree(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
      
      }
       if (strcmp (four,input_buf3) == 0)
{  
      CalculateTransformDCTRowFour(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
      
      }
  
  //one
	//free(hostB);
    } 

    }//row
    ///////////////////////////
}// direct

 if (strcmp (inverse,input_buf2) == 0)
{
    if (strcmp (column,input_buf4) == 0)
{      
///  input standard GPUarray 
    if (mxIsGPUArray(prhs[0])) {
		//mexErrMsgIdAndTxt(errId, errMsg);
           /* Declare all variables.*/
    mxGPUArray const *A;
    mxGPUArray *B;
    float const *d_A;
    float *d_B;
    int numARows, numAColumns, numCRows,  numCColumns;
    


    /* Initialize the MathWorks GPU API. */
    mxInitGPU();
    hipError_t error;
    int devID = 0;
    // get number of SMs on this GPU
    error = hipGetDevice(&devID);
    hipDeviceProp_t deviceProp;
     error = hipGetDeviceProperties(&deviceProp, devID);
      if (error != hipSuccess)
      {
          printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
          exit(EXIT_FAILURE);
      }
    int TILEDIM = (deviceProp.major < 2) ? 16 : 32;

    A = mxGPUCreateFromMxArray(prhs[0]);
    const mwSize *dims;
   dims=mxGPUGetDimensions(A);
   numARows = (int)dims[0]; /* gets number of rows of A */
   numAColumns = (int)dims[1]; /* gets number of columns of A */
 
 		
		numCRows = numARows;

		numCColumns = numAColumns;
  if (numARows==1)
 {   
 printf("Attention, this is a row vector, please try Inverse Discrete Cosine Transform in row wise \n");
 return;
 }
 
    char const * const errId = "parallel:gpu:mexGPUExample:InvalidInput";
    char const * const errMsg = "Invalid input to MEX file, input(FIRST ARGUMENT) must be single precision (float).";

    if (mxGPUGetClassID(A) != mxSINGLE_CLASS) {
        mexErrMsgIdAndTxt(errId, errMsg);
    }


    d_A = (float const *)(mxGPUGetDataReadOnly(A));
    
    
    B = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(A),
                            mxGPUGetDimensions(A),
                            mxGPUGetClassID(A),
                            mxGPUGetComplexity(A),
                            MX_GPU_DO_NOT_INITIALIZE);
    d_B = (float *)(mxGPUGetData(B));
    dim3 dimBlock;
	dim3 dimGrid;
     switch (TILEDIM){
        
        case 16:
            
     TILE_DIM= TILEDIM;
	 dimBlock.x=TILE_DIM;
	 dimBlock.y=TILE_DIM;
     dimBlock.z=1;
	dimGrid.x = (numCColumns + dimBlock.x - 1) / dimBlock.x;
	dimGrid.y = (numCRows + dimBlock.y - 1) / dimBlock.y;
 if (strcmp (one,input_buf3) == 0)
{
    
   DCTI_Column_Inverse_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
}
  if (strcmp (two,input_buf3) == 0)
{
    
   DCTII_Column_Inverse_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
}
 if (strcmp (three,input_buf3) == 0)
{
    
   DCTIII_Column_Inverse_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
}
 if (strcmp (four,input_buf3) == 0)
{
    
   DCTIV_Column_Inverse_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
}
  //	hipError_t err1 = hipPeekAtLastError();//To capture last error in function call

	//hipDeviceSynchronize();//To synchronize the device

      plhs[0] = mxGPUCreateMxArrayOnGPU(B);
      
    mxGPUDestroyGPUArray(A);
    
    mxGPUDestroyGPUArray(B);
     
    case 32:
            
     TILE_DIM= TILEDIM;
	 dimBlock.x=TILE_DIM;
	 dimBlock.y=TILE_DIM;
     dimBlock.z=1;
	dimGrid.x = (numCColumns + dimBlock.x - 1) / dimBlock.x;
	dimGrid.y = (numCRows + dimBlock.y - 1) / dimBlock.y;
     if (strcmp (one,input_buf3) == 0)
{
    
   DCTI_Column_Inverse_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
}
  if (strcmp (two,input_buf3) == 0)
{
    
   DCTII_Column_Inverse_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
}
 if (strcmp (three,input_buf3) == 0)
{
    
   DCTIII_Column_Inverse_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
}
 if (strcmp (four,input_buf3) == 0)
{
    
   DCTIV_Column_Inverse_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
}
  //	hipError_t err1 = hipPeekAtLastError();//To capture last error in function call

	//hipDeviceSynchronize();//To synchronize the device

      plhs[0] = mxGPUCreateMxArrayOnGPU(B);
      
    mxGPUDestroyGPUArray(A);
    
    mxGPUDestroyGPUArray(B);
    
    
    }
     
	}
///  input standard array 

    else if (!(mxIsGPUArray(prhs[0]))){
  
    if (mxGetClassID(prhs[0]) != mxSINGLE_CLASS) {
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Invalid input to MEX file, input(FIRST ARGUMENT) must be single precision (float).");
    }        
            
  	int numARows = (int)mxGetM(prhs[0]); 		// number of rows in the matrix A
	int numAColumns = (int)mxGetN(prhs[0]); 	// number of columns in the matrix A
	int numCRows;		// number of rows in the matrix C (you have to set this)
	int numCColumns;	// number of columns in the matrix C (you have to set this)
    	
		numCRows = numARows;

		numCColumns = numAColumns;
        
 if (numARows==1)
 {   
 printf("Attention, this is a row vector, please try Inverse Discrete Cosine Transform in row wise \n");
 return;
 }
    mxInitGPU();

	float  * hostA ; // The A matrix
	hostA = (float *)mxGetData(prhs[0]);
    
    plhs[0] = mxCreateNumericMatrix(numCRows, numCColumns, mxSINGLE_CLASS, mxREAL);
    float  *pointer = (float*)mxGetPr(plhs[0]);
    //CalculateTransform(float * A, float * C, int numARows, int numAColumns, int numCRows, int numCColumns) 
     if (strcmp (one,input_buf3) == 0)
   {
  
       CalculateTransformDCTInverseColumnOne(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
	
   }
     if (strcmp (two,input_buf3) == 0)
   {
  
       CalculateTransformDCTInverseColumnTwo(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
	
   }
     if (strcmp (three,input_buf3) == 0)
   {
  
       CalculateTransformDCTInverseColumnThree(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
	
   }
     if (strcmp (four,input_buf3) == 0)
   {
  
       CalculateTransformDCTInverseColumnFour(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
	
   }
	//free(hostB);
    } 
} // Column

    if (strcmp (row,input_buf4) == 0)
{
    
///  input standard GPUarray 
    if (mxIsGPUArray(prhs[0])) {
		//mexErrMsgIdAndTxt(errId, errMsg);
           /* Declare all variables.*/
   mxGPUArray const *A;
   

    mxGPUArray *B;
    float const *d_A;
   
    float *d_B;
   
    int numARows, numAColumns, numCRows,  numCColumns;
    


    /* Initialize the MathWorks GPU API. */
    mxInitGPU();
    hipError_t error;
    int devID = 0;
    // get number of SMs on this GPU
    error = hipGetDevice(&devID);
    hipDeviceProp_t deviceProp;
     error = hipGetDeviceProperties(&deviceProp, devID);
      if (error != hipSuccess)
      {
          printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
          exit(EXIT_FAILURE);
      }
    int TILEDIM = (deviceProp.major < 2) ? 16 : 32;
    

    A = mxGPUCreateFromMxArray(prhs[0]);
    const mwSize *dims;
    dims=mxGPUGetDimensions(A);
    numARows = (int)dims[0]; /* gets number of rows of A */
    numAColumns = (int)dims[1]; /* gets number of columns of A */
 
  if (numAColumns==1)
 {   
 printf("Attention, this is a column vector, please try Inverse Discrete Cosine Transform in column wise \n");
 return;
 }
 
    numCRows = numARows;
	numCColumns = numAColumns;
    char const * const errId = "parallel:gpu:mexGPUExample:InvalidInput";
    char const * const errMsg = "Invalid input to MEX file, input(FIRST ARGUMENT) must be single precision (float).";

    if (mxGPUGetClassID(A) != mxSINGLE_CLASS) {
        mexErrMsgIdAndTxt(errId, errMsg);
    }


    d_A = (float const *)(mxGPUGetDataReadOnly(A));
    
    
    B = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(A),
                            mxGPUGetDimensions(A),
                            mxGPUGetClassID(A),
                            mxGPUGetComplexity(A),
                            MX_GPU_DO_NOT_INITIALIZE);
    d_B = (float *)(mxGPUGetData(B));

    
    dim3 dimBlock;
	dim3 dimGrid;
     switch (TILEDIM){
        
        case 16:
            
     TILE_DIM= TILEDIM;
	 dimBlock.x=TILE_DIM;
	 dimBlock.y=TILE_DIM;
     dimBlock.z=1;
	dimGrid.x = (numCColumns + dimBlock.x - 1) / dimBlock.x;
	dimGrid.y = (numCRows + dimBlock.y - 1) / dimBlock.y;
 if (strcmp (one,input_buf3) == 0)
{
   DCTI_Row__InverseKernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);

   }
 if (strcmp (two,input_buf3) == 0)
{
   DCTII_Row__InverseKernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);

   }
 if (strcmp (three,input_buf3) == 0)
{
   DCTIII_Row__InverseKernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);

   }
 if (strcmp (four,input_buf3) == 0)
{
   DCTIV_Row__InverseKernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);

   }
// one
  //	hipError_t err1 = hipPeekAtLastError();//To capture last error in function call

	//hipDeviceSynchronize();//To synchronize the device

      plhs[0] = mxGPUCreateMxArrayOnGPU(B);
      
    mxGPUDestroyGPUArray(A);
    
    mxGPUDestroyGPUArray(B);
        case 32:
            
     TILE_DIM= TILEDIM;
	 dimBlock.x=TILE_DIM;
	 dimBlock.y=TILE_DIM;
     dimBlock.z=1;
	dimGrid.x = (numCColumns + dimBlock.x - 1) / dimBlock.x;
	dimGrid.y = (numCRows + dimBlock.y - 1) / dimBlock.y;
    
     if (strcmp (one,input_buf3) == 0)
{
   DCTI_Row__InverseKernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);

   }
 if (strcmp (two,input_buf3) == 0)
{
   DCTII_Row__InverseKernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);

   }
 if (strcmp (three,input_buf3) == 0)
{
   DCTIII_Row__InverseKernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);

   }
 if (strcmp (four,input_buf3) == 0)
{
   DCTIV_Row__InverseKernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);

   }
// one
  //	hipError_t err1 = hipPeekAtLastError();//To capture last error in function call

	//hipDeviceSynchronize();//To synchronize the device

      plhs[0] = mxGPUCreateMxArrayOnGPU(B);
      
    mxGPUDestroyGPUArray(A);
    
    mxGPUDestroyGPUArray(B);
    
    
    }
     
	}
///  input standard array 

    else if (!(mxIsGPUArray(prhs[0]))){
            
    if (mxGetClassID(prhs[0]) != mxSINGLE_CLASS) {
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Invalid input to MEX file, input(FIRST ARGUMENT) must be single precision (float).");
    }         
  	int numARows = (int)mxGetM(prhs[0]); 		// number of rows in the matrix A
	int numAColumns = (int)mxGetN(prhs[0]); 	// number of columns in the matrix A
	
	int numCRows;		// number of rows in the matrix C (you have to set this)
	int numCColumns;	// number of columns in the matrix C (you have to set this)
	
	
    if (numAColumns==1)
    {   
    printf("Attention, this is a column vector, please try Inverse Discrete Cosine Transform in column wise \n");
    return;
    }
    mxInitGPU();
  
    numCRows = numARows;
	numCColumns = numAColumns;
	//char const * const errId = "parallel:gpu:DCTTWO:InvalidInput";
	//char const * const errMsg = "Invalid input to MEX file, input(FIRST ARGUMENT) must be single precision (float).";

	float  * hostA ; // The A matrix
	
	
	
	hostA = (float *)mxGetData(prhs[0]);
    
    plhs[0] = mxCreateNumericMatrix(numCRows, numCColumns, mxSINGLE_CLASS, mxREAL);
    float  *pointer =(float*) mxGetPr(plhs[0]);
    
    
	//CalculateTransform(hostA, hostB, hostC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);
     if (strcmp (one,input_buf3) == 0)
{
      CalculateTransformDCTInverseRowOne(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);  
 } 
     if (strcmp (two,input_buf3) == 0)
{
      CalculateTransformDCTInverseRowTwo(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);  
 } 
     if (strcmp (three,input_buf3) == 0)
{
      CalculateTransformDCTInverseRowThree(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);  
 } 
     if (strcmp (four,input_buf3) == 0)
{
      CalculateTransformDCTInverseRowFour(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);  
 } 
//one
   //memcpy(pointer, hostC, numCRows*numCColumns*sizeof(float)); 
   
   // testing
    //    printf("\n  plhs[0]:");
    //    printf("\n");
    //    for (int i = 0; i<numCRows; i++){
	//	for (int j = 0; j<numCColumns; j++){
	//		printf(" %g ", round  (pointer[i * numCColumns + j]));
	//	}
	//	printf("\n");
//	}
	
	//free(hostB);
    }

    }//row

  } // inverse
}
// cosine

//SINE......................................................................................................................................................
 if (strcmp (sine,input_buf1) == 0)
{

 if (strcmp (direct,input_buf2) == 0)
{   
  if (strcmp (column,input_buf4) == 0)
{   
///  input standard GPUarray 
    if (mxIsGPUArray(prhs[0])) {
		//mexErrMsgIdAndTxt(errId, errMsg);
           /* Declare all variables.*/
    mxGPUArray const *A;
    mxGPUArray *B;
    float const *d_A;
    float *d_B;
    int numARows, numAColumns, numCRows,  numCColumns;
    


    /* Initialize the MathWorks GPU API. */
    mxInitGPU();
    hipError_t error;
    int devID = 0;
    // get number of SMs on this GPU
    error = hipGetDevice(&devID);
    hipDeviceProp_t deviceProp;
     error = hipGetDeviceProperties(&deviceProp, devID);
      if (error != hipSuccess)
      {
          printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
          exit(EXIT_FAILURE);
      }
    int TILEDIM = (deviceProp.major < 2) ? 16 : 32;

    A = mxGPUCreateFromMxArray(prhs[0]);
    const mwSize *dims;
   dims=mxGPUGetDimensions(A);
   numARows = (int)dims[0]; /* gets number of rows of A */
   numAColumns = (int)dims[1]; /* gets number of columns of A */
 
 		
		numCRows = numARows;

		numCColumns = numAColumns;
  if (numARows==1)
 {   
 printf("Attention, this is a row vector, please try Discrete Sine Transform in row wise \n");
 return;
 }
 
    char const * const errId = "parallel:gpu:mexGPUExample:InvalidInput";
    char const * const errMsg = "Invalid input to MEX file, input(FIRST ARGUMENT) must be single precision (float).";

    if (mxGPUGetClassID(A) != mxSINGLE_CLASS) {
        mexErrMsgIdAndTxt(errId, errMsg);
    }


    d_A = (float const *)(mxGPUGetDataReadOnly(A));
    
    
    B = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(A),
                            mxGPUGetDimensions(A),
                            mxGPUGetClassID(A),
                            mxGPUGetComplexity(A),
                            MX_GPU_DO_NOT_INITIALIZE);
    d_B = (float *)(mxGPUGetData(B));

    dim3 dimBlock;
	dim3 dimGrid;
     switch (TILEDIM){
        
        case 16:
            
     TILE_DIM= TILEDIM;
	 dimBlock.x=TILE_DIM;
	 dimBlock.y=TILE_DIM;
     dimBlock.z=1;
	dimGrid.x = (numCColumns + dimBlock.x - 1) / dimBlock.x;
	dimGrid.y = (numCRows + dimBlock.y - 1) / dimBlock.y;
 if (strcmp (one,input_buf3) == 0)
{
     
   DSTI_Column_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
 }  
 if (strcmp (two,input_buf3) == 0)
{
     
   DSTII_Column_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
 }
 if (strcmp (three,input_buf3) == 0)
{
     
   DSTIII_Column_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
 }
 if (strcmp (four,input_buf3) == 0)
{
     
   DSTIV_Column_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
 }

  //	hipError_t err1 = hipPeekAtLastError();//To capture last error in function call

	//hipDeviceSynchronize();//To synchronize the device

      plhs[0] = mxGPUCreateMxArrayOnGPU(B);
      
    mxGPUDestroyGPUArray(A);
    
    mxGPUDestroyGPUArray(B);
    
            case 32:
            
     TILE_DIM= TILEDIM;
	 dimBlock.x=TILE_DIM;
	 dimBlock.y=TILE_DIM;
     dimBlock.z=1;
	dimGrid.x = (numCColumns + dimBlock.x - 1) / dimBlock.x;
	dimGrid.y = (numCRows + dimBlock.y - 1) / dimBlock.y;
    
     if (strcmp (one,input_buf3) == 0)
{
     
   DSTI_Column_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
 }  
 if (strcmp (two,input_buf3) == 0)
{
     
   DSTII_Column_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
 }
 if (strcmp (three,input_buf3) == 0)
{
     
   DSTIII_Column_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
 }
 if (strcmp (four,input_buf3) == 0)
{
     
   DSTIV_Column_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
 }

  //	hipError_t err1 = hipPeekAtLastError();//To capture last error in function call

	//hipDeviceSynchronize();//To synchronize the device

      plhs[0] = mxGPUCreateMxArrayOnGPU(B);
      
    mxGPUDestroyGPUArray(A);
    
    mxGPUDestroyGPUArray(B);
    
    
    }
     
	}
///  input standard array 

    else if (!(mxIsGPUArray(prhs[0]))){
            
    if (mxGetClassID(prhs[0]) != mxSINGLE_CLASS) {
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Invalid input to MEX file, input(FIRST ARGUMENT) must be single precision (float).");
    }         
            
  	int numARows = (int)mxGetM(prhs[0]); 		// number of rows in the matrix A
	int numAColumns = (int)mxGetN(prhs[0]); 	// number of columns in the matrix A
	int numCRows;		// number of rows in the matrix C (you have to set this)
	int numCColumns;	// number of columns in the matrix C (you have to set this)
    	
		numCRows = numARows;

		numCColumns = numAColumns;
        
 if (numARows==1)
 {   
 printf("Attention, this is a row vector, please try Discrete Sine Transform in row wise \n");
 return;
 }
    mxInitGPU();



	float  * hostA ; // The A matrix
	hostA = (float *)mxGetData(prhs[0]);
    
    plhs[0] = mxCreateNumericMatrix(numCRows, numCColumns, mxSINGLE_CLASS, mxREAL);
    float  *pointer = (float*)mxGetPr(plhs[0]);
    //CalculateTransform(float * A, float * C, int numARows, int numAColumns, int numCRows, int numCColumns)  
     if (strcmp (one,input_buf3) == 0)
    {
     
       CalculateTransformDSTColumnOne(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
	
    }
     if (strcmp (two,input_buf3) == 0)
    {
     
       CalculateTransformDSTColumnTwo(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
	
    }
     if (strcmp (three,input_buf3) == 0)
    {
     
       CalculateTransformDSTColumnThree(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
	
    }
     if (strcmp (four,input_buf3) == 0)
    {
     
       CalculateTransformDSTColumnFour(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
	
    }

	//free(hostB);
    } 
    } // Column
///////////////////////////    
     if (strcmp (row,input_buf4) == 0)
{
   ///  input standard GPUarray 
    if (mxIsGPUArray(prhs[0])) {
		
   mxGPUArray const *A;
    

    mxGPUArray *B;
    float const *d_A;
   
    float *d_B;
   
    int numARows, numAColumns, numCRows,  numCColumns;
    


    /* Initialize the MathWorks GPU API. */
    mxInitGPU();
    hipError_t error;
    int devID = 0;
    // get number of SMs on this GPU
    error = hipGetDevice(&devID);
    hipDeviceProp_t deviceProp;
     error = hipGetDeviceProperties(&deviceProp, devID);
      if (error != hipSuccess)
      {
          printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
          exit(EXIT_FAILURE);
      }
    int TILEDIM = (deviceProp.major < 2) ? 16 : 32;

    A = mxGPUCreateFromMxArray(prhs[0]);
   const mwSize *dims;
 dims=mxGPUGetDimensions(A);
 numARows = (int)dims[0]; /* gets number of rows of A */
 numAColumns = (int)dims[1]; /* gets number of columns of A */
 
 if (numAColumns==1)
 {   
 printf("Attention, this is a column vector, please try Discrete Sine Transform in column wise \n");
 return;
 }
 
 
    numCRows = numARows;
	numCColumns = numAColumns;

    char const * const errId = "parallel:gpu:mexGPUExample:InvalidInput";
    char const * const errMsg = "Invalid input to MEX file, input(FIRST ARGUMENT) must be single precision (float).";
    
    if (mxGPUGetClassID(A) != mxSINGLE_CLASS) {
        mexErrMsgIdAndTxt(errId, errMsg);
    }


    d_A = (float const *)(mxGPUGetDataReadOnly(A));
    
    
    B = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(A),
                            mxGPUGetDimensions(A),
                            mxGPUGetClassID(A),
                            mxGPUGetComplexity(A),
                            MX_GPU_DO_NOT_INITIALIZE);
    d_B = (float *)(mxGPUGetData(B));

    dim3 dimBlock;
	dim3 dimGrid;
     switch (TILEDIM){
        
        case 16:
            
     TILE_DIM= TILEDIM;
	 dimBlock.x=TILE_DIM;
	 dimBlock.y=TILE_DIM;
     dimBlock.z=1;
	dimGrid.x = (numCColumns + dimBlock.x - 1) / dimBlock.x;
	dimGrid.y = (numCRows + dimBlock.y - 1) / dimBlock.y;
 if (strcmp (one,input_buf3) == 0)
{
    
   DSTI_Row_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
}
 if (strcmp (two,input_buf3) == 0)
{
    
   DSTII_Row_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
}
 if (strcmp (three,input_buf3) == 0)
{
    
   DSTIII_Row_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
}
 if (strcmp (four,input_buf3) == 0)
{
    
   DSTIV_Row_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
}
      plhs[0] = mxGPUCreateMxArrayOnGPU(B);
      
    mxGPUDestroyGPUArray(A);
   
    mxGPUDestroyGPUArray(B);
     
      case 32:
            
     TILE_DIM= TILEDIM;
	 dimBlock.x=TILE_DIM;
	 dimBlock.y=TILE_DIM;
     dimBlock.z=1;
	dimGrid.x = (numCColumns + dimBlock.x - 1) / dimBlock.x;
	dimGrid.y = (numCRows + dimBlock.y - 1) / dimBlock.y;
     if (strcmp (one,input_buf3) == 0)
{
    
   DSTI_Row_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
}
 if (strcmp (two,input_buf3) == 0)
{
    
   DSTII_Row_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
}
 if (strcmp (three,input_buf3) == 0)
{
    
   DSTIII_Row_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
}
 if (strcmp (four,input_buf3) == 0)
{
    
   DSTIV_Row_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
}
      plhs[0] = mxGPUCreateMxArrayOnGPU(B);
      
    mxGPUDestroyGPUArray(A);
   
    mxGPUDestroyGPUArray(B);
    
    }
	}
///  input standard array 

    else if (!(mxIsGPUArray(prhs[0]))){
            
    if (mxGetClassID(prhs[0]) != mxSINGLE_CLASS) {
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Invalid input to MEX file, input(FIRST ARGUMENT) must be single precision (float).");
    }         
  	int numARows = (int)mxGetM(prhs[0]); 		// number of rows in the matrix A
	int numAColumns = (int)mxGetN(prhs[0]); 	// number of columns in the matrix A
	
	int numCRows;		// number of rows in the matrix C (you have to set this)
	int numCColumns;	// number of columns in the matrix C (you have to set this)
	
	numCRows = numARows;

	numCColumns = numAColumns;
	

	float  * hostA ; // The A matrix
	
	
 if (numAColumns==1)
 {   
 printf("Attention, this is a column vector, please try Discrete Sine Transform in column wise \n");
 return;
 }
mxInitGPU();
    

	hostA = (float *)mxGetData(prhs[0]);
   
    plhs[0] = mxCreateNumericMatrix(numCRows, numCColumns, mxSINGLE_CLASS, mxREAL);
    float  *pointer = (float*)mxGetPr(plhs[0]);
     if (strcmp (one,input_buf3) == 0)
{  
      CalculateTransformDSTRowOne(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
      
      } 
       if (strcmp (two,input_buf3) == 0)
{  
      CalculateTransformDSTRowTwo(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
      
      }
       if (strcmp (three,input_buf3) == 0)
{  
      CalculateTransformDSTRowThree(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
      
      }
       if (strcmp (four,input_buf3) == 0)
{  
      CalculateTransformDSTRowFour(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
      
      }
  
  //one
	//free(hostB);
    } 

    }//row
    ///////////////////////////
}// direct

 if (strcmp (inverse,input_buf2) == 0)
{
    if (strcmp (column,input_buf4) == 0)
{      
///  input standard GPUarray 
    if (mxIsGPUArray(prhs[0])) {
		//mexErrMsgIdAndTxt(errId, errMsg);
           /* Declare all variables.*/
    mxGPUArray const *A;
    mxGPUArray *B;
    float const *d_A;
    float *d_B;
    int numARows, numAColumns, numCRows,  numCColumns;
    


    /* Initialize the MathWorks GPU API. */
    mxInitGPU();
    hipError_t error;
    int devID = 0;
    // get number of SMs on this GPU
    error = hipGetDevice(&devID);
    hipDeviceProp_t deviceProp;
     error = hipGetDeviceProperties(&deviceProp, devID);
      if (error != hipSuccess)
      {
          printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
          exit(EXIT_FAILURE);
      }
    int TILEDIM = (deviceProp.major < 2) ? 16 : 32;

    A = mxGPUCreateFromMxArray(prhs[0]);
    const mwSize *dims;
   dims=mxGPUGetDimensions(A);
   numARows = (int)dims[0]; /* gets number of rows of A */
   numAColumns = (int)dims[1]; /* gets number of columns of A */
 
 		
		numCRows = numARows;

		numCColumns = numAColumns;
  if (numARows==1)
 {   
 printf("Attention, this is a row vector, please try Inverse Discrete Sine Transform in row wise \n");
 return;
 }
 
    char const * const errId = "parallel:gpu:mexGPUExample:InvalidInput";
    char const * const errMsg = "Invalid input to MEX file, input(FIRST ARGUMENT) must be single precision (float).";

    if (mxGPUGetClassID(A) != mxSINGLE_CLASS) {
        mexErrMsgIdAndTxt(errId, errMsg);
    }


    d_A = (float const *)(mxGPUGetDataReadOnly(A));
    
    
    B = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(A),
                            mxGPUGetDimensions(A),
                            mxGPUGetClassID(A),
                            mxGPUGetComplexity(A),
                            MX_GPU_DO_NOT_INITIALIZE);
    d_B = (float *)(mxGPUGetData(B));
    dim3 dimBlock;
	dim3 dimGrid;
     switch (TILEDIM){
        
        case 16:
            
     TILE_DIM= TILEDIM;
	 dimBlock.x=TILE_DIM;
	 dimBlock.y=TILE_DIM;
     dimBlock.z=1;
	dimGrid.x = (numCColumns + dimBlock.x - 1) / dimBlock.x;
	dimGrid.y = (numCRows + dimBlock.y - 1) / dimBlock.y;
 if (strcmp (one,input_buf3) == 0)
{
    
   DSTI_Column_Inverse_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
}
  if (strcmp (two,input_buf3) == 0)
{
    
   DSTII_Column_Inverse_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
}
 if (strcmp (three,input_buf3) == 0)
{
    
   DSTIII_Column_Inverse_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
}
 if (strcmp (four,input_buf3) == 0)
{
    
   DSTIV_Column_Inverse_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
}
  //	hipError_t err1 = hipPeekAtLastError();//To capture last error in function call

	//hipDeviceSynchronize();//To synchronize the device

      plhs[0] = mxGPUCreateMxArrayOnGPU(B);
      
    mxGPUDestroyGPUArray(A);
    
    mxGPUDestroyGPUArray(B);
    
      case 32:
            
     TILE_DIM= TILEDIM;
	 dimBlock.x=TILE_DIM;
	 dimBlock.y=TILE_DIM;
     dimBlock.z=1;
	dimGrid.x = (numCColumns + dimBlock.x - 1) / dimBlock.x;
	dimGrid.y = (numCRows + dimBlock.y - 1) / dimBlock.y; 
    
     if (strcmp (one,input_buf3) == 0)
{
    
   DSTI_Column_Inverse_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
}
  if (strcmp (two,input_buf3) == 0)
{
    
   DSTII_Column_Inverse_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
}
 if (strcmp (three,input_buf3) == 0)
{
    
   DSTIII_Column_Inverse_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
}
 if (strcmp (four,input_buf3) == 0)
{
    
   DSTIV_Column_Inverse_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
}
  //	hipError_t err1 = hipPeekAtLastError();//To capture last error in function call

	//hipDeviceSynchronize();//To synchronize the device

      plhs[0] = mxGPUCreateMxArrayOnGPU(B);
      
    mxGPUDestroyGPUArray(A);
    
    mxGPUDestroyGPUArray(B);
    
    }
     
	}
///  input standard array 

    else if (!(mxIsGPUArray(prhs[0]))){
            
    if (mxGetClassID(prhs[0]) != mxSINGLE_CLASS) {
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Invalid input to MEX file, input(FIRST ARGUMENT) must be single precision (float).");
    }        
  	int numARows = (int)mxGetM(prhs[0]); 		// number of rows in the matrix A
	int numAColumns = (int)mxGetN(prhs[0]); 	// number of columns in the matrix A
	int numCRows;		// number of rows in the matrix C (you have to set this)
	int numCColumns;	// number of columns in the matrix C (you have to set this)
    	
		numCRows = numARows;

		numCColumns = numAColumns;
        
 if (numARows==1)
 {   
 printf("Attention, this is a row vector, please try Inverse Discrete Sine Transform in row wise \n");
 return;
 }
    mxInitGPU();
 

	float  * hostA ; // The A matrix
	hostA = (float *)mxGetData(prhs[0]);
    
    plhs[0] = mxCreateNumericMatrix(numCRows, numCColumns, mxSINGLE_CLASS, mxREAL);
    float  *pointer = (float*)mxGetPr(plhs[0]);
    //CalculateTransform(float * A, float * C, int numARows, int numAColumns, int numCRows, int numCColumns) 
     if (strcmp (one,input_buf3) == 0)
   {
  
       CalculateTransformDSTInverseColumnOne(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
	
   }
     if (strcmp (two,input_buf3) == 0)
   {
  
       CalculateTransformDSTInverseColumnTwo(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
	
   }
     if (strcmp (three,input_buf3) == 0)
   {
  
       CalculateTransformDSTInverseColumnThree(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
	
   }
     if (strcmp (four,input_buf3) == 0)
   {
  
       CalculateTransformDSTInverseColumnFour(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
	
   }
	//free(hostB);
    } 
} // Column

    if (strcmp (row,input_buf4) == 0)
{
    
///  input standard GPUarray 
    if (mxIsGPUArray(prhs[0])) {
		//mexErrMsgIdAndTxt(errId, errMsg);
           /* Declare all variables.*/
   mxGPUArray const *A;
   

    mxGPUArray *B;
    float const *d_A;
   
    float *d_B;
   
    int numARows, numAColumns, numCRows,  numCColumns;
    


    /* Initialize the MathWorks GPU API. */
    mxInitGPU();
    hipError_t error;
    int devID = 0;
    // get number of SMs on this GPU
    error = hipGetDevice(&devID);
    hipDeviceProp_t deviceProp;
     error = hipGetDeviceProperties(&deviceProp, devID);
      if (error != hipSuccess)
      {
          printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
          exit(EXIT_FAILURE);
      }
    int TILEDIM = (deviceProp.major < 2) ? 16 : 32;
    

    A = mxGPUCreateFromMxArray(prhs[0]);
    const mwSize *dims;
    dims=mxGPUGetDimensions(A);
    numARows = (int)dims[0]; /* gets number of rows of A */
    numAColumns = (int)dims[1]; /* gets number of columns of A */
 
  if (numAColumns==1)
 {   
 printf("Attention, this is a column vector, please try Inverse Discrete Sine Transform in column wise \n");
 return;
 }
 
    numCRows = numARows;
	numCColumns = numAColumns;
    
    char const * const errId = "parallel:gpu:mexGPUExample:InvalidInput";
    char const * const errMsg = "Invalid input to MEX file, input(FIRST ARGUMENT) must be single precision (float).";

    if (mxGPUGetClassID(A) != mxSINGLE_CLASS) {
        mexErrMsgIdAndTxt(errId, errMsg);
    }


    d_A = (float const *)(mxGPUGetDataReadOnly(A));
    
    
    B = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(A),
                            mxGPUGetDimensions(A),
                            mxGPUGetClassID(A),
                            mxGPUGetComplexity(A),
                            MX_GPU_DO_NOT_INITIALIZE);
    d_B = (float *)(mxGPUGetData(B));
    
    dim3 dimBlock;
	dim3 dimGrid;
     switch (TILEDIM){
        
        case 16:
            
     TILE_DIM= TILEDIM;
	 dimBlock.x=TILE_DIM;
	 dimBlock.y=TILE_DIM;
     dimBlock.z=1;
	dimGrid.x = (numCColumns + dimBlock.x - 1) / dimBlock.x;
	dimGrid.y = (numCRows + dimBlock.y - 1) / dimBlock.y;
 if (strcmp (one,input_buf3) == 0)
{
   DSTI_Row__InverseKernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);

   }
 if (strcmp (two,input_buf3) == 0)
{
   DSTII_Row__InverseKernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);

   }
 if (strcmp (three,input_buf3) == 0)
{
   DSTIII_Row__InverseKernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);

   }
 if (strcmp (four,input_buf3) == 0)
{
   DSTIV_Row__InverseKernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);

   }
// one
  //	hipError_t err1 = hipPeekAtLastError();//To capture last error in function call

	//hipDeviceSynchronize();//To synchronize the device

      plhs[0] = mxGPUCreateMxArrayOnGPU(B);
      
    mxGPUDestroyGPUArray(A);
    
    mxGPUDestroyGPUArray(B);
    
          case 32:
            
     TILE_DIM= TILEDIM;
	 dimBlock.x=TILE_DIM;
	 dimBlock.y=TILE_DIM;
     dimBlock.z=1;
	dimGrid.x = (numCColumns + dimBlock.x - 1) / dimBlock.x;
	dimGrid.y = (numCRows + dimBlock.y - 1) / dimBlock.y;
    
    if (strcmp (one,input_buf3) == 0)
{
   DSTI_Row__InverseKernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);

   }
 if (strcmp (two,input_buf3) == 0)
{
   DSTII_Row__InverseKernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);

   }
 if (strcmp (three,input_buf3) == 0)
{
   DSTIII_Row__InverseKernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);

   }
 if (strcmp (four,input_buf3) == 0)
{
   DSTIV_Row__InverseKernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);

   }
// one
  //	hipError_t err1 = hipPeekAtLastError();//To capture last error in function call

	//hipDeviceSynchronize();//To synchronize the device

      plhs[0] = mxGPUCreateMxArrayOnGPU(B);
      
    mxGPUDestroyGPUArray(A);
    
    mxGPUDestroyGPUArray(B); 
    
    
    } 
     
	}
///  input standard array 

    else if (!(mxIsGPUArray(prhs[0]))){
            
    if (mxGetClassID(prhs[0]) != mxSINGLE_CLASS) {
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Invalid input to MEX file, input(FIRST ARGUMENT) must be single precision (float).");
    }        
  	int numARows = (int)mxGetM(prhs[0]); 		// number of rows in the matrix A
	int numAColumns = (int)mxGetN(prhs[0]); 	// number of columns in the matrix A
	
	int numCRows;		// number of rows in the matrix C (you have to set this)
	int numCColumns;	// number of columns in the matrix C (you have to set this)
	
	
    if (numAColumns==1)
    {   
    printf("Attention, this is a column vector, please try Inverse Discrete Sine Transform in column wise \n");
    return;
    }
    mxInitGPU();
    
    
    numCRows = numARows;
	numCColumns = numAColumns;
	//char const * const errId = "parallel:gpu:DCTTWO:InvalidInput";
	//char const * const errMsg = "Invalid input to MEX file, input(FIRST ARGUMENT) must be single precision (float).";

	float  * hostA ; // The A matrix
	
	
	
	hostA = (float *)mxGetData(prhs[0]);
    
    plhs[0] = mxCreateNumericMatrix(numCRows, numCColumns, mxSINGLE_CLASS, mxREAL);
    float  *pointer =(float*) mxGetPr(plhs[0]);
    
    
	//CalculateTransform(hostA, hostB, hostC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);
     if (strcmp (one,input_buf3) == 0)
{
      CalculateTransformDSTInverseRowOne(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);  
 } 
     if (strcmp (two,input_buf3) == 0)
{
      CalculateTransformDSTInverseRowTwo(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);  
 } 
     if (strcmp (three,input_buf3) == 0)
{
      CalculateTransformDSTInverseRowThree(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);  
 } 
     if (strcmp (four,input_buf3) == 0)
{
      CalculateTransformDSTInverseRowFour(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);  
 } 
//one
   //memcpy(pointer, hostC, numCRows*numCColumns*sizeof(float)); 
   
   // testing
    //    printf("\n  plhs[0]:");
    //    printf("\n");
    //    for (int i = 0; i<numCRows; i++){
	//	for (int j = 0; j<numCColumns; j++){
	//		printf(" %g ", round  (pointer[i * numCColumns + j]));
	//	}
	//	printf("\n");
//	}
	
	//free(hostB);
    }

    }//row

  } // inverse
}

    //sine

/////////////END

}
