#include "hip/hip_runtime.h"
/*
 * Discrete Cosine Transform in Column wise (DCT two)
 * DCT_II_Column
 * This CUDA code can handle/work with  any type of the input mxArrays, 
 * GPUarray or standard matlab CPU array as input {prhs[0] := mxGPUArray or CPU Array}
 * gpuArray output, B=DCT_II_Column(A)=mexFunction(A).
 * Developed at UCL, Institute of Neurology, 12 Queen Square, WC1N 3AR, London
 * Wellcome Trust Centre for Neuroimaging
 * Part of the project SPM(http://www.fil.ion.ucl.ac.uk/spm)
 * Copyright 2018
 * Kevin Bronik
 */
#include "matrix.h"
#include "mex.h"
#include "gpu/mxGPUArray.h"
#define DEFAULT_DIM 32 
#define 	DELTA(i, j)   ((i==j)?1:0)

const double  PI_d = 3.141592653589793238462643383279502884; //pi

__global__ void DCTII_Column_Kernel_GPUA(double const * const A, double const * const B, double * const C,
	int numARows, int numAColumns,
	int numBRows, int numBColumns,
	int numCRows, int numCColumns)
{
	double CValue = 0.0;

	int Row = blockIdx.y*DEFAULT_DIM + threadIdx.y;
	int Col = blockIdx.x*DEFAULT_DIM + threadIdx.x;

	for (int k = 0; k < (DEFAULT_DIM + numAColumns - 1) / DEFAULT_DIM; k++) {

		for (int n = 0; n < DEFAULT_DIM; ++n)
		if ((k*DEFAULT_DIM + n < numAColumns && Row < numARows) && (k*DEFAULT_DIM + n < numBRows && Col < numBColumns))
			CValue += A[Row*numAColumns + k*DEFAULT_DIM + n] * B[(k*DEFAULT_DIM + n)*numBColumns + Col];

	}

	if (Row < numCRows && Col < numCColumns) C[((blockIdx.y * blockDim.y + threadIdx.y)*numCColumns) + (blockIdx.x*blockDim.x) + threadIdx.x] = CValue;

}

__global__ void DCTII_Column_Kernel(double  *A, double   *B, double  *C,
	int numARows, int numAColumns,
	int numBRows, int numBColumns,
	int numCRows, int numCColumns)
{
	double CValue = 0.0;

	int Row = blockIdx.y*DEFAULT_DIM + threadIdx.y;
	int Col = blockIdx.x*DEFAULT_DIM + threadIdx.x;

	for (int k = 0; k < (DEFAULT_DIM + numAColumns - 1) / DEFAULT_DIM; k++) {

		for (int n = 0; n < DEFAULT_DIM; ++n)
		if ((k*DEFAULT_DIM + n < numAColumns && Row < numARows) && (k*DEFAULT_DIM + n < numBRows && Col < numBColumns))
			CValue += A[Row*numAColumns + k*DEFAULT_DIM + n] * B[(k*DEFAULT_DIM + n)*numBColumns + Col];

	}

	if (Row < numCRows && Col < numCColumns) C[((blockIdx.y * blockDim.y + threadIdx.y)*numCColumns) + (blockIdx.x*blockDim.x) + threadIdx.x] = CValue;

}


// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void CalculateTransform(double  * A, double  * B, double  * C, int numARows,
	int numAColumns, int numBRows, int numBColumns,
	int numCRows, int numCColumns)
{


	double  * hostA = A; // The A matrix
	double  * hostB = B; // The B matrix
	double * hostC = C; // The output C matrix
	//double * hostComputedC;
	double  * deviceA=0;
	double  * deviceB=0;
	double  * deviceC=0;

	//hostA = (double *)malloc(sizeof(float)*numARows*numAColumns);
	//hostB = (v *)malloc(sizeof(float)*numBRows*numBColumns);

	// Setting numCRows and numCColumns
	numCRows = numARows;
	numCColumns = numBColumns;
	// Allocate GPU buffers for three vectors (two input, one output)    .
	//hostC = (float *)malloc(sizeof(float)*numCRows*numCColumns);
	//hostComputedC = (float *)malloc(sizeof(float)*numCRows*numCColumns);

	
	 hipMalloc((void **)&deviceA, sizeof(double )*numARows*numAColumns);


	 hipMalloc((void **)&deviceB, sizeof(double )*numBRows*numBColumns);


	 hipMalloc((void **)&deviceC, sizeof(double )*numCRows*numCColumns);


	 hipMemcpy(deviceA, hostA, sizeof(double )*numARows*numAColumns, hipMemcpyHostToDevice);
	

	 hipMemcpy(deviceB, hostB, sizeof(double )*numBRows*numBColumns, hipMemcpyHostToDevice);
	

    dim3 dimBlock(DEFAULT_DIM, DEFAULT_DIM, 1);
	dim3 dimGrid;

	dimGrid.x = (numCColumns + dimBlock.x - 1) / dimBlock.x;
	dimGrid.y = (numCRows + dimBlock.y - 1) / dimBlock.y;
	DCTII_Column_Kernel << <dimGrid, dimBlock >> >(deviceA, deviceB, deviceC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);



	 hipDeviceSynchronize();//To synchronize the device

	// Copy the results in GPU memory back to the CPU
	 hipMemcpy(hostC, deviceC, sizeof(double)*numCRows*numCColumns, hipMemcpyDeviceToHost);

	C = hostC;

	hipFree(deviceA);
	hipFree(deviceB);
	hipFree(deviceC);
    
}

void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
 
int nDevices;
hipError_t errCode =hipGetDeviceCount(&nDevices); 
//int nDevices;
//hipGetDeviceCount(&nDevices);

if (errCode != hipSuccess){
printf("Error! No CUDA devices found! \n");
return;
}


///  input standard GPUarray 
    if (mxIsGPUArray(prhs[0])) {
		//mexErrMsgIdAndTxt(errId, errMsg);
           /* Declare all variables.*/
   mxGPUArray const *A;
    mxGPUArray const *DCOS;

    mxGPUArray *B;
    double const *d_A, *d_DCOS;
   
    double *d_B;
   // mxArray  * hostcos;
    //test
   // double * hostcos, *pointer;
   double  *pointer;
    //int N;
    int numARows, numAColumns,  numDCOSRows,  numDCOSColumns, numCRows,  numCColumns;
    
    char const * const errId = "parallel:gpu:mexGPUExample:InvalidInput";
    char const * const errMsg = "Invalid input to MEX file.";

    /* Initialize the MathWorks GPU API. */
    mxInitGPU();

   
    if ((nrhs!=1)) {
        mexErrMsgIdAndTxt(errId, errMsg);
    }

    A = mxGPUCreateFromMxArray(prhs[0]);
const mwSize *dims;
 dims=mxGPUGetDimensions(A);
 numARows = (int)dims[0]; /* gets number of rows of A */
 numAColumns = (int)dims[1]; /* gets number of columns of A */
 
 		numDCOSRows=numDCOSColumns = numARows;
		numCRows = numARows;

		numCColumns = numAColumns;
 
 if (numARows==1)
 {   
 printf("Attention, this is a row vector, please try Discrete Cosine Transform in row wise \n");
 return;
 }
 
//  numDCOSRows=numDCOSColumns=numAColumns;
//     numCRows = numARows;
// 	numCColumns = numDCOSColumns;
 mxArray *COS= mxCreateNumericMatrix(numDCOSRows, numDCOSColumns, mxDOUBLE_CLASS, mxREAL);
pointer = mxGetPr(COS);
 

		for (int i = 0; i < numDCOSRows; i++){
			for (int j = 0; j < numDCOSColumns; j++){
				//hostB[i * numBColumns + j] = i + j* numAColumns;
				//hostB[i * numBColumns + j] = 1;
				//cosvalx[i * numBColumns + j] = cos(((2 * j + 1) / (2.0 * numBColumns))*3.14*i)*sqrt(1.0 / numBColumns);
				//hostB[i * numBColumns + j] = cosvalx[i + j* numAColumns];

				//hostB[i + j* numBColumns] = cos(((2 * i + 1) / (2.0 * numBColumns))*3.14*j)*sqrt(1.0 / (1 + DELTA(1, j + 1)))*sqrt(2.0 / numBColumns);
				pointer[i* numDCOSColumns + j] = cos(((2 * j + 1) / (2.0 * numDCOSColumns))*PI_d*i)*sqrt(1.0 / (1 + DELTA(i + 1, 1)))*sqrt(2.0 / numDCOSColumns); 
				//hostB[i + j* numBColumns] = 1;

				//hostL[i* numBColumns + j] = cos(((2 * j + 1) / (2.0 * numBColumns))*3.14*i)*sqrt(1.0 / numBColumns);
				//hostB[i + j* numBColumns] = 1;

			}
		}



// 		for (int i = 0; i < numDCOSRows; i++){
// 			for (int j = 0; j < numDCOSColumns; j++){
// 				//hostB[i * numDCOSColumns+ j] = i + j* numAColumns;
// 				//hostB[i * numDCOSColumns + j] = 1;
// 				//cosvalx[i * numDCOSColumns + j] = cos(((2 * j + 1) / (2.0 * numBColumns))*PI_d*i)*sqrt(1.0 / numBColumns);
// 				//hostB[i * numBColumns + j] = cosvalx[i + j* numAColumns];
// 				if (i == 0) {
// 					pointer[i* numDCOSColumns + j] = cos(((2 * j + 1) / (2.0 * numDCOSColumns))*PI_d*i)*sqrt(1.0 / numDCOSColumns);
// 					//pointer[i + j* numDCOSColumns] = 1;
// 				}
// 				else if (i != 0) {
// 					pointer[i* numDCOSColumns + j] = cos(((2 * j + 1) / (2.0 * numDCOSColumns))*PI_d*i)*sqrt(2.0 / numDCOSColumns);
// 					//pointer[i + j* numDCOSColumns] = 2;
// 				}
// 			}
// 		}




   
   DCOS=mxGPUCreateFromMxArray(COS);
  //  DCOS=mxGPUCreateFromMxArray(hostcos);

    if (mxGPUGetClassID(A) != mxDOUBLE_CLASS) {
        mexErrMsgIdAndTxt(errId, errMsg);
    }


    d_A = (double const *)(mxGPUGetDataReadOnly(A));
    d_DCOS=(double const *)(mxGPUGetDataReadOnly(DCOS));
    
    B = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(A),
                            mxGPUGetDimensions(A),
                            mxGPUGetClassID(A),
                            mxGPUGetComplexity(A),
                            MX_GPU_DO_NOT_INITIALIZE);
    d_B = (double *)(mxGPUGetData(B));

    
    dim3 dimBlock(DEFAULT_DIM, DEFAULT_DIM, 1);
	dim3 dimGrid;

	dimGrid.x = (numCColumns + dimBlock.x - 1) / dimBlock.x;
	dimGrid.y = (numCRows + dimBlock.y - 1) / dimBlock.y;
    //(hostL, hostA, hostC,  numBRows, numBColumns, numARows, numAColumns, numCRows, numCColumns);
   //DCTII_Column_Kernel_GPUA<< <dimGrid, dimBlock >> >(d_A, d_DCOS, d_B, numARows, numAColumns, numDCOSRows, numDCOSColumns, numCRows, numCColumns);
   DCTII_Column_Kernel_GPUA<< <dimGrid, dimBlock >> >(d_DCOS, d_A, d_B, numDCOSRows, numDCOSColumns, numARows, numAColumns, numCRows, numCColumns);
   
  //	hipError_t err1 = hipPeekAtLastError();//To capture last error in function call

	//hipDeviceSynchronize();//To synchronize the device

      plhs[0] = mxGPUCreateMxArrayOnGPU(B);
      
    mxGPUDestroyGPUArray(A);
    mxGPUDestroyGPUArray(DCOS);
    mxGPUDestroyGPUArray(B);
     
	}
///  input standard array 

    else if (!(mxIsGPUArray(prhs[0]))){
  	int numARows = (int)mxGetM(prhs[0]); 		// number of rows in the matrix A
	int numAColumns = (int)mxGetN(prhs[0]); 	// number of columns in the matrix A
	int numBRows; 		// number of rows in the matrix B
	int numBColumns; 	// number of columns in the matrix B
	int numCRows;		// number of rows in the matrix C (you have to set this)
	int numCColumns;	// number of columns in the matrix C (you have to set this)
    	numBRows = numBColumns = numARows;
		numCRows = numARows;

		numCColumns = numAColumns;
    
// 	numBRows = numBColumns = numAColumns;
// 	numCRows = numARows;
// 
// 	numCColumns = numBColumns;

        
 if (numARows==1)
 {   
 printf("Attention, this is a row vector, please try Discrete Cosine Transform in row wise \n");
 return;
 }

	//char const * const errId = "parallel:gpu:DCTTWO:InvalidInput";
	//char const * const errMsg = "Invalid input to MEX file.";

	double  * hostA ; // The A matrix
	double  * hostB ; // The B matrix
	


	/* Initialize the MathWorks GPU API. */
	//mxInitGPU();

	/* Throw an error if the input is not a GPU array. */
	//if ((nrhs != 1) || !(mxIsGPUArray(prhs[0]))) {
		//mexErrMsgIdAndTxt(errId, errMsg);
	//}

	//hostA = (double *)malloc(sizeof(double)*numARows*numAColumns);
	//hostAx = (double *)malloc(sizeof(double)*numARows*numAColumns);
	//hostAy = (double *)malloc(sizeof(double)*numARows*numAColumns);
	hostB = (double  *)malloc(sizeof(double)*numBRows*numBColumns);

    
  //const  mxArray *G =prhs[0];
   // if ((nrhs != 1) || (mxIsGPUArray(G))) {
		//mexErrMsgIdAndTxt(errId, errMsg);
    //    G = gather(G);
//	}
	hostA = (double *)mxGetData(prhs[0]);
    // hostA = (double *)mxGetData(G);
	//Discrete Cosine Transform in Columns wise
    
    		for (int i = 0; i < numBRows; i++){
			for (int j = 0; j < numBColumns; j++){
				//hostB[i * numBColumns + j] = i + j* numAColumns;
				//hostB[i * numBColumns + j] = 1;
				//cosvalx[i * numBColumns + j] = cos(((2 * j + 1) / (2.0 * numBColumns))*3.14*i)*sqrt(1.0 / numBColumns);
				//hostB[i * numBColumns + j] = cosvalx[i + j* numAColumns];

				//hostB[i + j* numBColumns] = cos(((2 * i + 1) / (2.0 * numBColumns))*3.14*j)*sqrt(1.0 / (1 + DELTA(1, j + 1)))*sqrt(2.0 / numBColumns);
				hostB[i* numBColumns + j] = cos(((2 * j + 1) / (2.0 * numBColumns))*PI_d*i)*sqrt(1.0 / (1 + DELTA(i + 1, 1)))*sqrt(2.0 / numBColumns); 
				//hostB[i + j* numBColumns] = 1;

				//hostL[i* numBColumns + j] = cos(((2 * j + 1) / (2.0 * numBColumns))*3.14*i)*sqrt(1.0 / numBColumns);
				//hostB[i + j* numBColumns] = 1;

			}
		}

    
//     		for (int i = 0; i < numBRows; i++){
// 			for (int j = 0; j < numBColumns; j++){
// 				//hostB[i * numBColumns + j] = i + j* numAColumns;
// 				//hostB[i * numBColumns + j] = 1;
// 				//cosvalx[i * numBColumns + j] = cos(((2 * j + 1) / (2.0 * numBColumns))*PI_d*i)*sqrt(1.0 / numBColumns);
// 				//hostB[i * numBColumns + j] = cosvalx[i + j* numAColumns];
// 				if (i == 0) {
// 					hostB[i* numBColumns + j] = cos(((2 * j + 1) / (2.0 * numBColumns))*PI_d*i)*sqrt(1.0 / numBColumns);
// 					//hostB[i + j* numBColumns] = 1;
// 				}
// 				else if (i != 0) {
// 					hostB[i* numBColumns + j] = cos(((2 * j + 1) / (2.0 * numBColumns))*PI_d*i)*sqrt(2.0 / numBColumns);
// 					//hostB[i + j* numBColumns] = 2;
//				}
// 			}
// 		}
    
    
    
	//plhs[0] = mxCreateNumericMatrix(numARows, numBColumns, mxDOUBLE_CLASS, mxREAL);

	//hostC = (double*)mxGetData(plhs[0]);
    plhs[0] = mxCreateNumericMatrix(numCRows, numCColumns, mxDOUBLE_CLASS, mxREAL);
    double  *pointer = mxGetPr(plhs[0]);
    
   // (hostL, hostA, hostC,  numBRows, numBColumns, numARows, numAColumns, numCRows, numCColumns);
	//CalculateTransform(hostA, hostB, hostC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns); 
     // CalculateTransform(hostA, hostB, pointer, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);  
       CalculateTransform( hostB, hostA, pointer, numBRows, numBColumns, numARows, numAColumns, numCRows, numCColumns);
   //memcpy(pointer, hostC, numCRows*numCColumns*sizeof(double)); 

	
	free(hostB);
    }
	
}
