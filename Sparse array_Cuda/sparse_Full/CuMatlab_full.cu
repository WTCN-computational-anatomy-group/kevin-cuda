
/*
 * This CUDA-Cusparse code can handle/work with  any type of the input mxArrays, 
 * GPUarray or standard matlab CPU array as input {prhs[0] := mxGPUArray or CPU Array}[double/complex double]
 * Sparse/Dense --> Dense,   Z=CuMatlab_full(Sparse/Dense(X)).
 * Developed at UCL, Institute of Neurology, 12 Queen Square, WC1N 3AR, London
 * Wellcome Trust Centre for Neuroimaging
 * Part of the project SPM(http://www.fil.ion.ucl.ac.uk/spm)
 * Copyright 2018
 * Kevin Bronik
 */

#include "matrix.h"
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <hipsparse.h>

#include "CuMatlab_fullD.cu"
#include "CuMatlab_fullZ.cu"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


extern "C" static void mexCuMatlab_fullD(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[]);
                 
extern "C" static void mexCuMatlab_fullZ(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[]);
                 

                 
void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
 int nDevices;
hipError_t errCode =hipGetDeviceCount(&nDevices); 
//int nDevices;
//hipGetDeviceCount(&nDevices);

if (errCode != hipSuccess){
printf("Error! No CUDA devices found! \n");
return;
}

    
   if (nrhs==1 && nlhs==1) {

        if (mxIsGPUArray(prhs[0])) {
        
    mxGPUArray const *tempGPU;
    tempGPU = mxGPUCreateFromMxArray(prhs[0]);
        
       if (mxGPUGetClassID(tempGPU) == mxDOUBLE_CLASS && mxGPUGetComplexity(tempGPU) == mxREAL){ 
           
       mexCuMatlab_fullD(nlhs, plhs,
                 nrhs, prhs);
           
       mxGPUDestroyGPUArray(tempGPU);
       return;
           }
       else if (mxGPUGetClassID(tempGPU) == mxDOUBLE_CLASS  && mxGPUGetComplexity(tempGPU) == mxCOMPLEX){ 
              mexCuMatlab_fullZ(nlhs, plhs,
                 nrhs, prhs);
       
       mxGPUDestroyGPUArray(tempGPU);
       return;
           
           }
           
       else{ 
               
       mxGPUDestroyGPUArray(tempGPU);
       mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Incorrect input arguments! %s\n");    
           
           }
    }
    //
     else if(!mxIsGPUArray(prhs[0])) {  
       if (mxGetClassID(prhs[0]) == mxDOUBLE_CLASS && (!mxIsComplex(prhs[0]))){ 
            
           mexCuMatlab_fullD(nlhs, plhs,
                 nrhs, prhs);
                 return; 
       
           }
       else if (mxGetClassID(prhs[0]) == mxDOUBLE_CLASS && (mxIsComplex(prhs[0]))){ 
           
           mexCuMatlab_fullZ(nlhs, plhs,
                 nrhs, prhs);
                 return; 
           }
           
           
       else{  
               
       mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Incorrect input arguments! %s\n");    
           
           }
           
       }           
                 
    }
    
 
    
     else if  ((nrhs<1) || (nrhs>1)  ||  (nlhs<1)  || (nlhs>1) ) {
            
        mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Incorrect input/output arguments! input argument must be one and output argument must be one\n"); 
                return;     
        }

}
