
/*
 * This CUDA-Cusparse code can handle/work with  any type of the input mxArrays, 
 * GPUarray or standard matlab CPU array as input {prhs[0]/prhs[1] := mxGPUArray or CPU Array}[double/complex double]
 * Sparse/Dense matrix-sparse/dense vector multiplication   Z=CuMatlab_solve(Sparse/Dense(A),Sparse/Dense(Y)).
 * AZ=Y -->Z=A\Y
 * Developed at UCL, Institute of Neurology, 12 Queen Square, WC1N 3AR, London
 * Wellcome Trust Centre for Neuroimaging
 * Part of the project SPM(http://www.fil.ion.ucl.ac.uk/spm)
 * Copyright 2018
 * Kevin Bronik
 */

#include "matrix.h"
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <hipsparse.h>
#include <hipsolver.h>
#include <hip/hip_runtime_api.h>
#include "hipsolver.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "SPARSEHELPER.h"
#include "ERRORCHK.h"
#include <omp.h>

// Input Arguments
#define	INPUTSPARSEA   prhs[0]
#define	INPUTDENSEB   prhs[1]

// Output Arguments
#define	OUTPUTMATRIX  plhs[0]



  
    
extern "C" static void mexCuMatlab_sparseSDR(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
 int nDevices;
hipError_t errCode =hipGetDeviceCount(&nDevices); 
//int nDevices;
//hipGetDeviceCount(&nDevices);

if (errCode != hipSuccess){
printf("Error! No CUDA devices found! \n");
return;
}

    char const * const InputErrMsg = "Invalid input to MEX file, number of input arguments must be two.";
    char const * const OutputErrMsg = "Invalid output to MEX file, number of output arguments must be one.";
   if ((nrhs!=2)) {
        mexErrMsgIdAndTxt("MATLAB:mexatexit:invalidInput", InputErrMsg);
    }
   if ((nlhs!=1)) {
        mexErrMsgIdAndTxt("MATLAB:mexatexit:invalidInput", OutputErrMsg);
    }
 char *input_buf0;
 input_buf0 = mxArrayToString(INPUTSPARSEA);

      if ((mxIsChar(INPUTSPARSEA))){
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Input(FIRST ARGUMENT) must be array, or gpuArray object not  %s\n",input_buf0);
    }
    
 char *input_buf1;
 input_buf1 = mxArrayToString(INPUTDENSEB);

      if ((mxIsChar(INPUTDENSEB))){
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Input(SECOND ARGUMENT) must be array, or gpuArray object not  %s\n",input_buf1);
    } 


if (mxIsGPUArray(INPUTSPARSEA) && mxIsGPUArray(INPUTDENSEB)) {
    
    mxGPUArray const *INPUTSPARSEGPUA;
    mxGPUArray const *INPUTDENSEGPUB;
    
    
    /* Initialize the MathWorks GPU API. */
    mxInitGPU();
    INPUTSPARSEGPUA = mxGPUCreateFromMxArray(INPUTSPARSEA);
    INPUTDENSEGPUB = mxGPUCreateFromMxArray(INPUTDENSEB);
    
   
	
    if((mxGPUIsSparse(INPUTSPARSEGPUA))&& (!mxGPUIsSparse(INPUTDENSEGPUB)) ){
        
    const mwSize *dimsGPUSA;
    dimsGPUSA=mxGPUGetDimensions(INPUTSPARSEGPUA);
    int numARows, numAColumns;
    numARows = (int)dimsGPUSA[0]; /* gets number of rows of A */
    numAColumns = (int)dimsGPUSA[1]; /* gets number of columns of A */
    
    const mwSize *dimsGPUSB;
    dimsGPUSB=mxGPUGetDimensions(INPUTDENSEGPUB);
    int numBRows, numBColumns;
    numBRows = (int)dimsGPUSB[0]; /* gets number of rows of B */
    numBColumns = (int)dimsGPUSB[1]; /* gets number of columns of B */
    if ( numARows != numAColumns) {
       
        mxGPUDestroyGPUArray(INPUTDENSEGPUB);
        mxGPUDestroyGPUArray(INPUTSPARSEGPUA);   
       
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                        "Invalid input to MEX file,first argument must be a sparse/dense square matrix.");
             
    } 
     if ( (numBColumns!= 1) ) {
         
        mxGPUDestroyGPUArray(INPUTDENSEGPUB);
        mxGPUDestroyGPUArray(INPUTSPARSEGPUA);
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                        "Invalid input to MEX file, second argument must be a dense/sparse column vector.");
             
    }
    if ( (numBRows!= numARows) ) {
        mxGPUDestroyGPUArray(INPUTDENSEGPUB);
        mxGPUDestroyGPUArray(INPUTSPARSEGPUA);
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                        "Invalid input to MEX file, array (matrix-vector) dimensions must agree.");
             
    }
    


    
    double const *d_B_dense;
   d_B_dense = (double const *)(mxGPUGetDataReadOnly(INPUTDENSEGPUB));
    

    mwIndex nnz1;
     mxArray * VLSXY1 = mxGPUCreateMxArrayOnCPU(INPUTSPARSEGPUA);
    nnz1 = *(mxGetJc(VLSXY1) + numAColumns);
    int nnzA = static_cast<int> (nnz1);
    
   
   mxArray *  ROW_SORTA = mxCreateNumericMatrix(nnzA, 1,mxINT32_CLASS, mxREAL);
    int *ROWSORTA  = (int *)mxGetInt32s(ROW_SORTA);
       SetIr_Data(VLSXY1, ROWSORTA);
    
   mxArray *  COL_SORTA = mxCreateNumericMatrix(nnzA, 1, mxINT32_CLASS, mxREAL);
    int  *COLSORTA = (int *)mxGetInt32s(COL_SORTA);
          SetJc_Int(VLSXY1, COLSORTA);
      
 
    double  *VALSORTA = (double *)mxGetDoubles(VLSXY1);
           
    hipsparseHandle_t handle;    cusparseSafeCall(hipsparseCreate(&handle));

	hipsparseMatDescr_t descrA;      cusparseSafeCall(hipsparseCreateMatDescr(&descrA));
	hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE);


		mxGPUDestroyGPUArray(INPUTSPARSEGPUA);
        mxGPUDestroyGPUArray(INPUTDENSEGPUB);
		
		
	//double *d_A;            gpuErrchk(hipMalloc(&d_A, nnzA * sizeof(*d_A)));
	//int *d_A_RowIndices;    gpuErrchk(hipMalloc(&d_A_RowIndices, (numARows + 1) * sizeof(*d_A_RowIndices)));
	//int *d_A_ColIndices;    gpuErrchk(hipMalloc(&d_A_ColIndices, nnzA * sizeof(*d_A_ColIndices)));
	//int *d_cooRowIndA;       gpuErrchk(hipMalloc(&d_cooRowIndA, nnzA * sizeof(*d_cooRowIndA)));
   size_t pivot_dimensA[1] = {nnzA};
   size_t pivot_dimensROW_A[1] = {numARows+1};
   size_t pivot_dimensCOL_A[1] = {nnzA};
   size_t pivot_dimensCOO_A[1] = {nnzA};
   
   mxGPUArray *A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensA, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    double  *d_A = (double *)mxGPUGetData(A);
   mxGPUArray * ROW_A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensROW_A, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_A_RowIndices = (int *)mxGPUGetData(ROW_A);
   mxGPUArray * COL_A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensCOL_A, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_A_ColIndices = (int *)mxGPUGetData(COL_A);
    mxGPUArray * COO_A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensCOO_A, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_cooRowIndA = (int *)mxGPUGetData(COO_A); 
	
	// --- Descriptor for sparse matrix B
	gpuErrchk(hipMemcpy(d_A, VALSORTA, nnzA * sizeof(*d_A), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_A_ColIndices, COLSORTA, nnzA * sizeof(*d_A_ColIndices), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_cooRowIndA, ROWSORTA, nnzA * sizeof(*d_cooRowIndA), hipMemcpyHostToDevice));
    
         mxDestroyArray(COL_SORTA);
         mxDestroyArray(ROW_SORTA);
         mxDestroyArray(VLSXY1);
         
         
	int *Pa = NULL;
	void *pBuffera = NULL;
	size_t pBufferSizeInBytesa = 0;
	hipsparseXcoosort_bufferSizeExt(handle, numARows, numAColumns,
		nnzA,
		d_cooRowIndA,
		d_A_ColIndices, &pBufferSizeInBytesa);

	gpuErrchk(hipMalloc(&pBuffera, sizeof(char)*pBufferSizeInBytesa));
	gpuErrchk(hipMalloc(&Pa, sizeof(int)*nnzA));
	hipsparseCreateIdentityPermutation(handle, nnzA, Pa);
	cusparseSafeCall(hipsparseXcoosortByRow(handle, numARows, numAColumns,
		nnzA,
		d_cooRowIndA,
		d_A_ColIndices,
		Pa,
		pBuffera));

	cusparseSafeCall(hipsparseDgthr(handle, nnzA, d_A, d_A, Pa, HIPSPARSE_INDEX_BASE_ZERO));

	cusparseSafeCall(hipsparseXcoo2csr(handle,
		d_cooRowIndA,
		nnzA,
		numARows,
		d_A_RowIndices,
		HIPSPARSE_INDEX_BASE_ONE));
    mxGPUDestroyGPUArray(COO_A);
    gpuErrchk(hipFree(pBuffera));
	gpuErrchk(hipFree(Pa));
    
hipsolverSpHandle_t handle_cusolver; 
hipsolverSpCreate(&handle_cusolver);
csrcholInfo_t chl_info = NULL;
    const double tol = 1.e-14; 
    int singularity = 0;
    size_t size_internal = 0; 
    size_t size_chol = 0;
  cusolverSafeCall(cusolverSpCreateCsrcholInfo(&chl_info));           

  cusolverSafeCall(cusolverSpXcsrcholAnalysis(
        handle_cusolver, numARows, nnzA,
        descrA, d_A_RowIndices, d_A_ColIndices,
        chl_info));  
    
  cusolverSafeCall(cusolverSpDcsrcholBufferInfo(
        handle_cusolver, numARows, nnzA,
        descrA, d_A, d_A_RowIndices, d_A_ColIndices,
        chl_info,
        &size_internal,
        &size_chol));   
     
    void *buffer_gpu = NULL; 

    gpuErrchk(hipMalloc(&buffer_gpu, sizeof(char)*size_chol)); 
   
     cusolverSafeCall(cusolverSpDcsrcholFactor(
        handle_cusolver, numARows, nnzA,
        descrA, d_A, d_A_RowIndices, d_A_ColIndices,
        chl_info,
        buffer_gpu));
    
    cusolverSafeCall(cusolverSpDcsrcholZeroPivot(
        handle_cusolver, chl_info, tol, &singularity));
    
    if ( 0 <= singularity){
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                       "Invalid input to MEX file, (fatal error:) A is not invertible, singularity=%d\n", singularity);
       
    }
    
    
    size_t pivot_dimensionsvalueVa[1] = {numAColumns};

    mxGPUArray *VAL = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionsvalueVa, mxDOUBLE_CLASS, mxREAL, MX_GPU_INITIALIZE_VALUES);
    double  *VALOUT = (double *)mxGPUGetData(VAL);

    cusolverSafeCall(cusolverSpDcsrcholSolve(
        handle_cusolver, numARows, d_B_dense, VALOUT, chl_info, buffer_gpu));
    

        mxGPUDestroyGPUArray(A);
        mxGPUDestroyGPUArray(ROW_A);
        mxGPUDestroyGPUArray(COL_A);   
   
OUTPUTMATRIX = mxGPUCreateMxArrayOnGPU(VAL);             
gpuErrchk(hipFree(buffer_gpu));       
mxGPUDestroyGPUArray(VAL);
cusolverSpDestroyCsrcholInfo(chl_info);      
hipsparseDestroyMatDescr(descrA);
hipsolverSpDestroy(handle_cusolver);
hipsparseDestroy(handle);
    
        }
    
        else{
        mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Incorrect input arguments! %s\n");    
        }
    
   }
     
////////////////////////////////////////////////////////////////////////////////////  
    else if (!(mxIsGPUArray(INPUTSPARSEA)) && !(mxIsGPUArray(INPUTDENSEB))){
   
     // if ((mxGetClassID(INPUTSPARSEA) != mxDOUBLE_CLASS) || (mxGetClassID(INPUTSPARSEB) != mxDOUBLE_CLASS)) {
       //  mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
           //     "Invalid input to MEX file, input(FIRST and SECOND  ARGUMENTS) must be  double precision.");
             
   // }
    if((mxIsSparse(INPUTSPARSEA))&& (!mxIsSparse(INPUTDENSEB)) ){
    
     mxInitGPU();
    const mwSize *dimsCPUA;
    dimsCPUA=mxGetDimensions(INPUTSPARSEA);
    
    int  numARows = (int)dimsCPUA[0]; /* gets number of rows of A */
    int  numAColumns = (int)dimsCPUA[1]; /* gets number of columns of A */
   
    const mwSize *dimsCPUB;
    dimsCPUB=mxGetDimensions(INPUTDENSEB);
    
    int  numBRows = (int)dimsCPUB[0]; /* gets number of rows of B */
    int  numBColumns = (int)dimsCPUB[1]; /* gets number of columns of B */
    if ( numARows != numAColumns ) {
      
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                        "Invalid input to MEX file,first argument must be a sparse/dense square matrix.");
             
    } 
     if ( (numBColumns!= 1) ) {
  
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                        "Invalid input to MEX file, second argument must be a dense/sparse column vector.");
             
    }
    if ( (numBRows!= numARows) ) {

         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                        "Invalid input to MEX file, array (matrix-vector) dimensions must agree.");
             
    }

      
    
    mwIndex nnz1;
 
    nnz1 = *(mxGetJc(INPUTSPARSEA) + numAColumns);
    int nnzA = static_cast<int> (nnz1);
    

   
   mxArray *  ROW_SORTA = mxCreateNumericMatrix(nnzA, 1,mxINT32_CLASS, mxREAL);
    int *ROWSORTA  = (int *)mxGetInt32s(ROW_SORTA);
       SetIr_Data(INPUTSPARSEA, ROWSORTA);

    
   mxArray *  COL_SORTA = mxCreateNumericMatrix(nnzA, 1, mxINT32_CLASS, mxREAL);
    int  *COLSORTA = (int *)mxGetInt32s(COL_SORTA);
          SetJc_Int(INPUTSPARSEA, COLSORTA);

      
    double  *VALSORTA = (double  *)mxGetDoubles(INPUTSPARSEA);
           
    hipsparseHandle_t handle;    cusparseSafeCall(hipsparseCreate(&handle));

	hipsparseMatDescr_t descrA;      cusparseSafeCall(hipsparseCreateMatDescr(&descrA));
	hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE);

	
	  size_t pivot_dimensionsvalueDB[1] = {numBRows};
      mxGPUArray *OUTMB = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionsvalueDB, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
      double *d_B_dense = (double *)mxGPUGetData(OUTMB);
	
   double *h_B_dense1;
   h_B_dense1 = (double *)mxGetDoubles(INPUTDENSEB);
	gpuErrchk(hipMemcpy(d_B_dense, h_B_dense1, numBRows * sizeof(*d_B_dense), hipMemcpyHostToDevice));
    
    
    	//double *d_A;            gpuErrchk(hipMalloc(&d_A, nnzA * sizeof(*d_A)));
	//int *d_A_RowIndices;    gpuErrchk(hipMalloc(&d_A_RowIndices, (numARows + 1) * sizeof(*d_A_RowIndices)));
	//int *d_A_ColIndices;    gpuErrchk(hipMalloc(&d_A_ColIndices, nnzA * sizeof(*d_A_ColIndices)));
	//int *d_cooRowIndA;       gpuErrchk(hipMalloc(&d_cooRowIndA, nnzA * sizeof(*d_cooRowIndA)));
   size_t pivot_dimensA[1] = {nnzA};
   size_t pivot_dimensROW_A[1] = {numARows+1};
   size_t pivot_dimensCOL_A[1] = {nnzA};
   size_t pivot_dimensCOO_A[1] = {nnzA};
   
   mxGPUArray *A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensA, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    double  *d_A = (double *)mxGPUGetData(A);
   mxGPUArray * ROW_A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensROW_A, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_A_RowIndices = (int *)mxGPUGetData(ROW_A);
   mxGPUArray * COL_A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensCOL_A, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_A_ColIndices = (int *)mxGPUGetData(COL_A);
    mxGPUArray * COO_A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensCOO_A, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_cooRowIndA = (int *)mxGPUGetData(COO_A); 
	
	// --- Descriptor for sparse matrix B
	gpuErrchk(hipMemcpy(d_A, VALSORTA, nnzA * sizeof(*d_A), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_A_ColIndices, COLSORTA, nnzA * sizeof(*d_A_ColIndices), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_cooRowIndA, ROWSORTA, nnzA * sizeof(*d_cooRowIndA), hipMemcpyHostToDevice));
    
         mxDestroyArray(COL_SORTA);
         mxDestroyArray(ROW_SORTA);
        
         
         
	int *Pa = NULL;
	void *pBuffera = NULL;
	size_t pBufferSizeInBytesa = 0;
	hipsparseXcoosort_bufferSizeExt(handle, numARows, numAColumns,
		nnzA,
		d_cooRowIndA,
		d_A_ColIndices, &pBufferSizeInBytesa);

	gpuErrchk(hipMalloc(&pBuffera, sizeof(char)*pBufferSizeInBytesa));
	gpuErrchk(hipMalloc(&Pa, sizeof(int)*nnzA));
	hipsparseCreateIdentityPermutation(handle, nnzA, Pa);
	cusparseSafeCall(hipsparseXcoosortByRow(handle, numARows, numAColumns,
		nnzA,
		d_cooRowIndA,
		d_A_ColIndices,
		Pa,
		pBuffera));

	cusparseSafeCall(hipsparseDgthr(handle, nnzA, d_A, d_A, Pa, HIPSPARSE_INDEX_BASE_ZERO));

	cusparseSafeCall(hipsparseXcoo2csr(handle,
		d_cooRowIndA,
		nnzA,
		numARows,
		d_A_RowIndices,
		HIPSPARSE_INDEX_BASE_ONE));
    mxGPUDestroyGPUArray(COO_A);
    gpuErrchk(hipFree(pBuffera));
	gpuErrchk(hipFree(Pa));
    
 hipsolverSpHandle_t handle_cusolver; 
hipsolverSpCreate(&handle_cusolver);
csrcholInfo_t chl_info = NULL;
    const double tol = 1.e-14; 
    int singularity = 0;
    size_t size_internal = 0; 
    size_t size_chol = 0;
  cusolverSafeCall(cusolverSpCreateCsrcholInfo(&chl_info));           

  cusolverSafeCall(cusolverSpXcsrcholAnalysis(
        handle_cusolver, numARows, nnzA,
        descrA, d_A_RowIndices, d_A_ColIndices,
        chl_info));  
    
  cusolverSafeCall(cusolverSpDcsrcholBufferInfo(
        handle_cusolver, numARows, nnzA,
        descrA, d_A, d_A_RowIndices, d_A_ColIndices,
        chl_info,
        &size_internal,
        &size_chol));   
     
    void *buffer_gpu = NULL; 

    gpuErrchk(hipMalloc(&buffer_gpu, sizeof(char)*size_chol)); 
   
     cusolverSafeCall(cusolverSpDcsrcholFactor(
        handle_cusolver, numARows, nnzA,
        descrA, d_A, d_A_RowIndices, d_A_ColIndices,
        chl_info,
        buffer_gpu));
    
    cusolverSafeCall(cusolverSpDcsrcholZeroPivot(
        handle_cusolver, chl_info, tol, &singularity));
    
    if ( 0 <= singularity){
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                       "Invalid input to MEX file, (fatal error:) A is not invertible, singularity=%d\n", singularity);
       
    }
    
    size_t pivot_dimensionsvalueVa[1] = {numAColumns};

    mxGPUArray *VAL = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionsvalueVa, mxDOUBLE_CLASS, mxREAL, MX_GPU_INITIALIZE_VALUES);
    double  *VALOUT = (double *)mxGPUGetData(VAL);

    cusolverSafeCall(cusolverSpDcsrcholSolve(
        handle_cusolver, numARows, d_B_dense, VALOUT, chl_info, buffer_gpu));
    

        mxGPUDestroyGPUArray(A);
        mxGPUDestroyGPUArray(ROW_A);
        mxGPUDestroyGPUArray(COL_A);   
        mxGPUDestroyGPUArray(OUTMB);
		
OUTPUTMATRIX = mxGPUCreateMxArrayOnGPU(VAL);             
gpuErrchk(hipFree(buffer_gpu));       
mxGPUDestroyGPUArray(VAL);
cusolverSpDestroyCsrcholInfo(chl_info);      
hipsparseDestroyMatDescr(descrA);
hipsolverSpDestroy(handle_cusolver);
hipsparseDestroy(handle);  
  
    }
    else{
        mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Incorrect input arguments! %s\n");    
        }
    
 }
        //
    else{
        mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Incorrect input arguments! %s\n");    
        }

}
