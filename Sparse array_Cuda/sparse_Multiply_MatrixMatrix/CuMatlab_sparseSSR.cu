
/*
 * This CUDA-Cusparse code can handle/work with  any type of the input mxArrays, 
 * GPUarray or standard matlab CPU array as input {prhs[0]/prhs[1] := mxGPUArray or CPU Array}[double/complex double]
 * Sparse/Dense matrix-sparse/dense matrix multiplication   Z=CuMatlab_multiply(Sparse/Dense(X),Sparse/Dense(Y)).
 * Developed at UCL, Institute of Neurology, 12 Queen Square, WC1N 3AR, London
 * Wellcome Trust Centre for Neuroimaging
 * Part of the project SPM(http://www.fil.ion.ucl.ac.uk/spm)
 * Copyright 2018
 * Kevin Bronik
 */

#include "matrix.h"
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <hipsparse.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "SPARSEHELPER.h"
#include "ERRORCHK.h"
#include <omp.h>
// Input Arguments
#define	INPUTSPARSEA   prhs[0]
#define	INPUTSPARSEB   prhs[1]

// Output Arguments
#define	OUTPUTMATRIX  plhs[0]



  
    
extern "C" static void mexCuMatlab_sparseSSR(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
 int nDevices;
hipError_t errCode =hipGetDeviceCount(&nDevices); 
//int nDevices;
//hipGetDeviceCount(&nDevices);

if (errCode != hipSuccess){
printf("Error! No CUDA devices found! \n");
return;
}

    char const * const InputErrMsg = "Invalid input to MEX file, number of input arguments must be two.";
    char const * const OutputErrMsg = "Invalid output to MEX file, number of output arguments must be one.";
   if ((nrhs!=2)) {
        mexErrMsgIdAndTxt("MATLAB:mexatexit:invalidInput", InputErrMsg);
    }
   if ((nlhs!=1)) {
        mexErrMsgIdAndTxt("MATLAB:mexatexit:invalidInput", OutputErrMsg);
    }
 char *input_buf0;
 input_buf0 = mxArrayToString(INPUTSPARSEA);

      if ((mxIsChar(INPUTSPARSEA))){
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Input(FIRST ARGUMENT) must be array, or gpuArray object not  %s\n",input_buf0);
    }
    
 char *input_buf1;
 input_buf1 = mxArrayToString(INPUTSPARSEB);

      if ((mxIsChar(INPUTSPARSEB))){
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Input(SECOND ARGUMENT) must be array, or gpuArray object not  %s\n",input_buf1);
    } 



if (mxIsGPUArray(INPUTSPARSEA) && mxIsGPUArray(INPUTSPARSEB)) {
    
    mxGPUArray const *INPUTSPARSEGPUA;
    mxGPUArray const *INPUTSPARSEGPUB;
    
    
    /* Initialize the MathWorks GPU API. */
    mxInitGPU();
    INPUTSPARSEGPUA = mxGPUCreateFromMxArray(INPUTSPARSEA);
    INPUTSPARSEGPUB = mxGPUCreateFromMxArray(INPUTSPARSEB);
    
   
	
    if((mxGPUIsSparse(INPUTSPARSEGPUA))&& (mxGPUIsSparse(INPUTSPARSEGPUB)) ){
        
    const mwSize *dimsGPUSA;
    dimsGPUSA=mxGPUGetDimensions(INPUTSPARSEGPUA);
    int numARows, numAColumns;
    numARows = (int)dimsGPUSA[0]; /* gets number of rows of A */
    numAColumns = (int)dimsGPUSA[1]; /* gets number of columns of A */
    
    const mwSize *dimsGPUSB;
    dimsGPUSB=mxGPUGetDimensions(INPUTSPARSEGPUB);
    int numBRows, numBColumns;
    numBRows = (int)dimsGPUSB[0]; /* gets number of rows of B */
    numBColumns = (int)dimsGPUSB[1]; /* gets number of columns of B */
    if ( numAColumns != numBRows) {
		
		mxGPUDestroyGPUArray(INPUTSPARSEGPUA);
        mxGPUDestroyGPUArray(INPUTSPARSEGPUB);
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Invalid input to MEX file, column number of sparse matrix(first argument) must match the row number of sparse matrix(second argument).");
             
    }
    mwIndex nnz1;
     mxArray * VLSXY1 = mxGPUCreateMxArrayOnCPU(INPUTSPARSEGPUA);
    nnz1 = *(mxGetJc(VLSXY1) + numAColumns);
    int nnzA = (int)nnz1;
    
    mwIndex nnz2;
    mxArray * VLSXY2 = mxGPUCreateMxArrayOnCPU(INPUTSPARSEGPUB);
    nnz2 = *(mxGetJc(VLSXY2) + numBColumns);
    int nnzB = (int)nnz2;
    
    
    
   
   mxArray *  ROW_SORTA = mxCreateNumericMatrix(nnzA, 1,mxINT32_CLASS, mxREAL);
    int *ROWSORTA  = (int *)mxGetInt32s(ROW_SORTA);
       SetIr_Data(VLSXY1, ROWSORTA);
    
   mxArray *  COL_SORTA = mxCreateNumericMatrix(nnzA, 1, mxINT32_CLASS, mxREAL);
    int  *COLSORTA = (int *)mxGetInt32s(COL_SORTA);
          SetJc_Int(VLSXY1, COLSORTA);
      
 
    double  *VALSORTA = (double *)mxGetDoubles(VLSXY1);
           
           
    mxArray *  ROW_SORTB = mxCreateNumericMatrix(nnzB, 1,mxINT32_CLASS, mxREAL);
    int *ROWSORTB  = (int *)mxGetInt32s(ROW_SORTB);
       SetIr_Data(VLSXY2, ROWSORTB);
    
   mxArray *  COL_SORTB = mxCreateNumericMatrix(nnzB, 1, mxINT32_CLASS, mxREAL);
    int  *COLSORTB = (int *)mxGetInt32s(COL_SORTB);
          SetJc_Int(VLSXY2, COLSORTB);
      
    double  *VALSORTB = (double *)mxGetDoubles(VLSXY2);

      
	hipsparseHandle_t handle;    cusparseSafeCall(hipsparseCreate(&handle));

	//double *d_A;            //gpuErrchk(hipMalloc(&d_A, nnzA * sizeof(*d_A)));
	//int *d_A_RowIndices;    //gpuErrchk(hipMalloc(&d_A_RowIndices, (numARows + 1) * sizeof(*d_A_RowIndices)));
	//int *d_A_ColIndices;   // gpuErrchk(hipMalloc(&d_A_ColIndices, nnzA * sizeof(*d_A_ColIndices)));
	//int *d_cooRowIndA;      // gpuErrchk(hipMalloc(&d_cooRowIndA, nnzA * sizeof(*d_cooRowIndA)));
   
   size_t pivot_dimensA[1] = {nnzA};
   size_t pivot_dimensROW_A[1] = {numARows+1};
   size_t pivot_dimensCOL_A[1] = {nnzA};
   size_t pivot_dimensCOO_A[1] = {nnzA};
   
   mxGPUArray *A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensA, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    double  *d_A = (double *)mxGPUGetData(A);
   mxGPUArray * ROW_A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensROW_A, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_A_RowIndices = (int *)mxGPUGetData(ROW_A);
   mxGPUArray * COL_A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensCOL_A, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_A_ColIndices = (int *)mxGPUGetData(COL_A);
    mxGPUArray * COO_A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensCOO_A, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_cooRowIndA = (int *)mxGPUGetData(COO_A);  

    
    
    
    
	// --- Descriptor for sparse matrix A
	gpuErrchk(hipMemcpy(d_A, VALSORTA, nnzA * sizeof(*d_A), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_A_ColIndices, COLSORTA, nnzA * sizeof(*d_A_ColIndices), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_cooRowIndA, ROWSORTA, nnzA * sizeof(*d_cooRowIndA), hipMemcpyHostToDevice));
         mxGPUDestroyGPUArray(INPUTSPARSEGPUA);
         

	hipsparseMatDescr_t descrA;      cusparseSafeCall(hipsparseCreateMatDescr(&descrA));
	hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE);

	int *Pa = NULL;
	void *pBuffera = NULL;
	size_t pBufferSizeInBytesa = 0;
	hipsparseXcoosort_bufferSizeExt(handle, numARows, numAColumns,
		nnzA,
		d_cooRowIndA,
		d_A_ColIndices, &pBufferSizeInBytesa);

	gpuErrchk(hipMalloc(&pBuffera, sizeof(char)*pBufferSizeInBytesa));
	gpuErrchk(hipMalloc(&Pa, sizeof(int)*nnzA));
	hipsparseCreateIdentityPermutation(handle, nnzA, Pa);
	cusparseSafeCall(hipsparseXcoosortByRow(handle, numARows, numAColumns,
		nnzA,
		d_cooRowIndA,
		d_A_ColIndices,
		Pa,
		pBuffera));

	cusparseSafeCall(hipsparseDgthr(handle, nnzA, d_A, d_A, Pa, HIPSPARSE_INDEX_BASE_ZERO));

	cusparseSafeCall(hipsparseXcoo2csr(handle,
		d_cooRowIndA,
		nnzA,
		numARows,
		d_A_RowIndices,
		HIPSPARSE_INDEX_BASE_ONE));
		
	//gpuErrchk(hipFree(d_cooRowIndA));	
	gpuErrchk(hipFree(pBuffera));
	gpuErrchk(hipFree(Pa));

	
	///double *d_B;           // gpuErrchk(hipMalloc(&d_B, nnzB * sizeof(*d_B)));
	////int *d_B_RowIndices;   // gpuErrchk(hipMalloc(&d_B_RowIndices, (numBRows + 1) * sizeof(*d_B_RowIndices)));
	//int *d_B_ColIndices;   // gpuErrchk(hipMalloc(&d_B_ColIndices, nnzB * sizeof(*d_B_ColIndices)));
	//int *d_cooRowIndB;     //  gpuErrchk(hipMalloc(&d_cooRowIndB, nnzB * sizeof(*d_cooRowIndB)));
    
   size_t pivot_dimensB[1] = {nnzB};
   size_t pivot_dimensROW_B[1] = {numBRows+1};
   size_t pivot_dimensCOL_B[1] = {nnzB};
   size_t pivot_dimensCOO_B[1] = {nnzB};
   
   mxGPUArray *B = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensB, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    double  *d_B = (double *)mxGPUGetData(B);
   mxGPUArray * ROW_B = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensROW_B, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_B_RowIndices = (int *)mxGPUGetData(ROW_B);
   mxGPUArray * COL_B = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensCOL_B, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_B_ColIndices = (int *)mxGPUGetData(COL_B);
    mxGPUArray * COO_B = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensCOO_B, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_cooRowIndB = (int *)mxGPUGetData(COO_B);
    
    
	// --- Descriptor for sparse matrix A
	gpuErrchk(hipMemcpy(d_B, VALSORTB, nnzB * sizeof(*d_B), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_B_ColIndices, COLSORTB, nnzB * sizeof(*d_B_ColIndices), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_cooRowIndB, ROWSORTB, nnzB * sizeof(*d_cooRowIndB), hipMemcpyHostToDevice));
	
		mxGPUDestroyGPUArray(INPUTSPARSEGPUB);

	int *Pb = NULL;
	void *pBufferb = NULL;
	size_t pBufferSizeInBytesb = 0;
	hipsparseXcoosort_bufferSizeExt(handle, numBRows, numBColumns,
		nnzB,
		d_cooRowIndB,
		d_B_ColIndices, &pBufferSizeInBytesb);

	gpuErrchk(hipMalloc(&pBufferb, sizeof(char)*pBufferSizeInBytesb));
	gpuErrchk(hipMalloc(&Pb, sizeof(int)*nnzB));
	hipsparseCreateIdentityPermutation(handle, nnzB, Pb);
	cusparseSafeCall(hipsparseXcoosortByRow(handle, numBRows, numBColumns,
		nnzB,
		d_cooRowIndB,
		d_B_ColIndices,
		Pb,
		pBufferb));

	cusparseSafeCall(hipsparseDgthr(handle, nnzB, d_B, d_B, Pb, HIPSPARSE_INDEX_BASE_ZERO));

	cusparseSafeCall(hipsparseXcoo2csr(handle,
		d_cooRowIndB,
		nnzB,
		numBRows,
		d_B_RowIndices,
		HIPSPARSE_INDEX_BASE_ONE));
		
		
    gpuErrchk(hipFree(pBufferb));
	gpuErrchk(hipFree(Pb));
	//gpuErrchk(hipFree(d_cooRowIndB));
		

	cusparseSafeCall(hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST));

	int nnzC = 0;
	int baseC = 0;
// nnzTotalDevHostPtr points to host memory
    int *nnzTotalDevHostPtr = &nnzC;
	//int *d_C_RowIndices;   // gpuErrchk(hipMalloc((void **)&d_C_RowIndices, sizeof(int)*(numARows + 1)));
   
   size_t pivot_dimensROW_C[1] = {numARows+1};
   
   mxGPUArray * ROW_C = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensROW_C, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_C_RowIndices = (int *)mxGPUGetData(ROW_C);
    
    
    
    
	// where op ( A ) , op ( B ) and C are m�k (numARows�numAColumns), k�n(numBRows�numBColumns), and m�n(numARows�numBColumns) sparse matrices
	cusparseSafeCall(hipsparseXcsrgemmNnz(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, numARows, numBColumns, numAColumns,
		descrA, nnzA, d_A_RowIndices, d_A_ColIndices,
		descrA, nnzB, d_B_RowIndices, d_B_ColIndices,
		descrA, d_C_RowIndices, nnzTotalDevHostPtr ));
       
        
        
     if (NULL != nnzTotalDevHostPtr){
    nnzC = *nnzTotalDevHostPtr;
    }else{
    hipMemcpy(&nnzC, d_C_RowIndices+numARows, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&baseC, d_C_RowIndices, sizeof(int), hipMemcpyDeviceToHost);
    nnzC -= baseC;
     }


    
   if (nnzC==0) {
          
         OUTPUTMATRIX = mxCreateSparse(numARows,numBColumns,0,mxREAL);
         
         return;
             
    } 
    
    
   size_t pivot_dimensionsrow[1] = {nnzC};
   size_t pivot_dimensionscolumn[1] = {numBColumns+1}; 
   size_t pivot_dimensionsvalue[1] = {nnzC};
   mxGPUArray * ROW_SORTC = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionsrow, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *ROWSORTC = (int *)mxGPUGetData(ROW_SORTC);
   mxGPUArray * COL_SORTC = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionscolumn, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *COLSORTC = (int *)mxGPUGetData(COL_SORTC);
    mxGPUArray *VAL_SORTC = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionsvalue, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    double  *VALSORTC = (double *)mxGPUGetData(VAL_SORTC);
   mwSize nnzm=(mwSize)nnzC;
   
   OUTPUTMATRIX = mxCreateSparse(numARows,numBColumns,nnzm,mxREAL);
    
    

        
    //double *d_C;        //  gpuErrchk(hipMalloc((void **)&d_C, sizeof(double)*(nnzC)));
   // int *d_C_ColIndices;   //gpuErrchk(hipMalloc((void **)&d_C_ColIndices, sizeof(int)*(nnzC)));
	size_t pivot_dimensC[1] = {nnzC};
    size_t pivot_dimensCOL_C[1] = {nnzC};
   
   
   mxGPUArray *C = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensC, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    double  *d_C = (double *)mxGPUGetData(C);
   mxGPUArray * COL_C = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensCOL_C, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_C_ColIndices = (int *)mxGPUGetData(COL_C);
    
    
    
   	cusparseSafeCall(hipsparseDcsrgemm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, numARows, numBColumns, numAColumns,
		descrA, nnzA,
		d_A, d_A_RowIndices, d_A_ColIndices,
		descrA, nnzB,
		d_B, d_B_RowIndices, d_B_ColIndices,
		descrA,
		d_C, d_C_RowIndices, d_C_ColIndices));
        
	//gpuErrchk(hipFree(d_A));
	//gpuErrchk(hipFree(d_A_RowIndices));
	//gpuErrchk(hipFree(d_A_ColIndices));
	
	//gpuErrchk(hipFree(d_B));
	//gpuErrchk(hipFree(d_B_RowIndices));
	//gpuErrchk(hipFree(d_B_ColIndices));
    

    mxGPUDestroyGPUArray(A);
    mxGPUDestroyGPUArray(ROW_A);
    mxGPUDestroyGPUArray(COL_A);
    mxGPUDestroyGPUArray(COO_A);
    mxGPUDestroyGPUArray(B);
    mxGPUDestroyGPUArray(ROW_B);
    mxGPUDestroyGPUArray(COL_B);
    mxGPUDestroyGPUArray(COO_B);
    
    
    
	cusparseSafeCall(hipsparseDcsr2csc(handle, numARows, numBColumns, nnzC, d_C, d_C_RowIndices, d_C_ColIndices, VALSORTC, ROWSORTC, COLSORTC, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ONE));

  
    
   mxArray *RS= mxGPUCreateMxArrayOnCPU(ROW_SORTC);
   int * rs= (int *)mxGetInt32s(RS);
   mxArray *CS= mxGPUCreateMxArrayOnCPU(COL_SORTC);
   int * cs= (int *)mxGetInt32s(CS);

    
      mwIndex *irs,*jcs;
  
        irs = static_cast<mwIndex *> (mxMalloc (nnzC * sizeof(mwIndex)));
                          int i;
	   #pragma omp parallel for shared(nnzC) private(i)
         for (i = 0; i < nnzC; ++i) {
           irs[i] = static_cast<mwIndex> (rs[i])-1;  
            }
      
      jcs = static_cast<mwIndex *> (mxMalloc ((numBColumns+1) * sizeof(mwIndex)));
      int nc1= numBColumns+1;
       #pragma omp parallel for shared(nc1) private(i)
            for (i = 0; i < nc1; ++i) {
           jcs[i] = static_cast<mwIndex> (cs[i])-1;
            }
             
        mxDouble* PRS = (mxDouble*) mxMalloc (nnzC * sizeof(mxDouble));
        gpuErrchk(hipMemcpy(PRS, VALSORTC, nnzC * sizeof(mxDouble), hipMemcpyDeviceToHost));          

           
    
   
        mxFree (mxGetJc (OUTPUTMATRIX)) ;
        mxFree (mxGetIr (OUTPUTMATRIX)) ;
        mxFree (mxGetDoubles (OUTPUTMATRIX)) ;
    
        mxSetIr(OUTPUTMATRIX, (mwIndex *)irs);
        mxSetJc(OUTPUTMATRIX, (mwIndex *)jcs);
        int s=mxSetDoubles(OUTPUTMATRIX, (mxDouble *)PRS);
         if ( s == 0) {
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "the function is unsuccessful, either mxArray is not an unshared mxDOUBLE_CLASS array, or the data is not allocated with mxCalloc.");
             
         }
    
	//gpuErrchk(hipFree(d_C));
	//gpuErrchk(hipFree(d_C_RowIndices));
	//gpuErrchk(hipFree(d_C_ColIndices));
      mxGPUDestroyGPUArray(C);
      mxGPUDestroyGPUArray(ROW_C);
      mxGPUDestroyGPUArray(COL_C);
	//gpuErrchk(hipFree(d_value_csc));
	//gpuErrchk(hipFree(d_row_csc));
	//gpuErrchk(hipFree(d_col_csc));



         mxDestroyArray(VLSXY1);
         mxDestroyArray(VLSXY2);
         mxGPUDestroyGPUArray(VAL_SORTC);
         mxGPUDestroyGPUArray(ROW_SORTC);
         mxGPUDestroyGPUArray(COL_SORTC);
         mxDestroyArray(RS);
         mxDestroyArray(CS);

         
         mxDestroyArray(COL_SORTA);
         mxDestroyArray(ROW_SORTA);
         
         mxDestroyArray(COL_SORTB);
         mxDestroyArray(ROW_SORTB);
        hipsparseDestroyMatDescr(descrA);  
		hipsparseDestroy(handle);
        
        }
    
        else{
        mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Incorrect input arguments! %s\n");    
        }
    
   }
     
////////////////////////////////////////////////////////////////////////////////////  
    else if (!(mxIsGPUArray(INPUTSPARSEA)) && !(mxIsGPUArray(INPUTSPARSEB))){
   
     // if ((mxGetClassID(INPUTSPARSEA) != mxDOUBLE_CLASS) || (mxGetClassID(INPUTSPARSEB) != mxDOUBLE_CLASS)) {
       //  mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
           //     "Invalid input to MEX file, input(FIRST and SECOND  ARGUMENTS) must be  double precision.");
             
   // }
    if((mxIsSparse(INPUTSPARSEA))&& (mxIsSparse(INPUTSPARSEB)) ){
    
     mxInitGPU();
    const mwSize *dimsCPUA;
    dimsCPUA=mxGetDimensions(INPUTSPARSEA);
    
    int  numARows = (int)dimsCPUA[0]; /* gets number of rows of A */
    int  numAColumns = (int)dimsCPUA[1]; /* gets number of columns of A */
   
    const mwSize *dimsCPUB;
    dimsCPUB=mxGetDimensions(INPUTSPARSEB);
    
    int  numBRows = (int)dimsCPUB[0]; /* gets number of rows of B */
    int  numBColumns = (int)dimsCPUB[1]; /* gets number of columns of B */
    if ( numAColumns != numBRows) {
	
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Invalid input to MEX file, column number of sparse matrix(first argument) must match the row number of sparse matrix(second argument).");
             
    }
    
    mwIndex nnz1;
 
    nnz1 = *(mxGetJc(INPUTSPARSEA) + numAColumns);
    int nnzA = (int)nnz1;
    
    mwIndex nnz2;
 
    nnz2 = *(mxGetJc(INPUTSPARSEB) + numBColumns);
    int nnzB = (int)nnz2;
    
    
    
   
   mxArray *  ROW_SORTA = mxCreateNumericMatrix(nnzA, 1,mxINT32_CLASS, mxREAL);
    int *ROWSORTA  = (int *)mxGetInt32s(ROW_SORTA);
       SetIr_Data(INPUTSPARSEA, ROWSORTA);

    
   mxArray *  COL_SORTA = mxCreateNumericMatrix(nnzA, 1, mxINT32_CLASS, mxREAL);
    int  *COLSORTA = (int *)mxGetInt32s(COL_SORTA);
          SetJc_Int(INPUTSPARSEA, COLSORTA);

      
    double  *VALSORTA = (double  *)mxGetDoubles(INPUTSPARSEA);

       
    mxArray *  ROW_SORTB = mxCreateNumericMatrix(nnzB, 1,mxINT32_CLASS, mxREAL);
    int *ROWSORTB  = (int *)mxGetInt32s(ROW_SORTB);
       SetIr_Data(INPUTSPARSEB, ROWSORTB);

    
   mxArray *  COL_SORTB = mxCreateNumericMatrix(nnzB, 1, mxINT32_CLASS, mxREAL);
    int  *COLSORTB = (int *)mxGetInt32s(COL_SORTB);
          SetJc_Int(INPUTSPARSEB, COLSORTB);

      
    double  *VALSORTB = (double *)mxGetDoubles(INPUTSPARSEB);

     	hipsparseHandle_t handle;    cusparseSafeCall(hipsparseCreate(&handle));

	//double *d_A;            //gpuErrchk(hipMalloc(&d_A, nnzA * sizeof(*d_A)));
	//int *d_A_RowIndices;    //gpuErrchk(hipMalloc(&d_A_RowIndices, (numARows + 1) * sizeof(*d_A_RowIndices)));
	//int *d_A_ColIndices;   // gpuErrchk(hipMalloc(&d_A_ColIndices, nnzA * sizeof(*d_A_ColIndices)));
	//int *d_cooRowIndA;      // gpuErrchk(hipMalloc(&d_cooRowIndA, nnzA * sizeof(*d_cooRowIndA)));
   
   size_t pivot_dimensA[1] = {nnzA};
   size_t pivot_dimensROW_A[1] = {numARows+1};
   size_t pivot_dimensCOL_A[1] = {nnzA};
   size_t pivot_dimensCOO_A[1] = {nnzA};
   
   mxGPUArray *A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensA, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    double  *d_A = (double *)mxGPUGetData(A);
   mxGPUArray * ROW_A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensROW_A, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_A_RowIndices = (int *)mxGPUGetData(ROW_A);
   mxGPUArray * COL_A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensCOL_A, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_A_ColIndices = (int *)mxGPUGetData(COL_A);
    mxGPUArray * COO_A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensCOO_A, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_cooRowIndA = (int *)mxGPUGetData(COO_A);  

    
    
    
    
	// --- Descriptor for sparse matrix A
	gpuErrchk(hipMemcpy(d_A, VALSORTA, nnzA * sizeof(*d_A), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_A_ColIndices, COLSORTA, nnzA * sizeof(*d_A_ColIndices), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_cooRowIndA, ROWSORTA, nnzA * sizeof(*d_cooRowIndA), hipMemcpyHostToDevice));


	hipsparseMatDescr_t descrA;      cusparseSafeCall(hipsparseCreateMatDescr(&descrA));
	hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE);

	int *Pa = NULL;
	void *pBuffera = NULL;
	size_t pBufferSizeInBytesa = 0;
	hipsparseXcoosort_bufferSizeExt(handle, numARows, numAColumns,
		nnzA,
		d_cooRowIndA,
		d_A_ColIndices, &pBufferSizeInBytesa);

	gpuErrchk(hipMalloc(&pBuffera, sizeof(char)*pBufferSizeInBytesa));
	gpuErrchk(hipMalloc(&Pa, sizeof(int)*nnzA));
	hipsparseCreateIdentityPermutation(handle, nnzA, Pa);
	cusparseSafeCall(hipsparseXcoosortByRow(handle, numARows, numAColumns,
		nnzA,
		d_cooRowIndA,
		d_A_ColIndices,
		Pa,
		pBuffera));

	cusparseSafeCall(hipsparseDgthr(handle, nnzA, d_A, d_A, Pa, HIPSPARSE_INDEX_BASE_ZERO));

	cusparseSafeCall(hipsparseXcoo2csr(handle,
		d_cooRowIndA,
		nnzA,
		numARows,
		d_A_RowIndices,
		HIPSPARSE_INDEX_BASE_ONE));
		
	//gpuErrchk(hipFree(d_cooRowIndA));	
	gpuErrchk(hipFree(pBuffera));
	gpuErrchk(hipFree(Pa));

	
	///double *d_B;           // gpuErrchk(hipMalloc(&d_B, nnzB * sizeof(*d_B)));
	////int *d_B_RowIndices;   // gpuErrchk(hipMalloc(&d_B_RowIndices, (numBRows + 1) * sizeof(*d_B_RowIndices)));
	//int *d_B_ColIndices;   // gpuErrchk(hipMalloc(&d_B_ColIndices, nnzB * sizeof(*d_B_ColIndices)));
	//int *d_cooRowIndB;     //  gpuErrchk(hipMalloc(&d_cooRowIndB, nnzB * sizeof(*d_cooRowIndB)));
    
   size_t pivot_dimensB[1] = {nnzB};
   size_t pivot_dimensROW_B[1] = {numBRows+1};
   size_t pivot_dimensCOL_B[1] = {nnzB};
   size_t pivot_dimensCOO_B[1] = {nnzB};
   
   mxGPUArray *B = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensB, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    double  *d_B = (double *)mxGPUGetData(B);
   mxGPUArray * ROW_B = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensROW_B, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_B_RowIndices = (int *)mxGPUGetData(ROW_B);
   mxGPUArray * COL_B = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensCOL_B, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_B_ColIndices = (int *)mxGPUGetData(COL_B);
    mxGPUArray * COO_B = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensCOO_B, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_cooRowIndB = (int *)mxGPUGetData(COO_B);
    
    
	// --- Descriptor for sparse matrix A
	gpuErrchk(hipMemcpy(d_B, VALSORTB, nnzB * sizeof(*d_B), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_B_ColIndices, COLSORTB, nnzB * sizeof(*d_B_ColIndices), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_cooRowIndB, ROWSORTB, nnzB * sizeof(*d_cooRowIndB), hipMemcpyHostToDevice));
	
		

	int *Pb = NULL;
	void *pBufferb = NULL;
	size_t pBufferSizeInBytesb = 0;
	hipsparseXcoosort_bufferSizeExt(handle, numBRows, numBColumns,
		nnzB,
		d_cooRowIndB,
		d_B_ColIndices, &pBufferSizeInBytesb);

	gpuErrchk(hipMalloc(&pBufferb, sizeof(char)*pBufferSizeInBytesb));
	gpuErrchk(hipMalloc(&Pb, sizeof(int)*nnzB));
	hipsparseCreateIdentityPermutation(handle, nnzB, Pb);
	cusparseSafeCall(hipsparseXcoosortByRow(handle, numBRows, numBColumns,
		nnzB,
		d_cooRowIndB,
		d_B_ColIndices,
		Pb,
		pBufferb));

	cusparseSafeCall(hipsparseDgthr(handle, nnzB, d_B, d_B, Pb, HIPSPARSE_INDEX_BASE_ZERO));

	cusparseSafeCall(hipsparseXcoo2csr(handle,
		d_cooRowIndB,
		nnzB,
		numBRows,
		d_B_RowIndices,
		HIPSPARSE_INDEX_BASE_ONE));
		
		
    gpuErrchk(hipFree(pBufferb));
	gpuErrchk(hipFree(Pb));
	//gpuErrchk(hipFree(d_cooRowIndB));
		

	cusparseSafeCall(hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST));

	int nnzC = 0;
	int baseC = 0;
// nnzTotalDevHostPtr points to host memory
    int *nnzTotalDevHostPtr = &nnzC;
	//int *d_C_RowIndices;   // gpuErrchk(hipMalloc((void **)&d_C_RowIndices, sizeof(int)*(numARows + 1)));
   
   size_t pivot_dimensROW_C[1] = {numARows+1};
   
   mxGPUArray * ROW_C = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensROW_C, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_C_RowIndices = (int *)mxGPUGetData(ROW_C);
    
    
    
    
	// where op ( A ) , op ( B ) and C are m�k (numARows�numAColumns), k�n(numBRows�numBColumns), and m�n(numARows�numBColumns) sparse matrices
	cusparseSafeCall(hipsparseXcsrgemmNnz(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, numARows, numBColumns, numAColumns,
		descrA, nnzA, d_A_RowIndices, d_A_ColIndices,
		descrA, nnzB, d_B_RowIndices, d_B_ColIndices,
		descrA, d_C_RowIndices, nnzTotalDevHostPtr ));
       
        
        
     if (NULL != nnzTotalDevHostPtr){
    nnzC = *nnzTotalDevHostPtr;
    }else{
    hipMemcpy(&nnzC, d_C_RowIndices+numARows, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&baseC, d_C_RowIndices, sizeof(int), hipMemcpyDeviceToHost);
    nnzC -= baseC;
     }


    
   if (nnzC==0) {
          
         OUTPUTMATRIX = mxCreateSparse(numARows,numBColumns,0,mxREAL);
         
         return;
             
    } 
    
    
   size_t pivot_dimensionsrow[1] = {nnzC};
   size_t pivot_dimensionscolumn[1] = {numBColumns+1}; 
   size_t pivot_dimensionsvalue[1] = {nnzC};
   mxGPUArray * ROW_SORTC = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionsrow, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *ROWSORTC = (int *)mxGPUGetData(ROW_SORTC);
   mxGPUArray * COL_SORTC = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionscolumn, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *COLSORTC = (int *)mxGPUGetData(COL_SORTC);
    mxGPUArray *VAL_SORTC = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionsvalue, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    double  *VALSORTC = (double *)mxGPUGetData(VAL_SORTC);
   mwSize nnzm=(mwSize)nnzC;
   
   OUTPUTMATRIX = mxCreateSparse(numARows,numBColumns,nnzm,mxREAL);
    
    

        
    //double *d_C;        //  gpuErrchk(hipMalloc((void **)&d_C, sizeof(double)*(nnzC)));
   // int *d_C_ColIndices;   //gpuErrchk(hipMalloc((void **)&d_C_ColIndices, sizeof(int)*(nnzC)));
	size_t pivot_dimensC[1] = {nnzC};
    size_t pivot_dimensCOL_C[1] = {nnzC};
   
   
   mxGPUArray *C = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensC, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    double  *d_C = (double *)mxGPUGetData(C);
   mxGPUArray * COL_C = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensCOL_C, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_C_ColIndices = (int *)mxGPUGetData(COL_C);
    
    
    
   	cusparseSafeCall(hipsparseDcsrgemm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, numARows, numBColumns, numAColumns,
		descrA, nnzA,
		d_A, d_A_RowIndices, d_A_ColIndices,
		descrA, nnzB,
		d_B, d_B_RowIndices, d_B_ColIndices,
		descrA,
		d_C, d_C_RowIndices, d_C_ColIndices));
        
	//gpuErrchk(hipFree(d_A));
	//gpuErrchk(hipFree(d_A_RowIndices));
	//gpuErrchk(hipFree(d_A_ColIndices));
	
	//gpuErrchk(hipFree(d_B));
	//gpuErrchk(hipFree(d_B_RowIndices));
	//gpuErrchk(hipFree(d_B_ColIndices));
    

    mxGPUDestroyGPUArray(A);
    mxGPUDestroyGPUArray(ROW_A);
    mxGPUDestroyGPUArray(COL_A);
    mxGPUDestroyGPUArray(COO_A);
    mxGPUDestroyGPUArray(B);
    mxGPUDestroyGPUArray(ROW_B);
    mxGPUDestroyGPUArray(COL_B);
    mxGPUDestroyGPUArray(COO_B);
    
    
    
	cusparseSafeCall(hipsparseDcsr2csc(handle, numARows, numBColumns, nnzC, d_C, d_C_RowIndices, d_C_ColIndices, VALSORTC, ROWSORTC, COLSORTC, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ONE));

  
    
   mxArray *RS= mxGPUCreateMxArrayOnCPU(ROW_SORTC);
   int * rs= (int *)mxGetInt32s(RS);
   mxArray *CS= mxGPUCreateMxArrayOnCPU(COL_SORTC);
   int * cs= (int *)mxGetInt32s(CS);

    
      mwIndex *irs,*jcs;
  
        irs = static_cast<mwIndex *> (mxMalloc (nnzC * sizeof(mwIndex)));
                          int i;
	   #pragma omp parallel for shared(nnzC) private(i)
         for (i = 0; i < nnzC; ++i) {
           irs[i] = static_cast<mwIndex> (rs[i])-1;  
            }
      
      jcs = static_cast<mwIndex *> (mxMalloc ((numBColumns+1) * sizeof(mwIndex)));
      int nc1= numBColumns+1;
       #pragma omp parallel for shared(nc1) private(i)
            for (i = 0; i < nc1; ++i) {
           jcs[i] = static_cast<mwIndex> (cs[i])-1;
            }
             
        mxDouble* PRS = (mxDouble*) mxMalloc (nnzC * sizeof(mxDouble));
        gpuErrchk(hipMemcpy(PRS, VALSORTC, nnzC * sizeof(mxDouble), hipMemcpyDeviceToHost));         

  
   
        mxFree (mxGetJc (OUTPUTMATRIX)) ;
        mxFree (mxGetIr (OUTPUTMATRIX)) ;
        mxFree (mxGetDoubles (OUTPUTMATRIX)) ;
    
        mxSetIr(OUTPUTMATRIX, (mwIndex *)irs);
        mxSetJc(OUTPUTMATRIX, (mwIndex *)jcs);
        int s=mxSetDoubles(OUTPUTMATRIX, (mxDouble *)PRS);
            if ( s == 0) {
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "the function is unsuccessful, either mxArray is not an unshared mxDOUBLE_CLASS array, or the data is not allocated with mxCalloc.");
             
         }
    
	//gpuErrchk(hipFree(d_C));
	//gpuErrchk(hipFree(d_C_RowIndices));
	//gpuErrchk(hipFree(d_C_ColIndices));
      mxGPUDestroyGPUArray(C);
      mxGPUDestroyGPUArray(ROW_C);
      mxGPUDestroyGPUArray(COL_C);
	//gpuErrchk(hipFree(d_value_csc));
	//gpuErrchk(hipFree(d_row_csc));
	//gpuErrchk(hipFree(d_col_csc));



         mxGPUDestroyGPUArray(VAL_SORTC);
         mxGPUDestroyGPUArray(ROW_SORTC);
         mxGPUDestroyGPUArray(COL_SORTC);
         mxDestroyArray(RS);
         mxDestroyArray(CS);

         
         mxDestroyArray(COL_SORTA);
         mxDestroyArray(ROW_SORTA);
         
         mxDestroyArray(COL_SORTB);
         mxDestroyArray(ROW_SORTB);
        hipsparseDestroyMatDescr(descrA);   
		hipsparseDestroy(handle);
          

    }
    else{
        mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Incorrect input arguments! %s\n");    
        }
    
 }
        //
    else{
        mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Incorrect input arguments! %s\n");    
        }

}
