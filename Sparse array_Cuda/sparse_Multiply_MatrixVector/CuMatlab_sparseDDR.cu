
/*
 * This CUDA-Cusparse code can handle/work with  any type of the input mxArrays, 
 * GPUarray or standard matlab CPU array as input {prhs[0]/prhs[1] := mxGPUArray or CPU Array}[double/complex double]
 * Sparse/Dense matrix-sparse/dense vector multiplication   Z=CuMatlab_multiplyV(Sparse/Dense(X),Sparse/Dense(Y), alpha).
 * Z= alpha*X*Y
 * Developed at UCL, Institute of Neurology, 12 Queen Square, WC1N 3AR, London
 * Wellcome Trust Centre for Neuroimaging
 * Part of the project SPM(http://www.fil.ion.ucl.ac.uk/spm)
 * Copyright 2018
 * Kevin Bronik
 */

#include "matrix.h"
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <hipsparse.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "SPARSEHELPER.h"
#include "ERRORCHK.h"
#include <omp.h>

// Input Arguments
#define	INPUTDENSEA   prhs[0]
#define	INPUTDENSEB   prhs[1]
#define	ALPHA   prhs[2]
//#define	BETA    prhs[3]
// Output Arguments
#define	OUTPUTMATRIX  plhs[0]



  
    
extern "C" static void mexCuMatlab_sparseDDR(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
 int nDevices;
hipError_t errCode =hipGetDeviceCount(&nDevices); 
//int nDevices;
//hipGetDeviceCount(&nDevices);

if (errCode != hipSuccess){
printf("Error! No CUDA devices found! \n");
return;
}

    char const * const InputErrMsg = "Invalid input to MEX file, number of input arguments must be three.";
    char const * const OutputErrMsg = "Invalid output to MEX file, number of output arguments must be one.";
   if ((nrhs!=3)) {
        mexErrMsgIdAndTxt("MATLAB:mexatexit:invalidInput", InputErrMsg);
    }
   if ((nlhs!=1)) {
        mexErrMsgIdAndTxt("MATLAB:mexatexit:invalidInput", OutputErrMsg);
    }
 char *input_buf0;
 input_buf0 = mxArrayToString(INPUTDENSEA);

      if ((mxIsChar(INPUTDENSEA))){
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Input(FIRST ARGUMENT) must be array, or gpuArray object not  %s\n",input_buf0);
    }
    
 char *input_buf1;
 input_buf1 = mxArrayToString(INPUTDENSEB);

      if ((mxIsChar(INPUTDENSEB))){
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Input(SECOND ARGUMENT) must be array, or gpuArray object not  %s\n",input_buf1);
    } 

 char *input_buf2;
 input_buf2 = mxArrayToString(ALPHA);

      if ((mxIsChar(ALPHA))){
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Input(THIRD ARGUMENT) must be scalar not  %s\n",input_buf2);
    } 


if (mxIsGPUArray(INPUTDENSEA) && mxIsGPUArray(INPUTDENSEB)) {
    
    mxGPUArray const *INPUTDENSEGPUA;
    mxGPUArray const *INPUTDENSEGPUB;
    
    
    /* Initialize the MathWorks GPU API. */
    mxInitGPU();
    INPUTDENSEGPUA = mxGPUCreateFromMxArray(INPUTDENSEA);
    INPUTDENSEGPUB = mxGPUCreateFromMxArray(INPUTDENSEB);
    
   
	
    if((!mxGPUIsSparse(INPUTDENSEGPUA))&& (!mxGPUIsSparse(INPUTDENSEGPUB)) ){
        
    const mwSize *dimsGPUSA;
    dimsGPUSA=mxGPUGetDimensions(INPUTDENSEGPUA);
    int numARows, numAColumns;
    numARows = (int)dimsGPUSA[0]; /* gets number of rows of A */
    numAColumns = (int)dimsGPUSA[1]; /* gets number of columns of A */
    
    const mwSize *dimsGPUSB;
    dimsGPUSB=mxGPUGetDimensions(INPUTDENSEGPUB);
    int numBRows, numBColumns;
    
    numBRows = (int)dimsGPUSB[0]; /* gets number of rows of B */
    numBColumns = (int)dimsGPUSB[1]; /* gets number of columns of B */
   if ( (numBColumns!= 1) ) {
              mxGPUDestroyGPUArray(INPUTDENSEGPUB);
              mxGPUDestroyGPUArray(INPUTDENSEGPUA);   
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                        "Invalid input to MEX file, second argument must be a dense column vector.");
             
    }
    
    if ( (numAColumns!= numBRows) ) {
              mxGPUDestroyGPUArray(INPUTDENSEGPUB);
              mxGPUDestroyGPUArray(INPUTDENSEGPUA);   
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                        "Invalid input to MEX file, column number of dense matrix(first argument) must be equal to row numbers of dense vector(second argument).");
             
    }
      const  double alpha= mxGetScalar(ALPHA);
      const  double beta = 0.0;

  double const *d_A_dense;
   d_A_dense = (double const *)(mxGPUGetDataReadOnly(INPUTDENSEGPUA));
        mxGPUDestroyGPUArray(INPUTDENSEGPUA); 
      
	hipsparseHandle_t handle;    cusparseSafeCall(hipsparseCreate(&handle));

	hipsparseMatDescr_t descrA;      cusparseSafeCall(hipsparseCreateMatDescr(&descrA));
	hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE);

	int nnzA = 0;                            // --- Number of nonzero elements in dense matrix A
	const int lda = numARows;
	//int *d_nnzPerVectorA;    //gpuErrchk(hipMalloc(&d_nnzPerVectorA, numARows * sizeof(*d_nnzPerVectorA)));
    size_t pivot_pervect1[1] = {numARows};
    mxGPUArray *PerVect1 = mxGPUCreateGPUArray(1, (mwSize*) pivot_pervect1, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
	int *d_nnzPerVectorA = (int*)mxGPUGetData(PerVect1);
	//double *d_A_dense;  gpuErrchk(hipMalloc(&d_A_dense, numARows * numAColumns * sizeof(*d_A_dense)));
	//gpuErrchk(hipMemcpy(d_A_dense, h_A_dense1, numARows * numAColumns * sizeof(*d_A_dense), hipMemcpyHostToDevice));
	cusparseSafeCall(hipsparseDnnz(handle, HIPSPARSE_DIRECTION_ROW, numARows, numAColumns, descrA, d_A_dense, lda, d_nnzPerVectorA, &nnzA));
        //double *d_A;            //gpuErrchk(hipMalloc(&d_A, nnzA * sizeof(*d_A)));
		//int *d_A_RowIndices;    //gpuErrchk(hipMalloc(&d_A_RowIndices, (numARows + 1) * sizeof(*d_A_RowIndices)));
		//int *d_A_ColIndices;    //gpuErrchk(hipMalloc(&d_A_ColIndices, nnzA * sizeof(*d_A_ColIndices)));
   size_t pivot_dimensA[1] = {nnzA};
   size_t pivot_dimensROW_A[1] = {numARows+1};
   size_t pivot_dimensCOL_A[1] = {nnzA};
   
   
   mxGPUArray *A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensA, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    double  *d_A = (double *)mxGPUGetData(A);
   mxGPUArray * ROW_A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensROW_A, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_A_RowIndices = (int *)mxGPUGetData(ROW_A);
   mxGPUArray * COL_A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensCOL_A, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_A_ColIndices = (int *)mxGPUGetData(COL_A);

		cusparseSafeCall(hipsparseDdense2csr(handle, numARows, numAColumns, descrA, d_A_dense, lda, d_nnzPerVectorA, d_A, d_A_RowIndices, d_A_ColIndices));
             

        mxGPUDestroyGPUArray(PerVect1);
        
   double const *d_B_dense;
   d_B_dense = (double const *)(mxGPUGetDataReadOnly(INPUTDENSEGPUB));   
             mxGPUDestroyGPUArray(INPUTDENSEGPUB);
   
   size_t pivot_dimensionsvalue[1] = {numARows};

    mxGPUArray *VAL = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionsvalue, mxDOUBLE_CLASS, mxREAL, MX_GPU_INITIALIZE_VALUES);
    double  *VALOUT = (double *)mxGPUGetData(VAL);

 cusparseSafeCall(hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
               numARows, numAColumns, nnzA, &alpha, 
               descrA, 
               d_A, 
               d_A_RowIndices, d_A_ColIndices,
               d_B_dense, &beta, 
               VALOUT));
          mxGPUDestroyGPUArray(A);  
          mxGPUDestroyGPUArray(ROW_A); 
          mxGPUDestroyGPUArray(COL_A);            
               
  OUTPUTMATRIX = mxGPUCreateMxArrayOnGPU(VAL);             

       
        mxGPUDestroyGPUArray(VAL);
  
        hipsparseDestroyMatDescr(descrA); 
		hipsparseDestroy(handle);
        
        }
    
        else{
        mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Incorrect input arguments! %s\n");    
        }
    
   }
     
////////////////////////////////////////////////////////////////////////////////////  
    else if (!(mxIsGPUArray(INPUTDENSEA)) && !(mxIsGPUArray(INPUTDENSEB))){
   
     // if ((mxGetClassID(INPUTSPARSEA) != mxDOUBLE_CLASS) || (mxGetClassID(INPUTSPARSEB) != mxDOUBLE_CLASS)) {
       //  mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
           //     "Invalid input to MEX file, input(FIRST and SECOND  ARGUMENTS) must be  double precision.");
             
   // }
    if((!mxIsSparse(INPUTDENSEA))&& (!mxIsSparse(INPUTDENSEB)) ){
    
     mxInitGPU();
    const mwSize *dimsCPUA;
    dimsCPUA=mxGetDimensions(INPUTDENSEA);
    
    int  numARows = (int)dimsCPUA[0]; /* gets number of rows of A */
    int  numAColumns = (int)dimsCPUA[1]; /* gets number of columns of A */
   
    const mwSize *dimsCPUB;
    dimsCPUB=mxGetDimensions(INPUTDENSEB);
    
    int  numBRows = (int)dimsCPUB[0]; /* gets number of rows of B */
    int  numBColumns = (int)dimsCPUB[1]; /* gets number of columns of B */
    
    if ( (numBColumns!= 1)) {   
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                        "Invalid input to MEX file, second argument must be a dense column vector.");
             
    }
    
    if ( (numAColumns!= numBRows) ) {
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Invalid input to MEX file, column number of dense matrix(first argument) must be equal to row numbers of dense vector(second argument).");
             
    }
      const  double alpha= mxGetScalar(ALPHA);
      const  double beta = 0.0;
    
   double *h_A_dense1;
   h_A_dense1 = (double *)mxGetDoubles(INPUTDENSEA);

      
	hipsparseHandle_t handle;    cusparseSafeCall(hipsparseCreate(&handle));

	hipsparseMatDescr_t descrA;      cusparseSafeCall(hipsparseCreateMatDescr(&descrA));
	hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE);

	int nnzA = 0;                            // --- Number of nonzero elements in dense matrix A
	const int lda = numARows;
	//int *d_nnzPerVectorA;    gpuErrchk(hipMalloc(&d_nnzPerVectorA, numARows * sizeof(*d_nnzPerVectorA)));
	size_t pivot_pervect1[1] = {numARows};
    mxGPUArray *PerVect1 = mxGPUCreateGPUArray(1, (mwSize*) pivot_pervect1, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
	int *d_nnzPerVectorA = (int*)mxGPUGetData(PerVect1);
	
	
	//double *d_A_dense;  gpuErrchk(hipMalloc(&d_A_dense, numARows * numAColumns * sizeof(*d_A_dense)));
	
	 size_t pivot_dimensionsvalueDA[2] = {numARows, numAColumns};
      mxGPUArray *OUTMA = mxGPUCreateGPUArray(2, (mwSize*) pivot_dimensionsvalueDA, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
      double *d_A_dense = (double *)mxGPUGetData(OUTMA);
	  
	  
	gpuErrchk(hipMemcpy(d_A_dense, h_A_dense1, numARows * numAColumns * sizeof(*d_A_dense), hipMemcpyHostToDevice));
	cusparseSafeCall(hipsparseDnnz(handle, HIPSPARSE_DIRECTION_ROW, numARows, numAColumns, descrA, d_A_dense, lda, d_nnzPerVectorA, &nnzA));
        
    	//double *d_A;            gpuErrchk(hipMalloc(&d_A, nnzA * sizeof(*d_A)));
		//int *d_A_RowIndices;    gpuErrchk(hipMalloc(&d_A_RowIndices, (numARows + 1) * sizeof(*d_A_RowIndices)));
		//int *d_A_ColIndices;    gpuErrchk(hipMalloc(&d_A_ColIndices, nnzA * sizeof(*d_A_ColIndices)));
   size_t pivot_dimensA[1] = {nnzA};
   size_t pivot_dimensROW_A[1] = {numARows+1};
   size_t pivot_dimensCOL_A[1] = {nnzA};
   
   
   mxGPUArray *A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensA, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    double  *d_A = (double *)mxGPUGetData(A);
   mxGPUArray * ROW_A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensROW_A, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_A_RowIndices = (int *)mxGPUGetData(ROW_A);
   mxGPUArray * COL_A = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensCOL_A, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int  *d_A_ColIndices = (int *)mxGPUGetData(COL_A);

		cusparseSafeCall(hipsparseDdense2csr(handle, numARows, numAColumns, descrA, d_A_dense, lda, d_nnzPerVectorA, d_A, d_A_RowIndices, d_A_ColIndices));
		       
		mxGPUDestroyGPUArray(OUTMA);
		mxGPUDestroyGPUArray(PerVect1);
       
        
   double *h_B_dense1;
   h_B_dense1 = (double *)mxGetDoubles(INPUTDENSEB);    
        
 
	  size_t pivot_dimensionsvalueDB[1] = {numBRows};
      mxGPUArray *OUTMB = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionsvalueDB, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
      double *d_B_dense = (double *)mxGPUGetData(OUTMB);
     gpuErrchk(hipMemcpy(d_B_dense, h_B_dense1, numBRows * sizeof(*d_B_dense), hipMemcpyHostToDevice));
     	
   size_t pivot_dimensionsvalue[1] = {numARows};

    mxGPUArray *VAL = mxGPUCreateGPUArray(1, (mwSize*) pivot_dimensionsvalue, mxDOUBLE_CLASS, mxREAL, MX_GPU_INITIALIZE_VALUES);
    double  *VALOUT = (double *)mxGPUGetData(VAL);

 cusparseSafeCall(hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
               numARows, numAColumns, nnzA, &alpha, 
               descrA, 
               d_A, 
               d_A_RowIndices, d_A_ColIndices,
               d_B_dense, &beta, 
               VALOUT));
          mxGPUDestroyGPUArray(A);  
          mxGPUDestroyGPUArray(ROW_A); 
          mxGPUDestroyGPUArray(COL_A);            
          mxGPUDestroyGPUArray(OUTMB);
          
  OUTPUTMATRIX = mxGPUCreateMxArrayOnGPU(VAL);             

       
        mxGPUDestroyGPUArray(VAL);
  
        hipsparseDestroyMatDescr(descrA);  

		hipsparseDestroy(handle);
    }
    else{
        mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Incorrect input arguments! %s\n");    
        }
    
 }
        //
    else{
        mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Incorrect input arguments! %s\n");    
        }

}
