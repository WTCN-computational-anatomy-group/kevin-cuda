#include "hip/hip_runtime.h"
/*
 * Inverse Discrete Sine Transform in row wise (DST four)
 * DST_IV_Row_Inverse
 * This CUDA code can handle/work with  any type of the input mxArrays, 
 * GPUarray or standard matlab CPU array as input {prhs[0] := mxGPUArray or CPU Array}
 * gpuArray output, B=DST_IV_Row_Inverse(A)=mexFunction(A).
 * Developed at UCL, Institute of Neurology, 12 Queen Square, WC1N 3AR, London
 * Wellcome Trust Centre for Neuroimaging
 * Part of the project SPM(http://www.fil.ion.ucl.ac.uk/spm)
 * Copyright 2018
 * Kevin Bronik
 */
#include "matrix.h"
#include "DST_IV_Row_Inverse.cuh"
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "ERRORCHK.h"
// #define TILE_DIM 16

#define DEFAULT_DIM 32                     // Tile dimension 
#define 	DELTA(i, j)   ((i==j)?1:0)

//const float  PI_d = 3.141592653589793238462643383279502884f; //pi



template <unsigned int TILE_DIM >  __global__ void DSTIV_Row__InverseKernelx(float  *A, float  *C,
	int numARows, int numAColumns,
	int numCRows, int numCColumns)
{
	float CValue = 0.0f;
	const float  PI_d = 3.141592653589793238462643383279502884f; //pi
	int Row = blockIdx.y*TILE_DIM + threadIdx.y;
	int Col = blockIdx.x*TILE_DIM + threadIdx.x;

	__shared__ float As[TILE_DIM][TILE_DIM];
	__shared__ float Bs[TILE_DIM][TILE_DIM];

	for (int k = 0; k < (TILE_DIM + numAColumns - 1) / TILE_DIM; k++) {

		if (k*TILE_DIM + threadIdx.x < numAColumns && Row < numARows)	{ As[threadIdx.y][threadIdx.x] = A[Row*numAColumns + k*TILE_DIM + threadIdx.x]; }
		else													{ As[threadIdx.y][threadIdx.x] = 0.0; }
                                                                           //Bs[threadIdx.y][threadIdx.x] = __cosf(((2 * (threadIdx.y + k*TILE_DIM) + 1)*PI_d*(2 * Col + 1) / (4.0 * numAColumns)))*sqrtf(2.0 / numAColumns);
		if (k*TILE_DIM + threadIdx.y < numAColumns && Col < numAColumns)	{ Bs[threadIdx.y][threadIdx.x] = __sinf((((threadIdx.y + k*TILE_DIM) + 0.5)*PI_d*(Col + 0.5)) / (numAColumns))*sqrtf(2.0 / (numAColumns)); }
		
		else													{ Bs[threadIdx.y][threadIdx.x] = 0.0; }

		__syncthreads();

		for (int n = 0; n < TILE_DIM; ++n) { CValue += As[threadIdx.y][n] * Bs[n][threadIdx.x]; }

		__syncthreads();
	}

	if (Row < numCRows && Col < numCColumns) { C[((blockIdx.y * blockDim.y + threadIdx.y)*numCColumns) + (blockIdx.x*blockDim.x) + threadIdx.x] = CValue; }

}


// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
extern "C" void  CalculateTransformDSTInverseRowFourS(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns)
{


	float * hostA = A; // The A matrix
	//float * hostB = B; // The B matrix
	float * hostC = C; // The output C matrix
	//float * hostComputedC;
	float * deviceA;
	//float * deviceB;
	float * deviceC;

	//hostA = (float *)malloc(sizeof(float)*numARows*numAColumns);
	hipError_t error;
    int devID = 0;
    // get number of SMs on this GPU
    error = hipGetDevice(&devID);
    hipDeviceProp_t deviceProp;
     error = hipGetDeviceProperties(&deviceProp, devID);
      if (error != hipSuccess)
      {
          printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
          exit(EXIT_FAILURE);
      }
    int TILEDIM = (deviceProp.major < 2) ? 16 : 32;
	
	


	// Setting numCRows and numCColumns
	numCRows = numARows;
	numCColumns = numAColumns;

	//hostC = (float *)malloc(sizeof(float)*numCRows*numCColumns);
	//hostComputedC = (float *)malloc(sizeof(float)*numCRows*numCColumns);

	// Allocating GPU memory
	gpuErrchk(hipMalloc((void **)&deviceA, sizeof(float)*numARows*numAColumns));
	//hipMalloc((void **)&deviceB, sizeof(float)*numBRows*numBColumns);
	gpuErrchk(hipMalloc((void **)&deviceC, sizeof(float)*numCRows*numCColumns));
	
	//thrust::device_ptr< float >dev_ptr_A(deviceA);
	//thrust::device_ptr< float >dev_ptr_C(deviceC);

	// Copy memory to the GPU
	gpuErrchk(hipMemcpy(deviceA, hostA, sizeof(float)*numARows*numAColumns, hipMemcpyHostToDevice));
	//hipMemcpy(deviceB, hostB, sizeof(float)*numBRows*numBColumns, hipMemcpyHostToDevice);

	/////////////////////////////////////////////////////////
    unsigned int TILE_DIM=16;
    dim3 dimBlock;
	dim3 dimGrid;
  switch (TILEDIM){
        
        case 16:
     TILE_DIM= TILEDIM;
	 dimBlock.x=TILE_DIM;
	 dimBlock.y=TILE_DIM;
     dimBlock.z=1;
	dimGrid.x = (numCColumns + dimBlock.x - 1) / dimBlock.x;
	dimGrid.y = (numCRows + dimBlock.y - 1) / dimBlock.y;
	DSTIV_Row__InverseKernelx <16> << <dimGrid, dimBlock >> >(deviceA, deviceC, numARows, numAColumns, numCRows, numCColumns);
	//matrixMultiplyShared << <dimGrid, dimBlock >> >(thrust::raw_pointer_cast(&dev_ptr_A[0]), thrust::raw_pointer_cast(&dev_ptr_C[0]), numARows, numAColumns, numCRows, numCColumns);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());

	// Copy the results in GPU memory back to the CPU
	gpuErrchk(hipMemcpy(hostC, deviceC, sizeof(float)*numCRows*numCColumns, hipMemcpyDeviceToHost));

	C = hostC;

	//thrust::device_free(dev_ptr_A);
	//thrust::device_free(dev_ptr_C);
	gpuErrchk(hipFree(deviceA));
	//hipFree(deviceB);
	gpuErrchk(hipFree(deviceC));
	return;
	
	     case 32:
     TILE_DIM= TILEDIM;
	 dimBlock.x=TILE_DIM;
	 dimBlock.y=TILE_DIM;
     dimBlock.z=1;
	

	dimGrid.x = (numCColumns + dimBlock.x - 1) / dimBlock.x;
	dimGrid.y = (numCRows + dimBlock.y - 1) / dimBlock.y;
		DSTIV_Row__InverseKernelx <32> << <dimGrid, dimBlock >> >(deviceA, deviceC, numARows, numAColumns, numCRows, numCColumns);
	//matrixMultiplyShared << <dimGrid, dimBlock >> >(thrust::raw_pointer_cast(&dev_ptr_A[0]), thrust::raw_pointer_cast(&dev_ptr_C[0]), numARows, numAColumns, numCRows, numCColumns);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());

	// Copy the results in GPU memory back to the CPU
	gpuErrchk(hipMemcpy(hostC, deviceC, sizeof(float)*numCRows*numCColumns, hipMemcpyDeviceToHost));

	C = hostC;

	//thrust::device_free(dev_ptr_A);
	//thrust::device_free(dev_ptr_C);
	gpuErrchk(hipFree(deviceA));
	//hipFree(deviceB);
	gpuErrchk(hipFree(deviceC));
	return;
	
	
	}
	
    
}
