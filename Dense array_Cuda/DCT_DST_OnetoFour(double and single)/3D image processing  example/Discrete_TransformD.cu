
/*
 * Discrete Cosine/Sine Transform(DCT/DST and IDCT/IDST one to four-all in one)
 * DCT/DST and IDCT/IDST I ---> IV
 * This CUDA code can handle/work with  any type of the input mxArrays, 
 * GPUarray or standard matlab CPU array as input {prhs[0] := mxGPUArray or CPU Array}
 * GpuArray/cpuArray output, B=Discrete_Transform(A, , type of Transform (sine or cosine), type of Transform(direct/inverse), type of DCT/DST or IDCT/IDST, dimensions).
 * Developed at UCL, Institute of Neurology, 12 Queen Square, WC1N 3AR, London
 * Wellcome Trust Centre for Neuroimaging
 * Part of the project SPM(http://www.fil.ion.ucl.ac.uk/spm)
 * Copyright 2018
 * Kevin Bronik
 */

#include "matrix.h"
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include "CuFilesD/Discrete_Transform_kernel.cuh"
#include "CuFilesD/DCT_I_Column.cu"
#include "CuFilesD/DCT_I_Row.cu"
#include "CuFilesD/DCT_I_Column_Inverse.cu"
#include "CuFilesD/DCT_I_Row_Inverse.cu"
#include "CuFilesD/DCT_II_Row.cu"
#include "CuFilesD/DCT_II_Row_Inverse.cu"
#include "CuFilesD/DCT_II_Column.cu"
#include "CuFilesD/DCT_II_Column_Inverse.cu"
#include "CuFilesD/DCT_III_Row.cu"
#include "CuFilesD/DCT_III_Row_Inverse.cu"
#include "CuFilesD/DCT_III_Column.cu"
#include "CuFilesD/DCT_III_Column_Inverse.cu"
#include "CuFilesD/DCT_IV_Row.cu"
#include "CuFilesD/DCT_IV_Row_Inverse.cu"
#include "CuFilesD/DCT_IV_Column.cu"
#include "CuFilesD/DCT_IV_Column_Inverse.cu"
#include "CuFilesD/DST_I_Column.cu"
#include "CuFilesD/DST_I_Row.cu"
#include "CuFilesD/DST_I_Column_Inverse.cu"
#include "CuFilesD/DST_I_Row_Inverse.cu"
#include "CuFilesD/DST_II_Row.cu"
#include "CuFilesD/DST_II_Row_Inverse.cu"
#include "CuFilesD/DST_II_Column.cu"
#include "CuFilesD/DST_II_Column_Inverse.cu"
#include "CuFilesD/DST_III_Row.cu"
#include "CuFilesD/DST_III_Row_Inverse.cu"
#include "CuFilesD/DST_III_Column.cu"
#include "CuFilesD/DST_III_Column_Inverse.cu"
#include "CuFilesD/DST_IV_Row.cu"
#include "CuFilesD/DST_IV_Row_Inverse.cu"
#include "CuFilesD/DST_IV_Column.cu"
#include "CuFilesD/DST_IV_Column_Inverse.cu"
//#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#define DEFAULT_DIM 32   
#define 	DELTA(i, j)   ((i==j)?1:0)
//#define TILE_DIM 16
unsigned int TILE_DIM=16;
// DCT
extern "C" void  CalculateTransformDCTColumnOne(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDCTInverseColumnOne(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDCTRowOne(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDCTInverseRowOne(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);    
    
extern "C" void  CalculateTransformDCTRowTwo(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDCTInverseRowTwo(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDCTColumnTwo(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDCTInverseColumnTwo(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDCTColumnThree(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns); 
    
extern "C" void  CalculateTransformDCTInverseColumnThree(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDCTRowThree(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDCTInverseRowThree(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDCTColumnFour(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);  
    
extern "C" void  CalculateTransformDCTInverseColumnFour(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDCTRowFour(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDCTInverseRowFour(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns); 
    
    // DST
extern "C" void  CalculateTransformDSTColumnOne(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDSTInverseColumnOne(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDSTRowOne(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDSTInverseRowOne(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);    
    
extern "C" void  CalculateTransformDSTRowTwo(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDSTInverseRowTwo(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDSTColumnTwo(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDSTInverseColumnTwo(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDSTColumnThree(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns); 
    
extern "C" void  CalculateTransformDSTInverseColumnThree(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDSTRowThree(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDSTInverseRowThree(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDSTColumnFour(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);  
    
extern "C" void  CalculateTransformDSTInverseColumnFour(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDSTRowFour(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns);
    
extern "C" void  CalculateTransformDSTInverseRowFour(float * A, float * C, int numARows,
	int numAColumns, int numCRows, int numCColumns); 
    
extern "C" static void mexTransD(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
 int nDevices;
hipError_t errCode =hipGetDeviceCount(&nDevices); 
//int nDevices;
//hipGetDeviceCount(&nDevices);

if (errCode != hipSuccess){
printf("Error! No CUDA devices found! \n");
return;
}

char row[] = "row";
char column[] = "column";
char one[] = "one";
char two[] = "two";
char three[] = "three";
char four[] = "four";

char direct[] = "direct";
char inverse[] = "inverse";
char cosine[] = "cosine";
char sine[] = "sine";
 
    char const * const InputErrMsg = "Invalid input to MEX file, input(FIRST ARGUMENT) must be single precision (float), and the number of input arguments must be five.";
    
    if ((nrhs!=5)) {
        mexErrMsgIdAndTxt("MATLAB:mexatexit:invalidInput", InputErrMsg);
    }

char *input_buf0;
 input_buf0 = mxArrayToString(prhs[0]);
 char *input_buf1;
 input_buf1 = mxArrayToString(prhs[1]);
char *input_buf2;
 input_buf2 = mxArrayToString(prhs[2]);
  char *input_buf3;
 input_buf3 = mxArrayToString(prhs[3]);
char *input_buf4;
 input_buf4 = mxArrayToString(prhs[4]);
    if ((mxIsChar(prhs[0]))){
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Input(FIRST ARGUMENT) must be array, or gpuArray object not  %s\n",input_buf0);
    }
     if (!(mxIsChar(prhs[1]))){
        mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Input(SECOND ARGUMENT) must be of type string.\n.");
    }
      if (!(mxIsChar(prhs[2]))){
        mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Input(THIRD ARGUMENT) must be of type string.\n.");
    }
        if (!(mxIsChar(prhs[3]))){
        mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Input(FOURTH ARGUMENT) must be of type string.\n.");
    }
      if (!(mxIsChar(prhs[4]))){
        mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Input(FIFTH ARGUMENT) must be of type string.\n.");
    } 
  
  if ((strcmp (cosine,input_buf1) != 0) &&(strcmp (sine,input_buf1) != 0) )
{
    mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Input(SECOND ARGUMENT) must be 'cosine' or 'sine'  not  %s\n",input_buf1);
   }
    
  if ((strcmp (direct,input_buf2) != 0)&& (strcmp (inverse,input_buf2) != 0) )
{
        mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Input(THIRD ARGUMENT) must be 'direct' or 'inverse' not  %s\n",input_buf2);
    }
    

   if ((strcmp (one,input_buf3) != 0)&& (strcmp (two,input_buf3) != 0) && (strcmp (three,input_buf3) != 0) && (strcmp (four,input_buf3) != 0))
{
                mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Input(FOURTH ARGUMENT) must be 'one' or 'two' or 'three' or 'four' not  %s\n",input_buf3);

    }
       if ((strcmp (column,input_buf4) != 0)&&(strcmp (row,input_buf4) != 0))
{  
            mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Input(FIFTH ARGUMENT) must be 'column' or 'row' not  %s\n",input_buf4);
    }
 //COSINE TRANSFORM   
 if (strcmp (cosine,input_buf1) == 0)
{

 if (strcmp (direct,input_buf2) == 0)
{   
  if (strcmp (column,input_buf4) == 0)
{   
 
    if (mxIsGPUArray(prhs[0])) {
    
  

    mxGPUArray const *A;
    mxGPUArray *B;
    float const *d_A;
    float *d_B;
    int numARows, numAColumns, numCRows,  numCColumns;
    mxInitGPU();
    hipError_t error;
    int devID = 0;
    error = hipGetDevice(&devID);
    hipDeviceProp_t deviceProp;
     error = hipGetDeviceProperties(&deviceProp, devID);
      if (error != hipSuccess)
      {
          printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
          exit(EXIT_FAILURE);
      }
    int TILEDIM = (deviceProp.major < 2) ? 16 : 32;

   A = mxGPUCreateFromMxArray(prhs[0]);
   if(mxGPUGetComplexity(A) != mxREAL){
       mxGPUDestroyGPUArray(A);
       mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Incorrect input arguments!, input matrix must be real %s\n");
       
       }
   
   const mwSize *dims;
   dims=mxGPUGetDimensions(A);
   numARows = (int)dims[0]; /* gets number of rows of A */
   numAColumns = (int)dims[1]; /* gets number of columns of A */
   size_t pivot_dimensA[2] = {numARows,numAColumns};
   mwSize NrOfDim=mxGPUGetNumberOfDimensions(A);
   
		numCRows = numARows;
		numCColumns = numAColumns;
  if (numARows==1)
 {   
 printf("Attention, this is a row vector, please try Discrete Cosine Transform in row wise \n");
 return;
 }
 
    char const * const errId = "parallel:gpu:mexGPUExample:InvalidInput";
    char const * const errMsg = "Invalid input to MEX file, input(FIRST ARGUMENT) must be single precision (float).";

    if (mxGPUGetClassID(A) != mxSINGLE_CLASS) {
        mexErrMsgIdAndTxt(errId, errMsg);
    }

    d_A = (float const *)(mxGPUGetDataReadOnly(A));
    mxGPUDestroyGPUArray(A);
    B = mxGPUCreateGPUArray(NrOfDim, (mwSize*) pivot_dimensA, mxSINGLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    d_B = (float *)(mxGPUGetData(B));
    dim3 dimBlock;
	dim3 dimGrid;
    
     switch (TILEDIM){
        
        case 16:
            
     TILE_DIM= TILEDIM;
	 dimBlock.x=TILE_DIM;
	 dimBlock.y=TILE_DIM;
     dimBlock.z=1;
	dimGrid.x = (numCColumns + dimBlock.x - 1) / dimBlock.x;
	dimGrid.y = (numCRows + dimBlock.y - 1) / dimBlock.y;
 if (strcmp (one,input_buf3) == 0)
{
     
   DCTI_Column_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
 }  
 if (strcmp (two,input_buf3) == 0)
{
     
   DCTII_Column_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
 }
 if (strcmp (three,input_buf3) == 0)
{
     
   DCTIII_Column_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
 }
 if (strcmp (four,input_buf3) == 0)
{
     
   DCTIV_Column_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
 }

    plhs[0] = mxGPUCreateMxArrayOnGPU(B);   
    mxGPUDestroyGPUArray(B);
    
         case 32:
            
     TILE_DIM= TILEDIM;
	 dimBlock.x=TILE_DIM;
	 dimBlock.y=TILE_DIM;
     dimBlock.z=1;
	dimGrid.x = (numCColumns + dimBlock.x - 1) / dimBlock.x;
	dimGrid.y = (numCRows + dimBlock.y - 1) / dimBlock.y;
    
     if (strcmp (one,input_buf3) == 0)
{
     
   DCTI_Column_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
 }  
 if (strcmp (two,input_buf3) == 0)
{
     
   DCTII_Column_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
 }
 if (strcmp (three,input_buf3) == 0)
{
     
   DCTIII_Column_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
 }
 if (strcmp (four,input_buf3) == 0)
{
     
   DCTIV_Column_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
 }

    plhs[0] = mxGPUCreateMxArrayOnGPU(B);  
    mxGPUDestroyGPUArray(B);
    
      } 
    }

    else if (!(mxIsGPUArray(prhs[0]))){
             
   if (mxGetClassID(prhs[0]) != mxSINGLE_CLASS) {
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Invalid input to MEX file, input(FIRST ARGUMENT) must be single precision (float).");
    } 
   if(mxIsComplex(prhs[0])){
       
       mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Incorrect input arguments!, input matrix must be real %s\n");
       
       }
    
  	int numARows = (int)mxGetM(prhs[0]); 		// number of rows in the matrix A
	int numAColumns = (int)mxGetN(prhs[0]); 	// number of columns in the matrix A
	int numCRows;		// number of rows in the matrix C (you have to set this)
	int numCColumns;	// number of columns in the matrix C (you have to set this)
    	
		numCRows = numARows;
		numCColumns = numAColumns;
        
 if (numARows==1)
 {   
 printf("Attention, this is a row vector, please try Discrete Cosine Transform in row wise \n");
 return;
 }
    mxInitGPU();
	float  * hostA ; // The A matrix
	hostA = (float *)mxGetData(prhs[0]);   
    plhs[0] = mxCreateNumericMatrix(numCRows, numCColumns, mxSINGLE_CLASS, mxREAL);
    float  *pointer =(float*) mxGetPr(plhs[0]);
  
     if (strcmp (one,input_buf3) == 0)
    {
     
       CalculateTransformDCTColumnOne(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
	
    }
     if (strcmp (two,input_buf3) == 0)
    {
     
       CalculateTransformDCTColumnTwo(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
	
    }
     if (strcmp (three,input_buf3) == 0)
    {
     
       CalculateTransformDCTColumnThree(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
	
    }
     if (strcmp (four,input_buf3) == 0)
    {
     
       CalculateTransformDCTColumnFour(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
	
         }	
       } 
    } 
   
     if (strcmp (row,input_buf4) == 0)
{

    if (mxIsGPUArray(prhs[0])) {
		
    mxGPUArray const *A;
    mxGPUArray *B;
    float const *d_A;
    float *d_B;
    int numARows, numAColumns, numCRows,  numCColumns;
    mxInitGPU();
    hipError_t error;
    int devID = 0;
    error = hipGetDevice(&devID);
    hipDeviceProp_t deviceProp;
     error = hipGetDeviceProperties(&deviceProp, devID);
      if (error != hipSuccess)
      {
          printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
          exit(EXIT_FAILURE);
      }
    int TILEDIM = (deviceProp.major < 2) ? 16 : 32;
    A = mxGPUCreateFromMxArray(prhs[0]);
       if(mxGPUGetComplexity(A) != mxREAL){
       mxGPUDestroyGPUArray(A);
       mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Incorrect input arguments!, input matrix must be real %s\n");
       
       }
    const mwSize *dims;
    dims=mxGPUGetDimensions(A);
    numARows = (int)dims[0]; /* gets number of rows of A */
    numAColumns = (int)dims[1]; /* gets number of columns of A */
   size_t pivot_dimensA[2] = {numARows,numAColumns};
   mwSize NrOfDim=mxGPUGetNumberOfDimensions(A); 
 
 if (numAColumns==1)
 {   
 printf("Attention, this is a column vector, please try Discrete Cosine Transform in column wise \n");
 return;
 }
 
    numCRows = numARows;
	numCColumns = numAColumns;

   char const * const errId = "parallel:gpu:mexGPUExample:InvalidInput";
   char const * const errMsg = "Invalid input to MEX file, input(FIRST ARGUMENT) must be single precision (float).";
    if (mxGPUGetClassID(A) != mxSINGLE_CLASS) {
        mexErrMsgIdAndTxt(errId, errMsg);
    }
    
    d_A = (float const *)(mxGPUGetDataReadOnly(A));
    mxGPUDestroyGPUArray(A);
    B = mxGPUCreateGPUArray(NrOfDim, (mwSize*) pivot_dimensA, mxSINGLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    d_B = (float *)(mxGPUGetData(B));
    dim3 dimBlock;
	dim3 dimGrid;
    
     switch (TILEDIM){
        
        case 16:
            
     TILE_DIM= TILEDIM;
	 dimBlock.x=TILE_DIM;
	 dimBlock.y=TILE_DIM;
     dimBlock.z=1;
	dimGrid.x = (numCColumns + dimBlock.x - 1) / dimBlock.x;
	dimGrid.y = (numCRows + dimBlock.y - 1) / dimBlock.y;
 if (strcmp (one,input_buf3) == 0)
{
    
   DCTI_Row_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
}
 if (strcmp (two,input_buf3) == 0)
{
    
   DCTII_Row_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
}
 if (strcmp (three,input_buf3) == 0)
{
    
   DCTIII_Row_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
}
 if (strcmp (four,input_buf3) == 0)
{
    
   DCTIV_Row_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
}
    
    plhs[0] = mxGPUCreateMxArrayOnGPU(B); 
    mxGPUDestroyGPUArray(B);
    
         case 32:
            
     TILE_DIM= TILEDIM;
	 dimBlock.x=TILE_DIM;
	 dimBlock.y=TILE_DIM;
     dimBlock.z=1;
	dimGrid.x = (numCColumns + dimBlock.x - 1) / dimBlock.x;
	dimGrid.y = (numCRows + dimBlock.y - 1) / dimBlock.y;
    
     if (strcmp (one,input_buf3) == 0)
{
    
   DCTI_Row_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
}
 if (strcmp (two,input_buf3) == 0)
{
    
   DCTII_Row_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
}
 if (strcmp (three,input_buf3) == 0)
{
    
   DCTIII_Row_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
}
 if (strcmp (four,input_buf3) == 0)
{
    
   DCTIV_Row_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
}

    plhs[0] = mxGPUCreateMxArrayOnGPU(B);
    mxGPUDestroyGPUArray(B);
     
        }    
	}

    else if (!(mxIsGPUArray(prhs[0]))){
            
   if (mxGetClassID(prhs[0]) != mxSINGLE_CLASS) {
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Invalid input to MEX file, input(FIRST ARGUMENT) must be single precision (float).");
    }  
   if(mxIsComplex(prhs[0])){
       
       mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Incorrect input arguments!, input matrix must be real %s\n");
       
       }
  	int numARows = (int)mxGetM(prhs[0]); 		// number of rows in the matrix A
	int numAColumns = (int)mxGetN(prhs[0]); 	// number of columns in the matrix A
	
	int numCRows;		// number of rows in the matrix C (you have to set this)
	int numCColumns;	// number of columns in the matrix C (you have to set this)
	
	numCRows = numARows;
	numCColumns = numAColumns;
	float  * hostA ; // The A matrix
	
 if (numAColumns==1)
 {   
 printf("Attention, this is a column vector, please try Discrete Cosine Transform in column wise \n");
 return;
 }

    mxInitGPU();
	hostA = (float *)mxGetData(prhs[0]);
    plhs[0] = mxCreateNumericMatrix(numCRows, numCColumns, mxSINGLE_CLASS, mxREAL);
    float  *pointer = (float*)mxGetPr(plhs[0]);
     if (strcmp (one,input_buf3) == 0)
{  
      CalculateTransformDCTRowOne(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
      
      } 
       if (strcmp (two,input_buf3) == 0)
{  
      CalculateTransformDCTRowTwo(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
      
      }
       if (strcmp (three,input_buf3) == 0)
{  
      CalculateTransformDCTRowThree(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
      
      }
       if (strcmp (four,input_buf3) == 0)
{  
      CalculateTransformDCTRowFour(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
      
        }
  
      } 

   }

}

 if (strcmp (inverse,input_buf2) == 0)
{
    if (strcmp (column,input_buf4) == 0)
{      
 
    if (mxIsGPUArray(prhs[0])) {

    mxGPUArray const *A;
    mxGPUArray *B;
    float const *d_A;
    float *d_B;
    int numARows, numAColumns, numCRows,  numCColumns;
    mxInitGPU();
    hipError_t error;
    int devID = 0;
    error = hipGetDevice(&devID);
    hipDeviceProp_t deviceProp;
     error = hipGetDeviceProperties(&deviceProp, devID);
      if (error != hipSuccess)
      {
          printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
          exit(EXIT_FAILURE);
      }
    int TILEDIM = (deviceProp.major < 2) ? 16 : 32;
    A = mxGPUCreateFromMxArray(prhs[0]);
       if(mxGPUGetComplexity(A) != mxREAL){
       mxGPUDestroyGPUArray(A);
       mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Incorrect input arguments!, input matrix must be real %s\n");
       
       }
    const mwSize *dims;
   dims=mxGPUGetDimensions(A);
   numARows = (int)dims[0]; /* gets number of rows of A */
   numAColumns = (int)dims[1]; /* gets number of columns of A */
   size_t pivot_dimensA[2] = {numARows,numAColumns};
   mwSize NrOfDim=mxGPUGetNumberOfDimensions(A);
   
		numCRows = numARows;
		numCColumns = numAColumns;
  if (numARows==1)
 {   
 printf("Attention, this is a row vector, please try Inverse Discrete Cosine Transform in row wise \n");
 return;
 }
 
    char const * const errId = "parallel:gpu:mexGPUExample:InvalidInput";
    char const * const errMsg = "Invalid input to MEX file, input(FIRST ARGUMENT) must be single precision (float).";

    if (mxGPUGetClassID(A) != mxSINGLE_CLASS) {
        mexErrMsgIdAndTxt(errId, errMsg);
    }

    d_A = (float const *)(mxGPUGetDataReadOnly(A));
    mxGPUDestroyGPUArray(A);
    B = mxGPUCreateGPUArray(NrOfDim, (mwSize*) pivot_dimensA, mxSINGLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    d_B = (float *)(mxGPUGetData(B));
    dim3 dimBlock;
	dim3 dimGrid;
    
     switch (TILEDIM){
        
        case 16:
            
     TILE_DIM= TILEDIM;
	 dimBlock.x=TILE_DIM;
	 dimBlock.y=TILE_DIM;
     dimBlock.z=1;
	dimGrid.x = (numCColumns + dimBlock.x - 1) / dimBlock.x;
	dimGrid.y = (numCRows + dimBlock.y - 1) / dimBlock.y;
 if (strcmp (one,input_buf3) == 0)
{
    
   DCTI_Column_Inverse_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
}
  if (strcmp (two,input_buf3) == 0)
{
    
   DCTII_Column_Inverse_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
}
 if (strcmp (three,input_buf3) == 0)
{
    
   DCTIII_Column_Inverse_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
}
 if (strcmp (four,input_buf3) == 0)
{
    
   DCTIV_Column_Inverse_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
}

    plhs[0] = mxGPUCreateMxArrayOnGPU(B);
    mxGPUDestroyGPUArray(B);
     
    case 32:
            
     TILE_DIM= TILEDIM;
	 dimBlock.x=TILE_DIM;
	 dimBlock.y=TILE_DIM;
     dimBlock.z=1;
	dimGrid.x = (numCColumns + dimBlock.x - 1) / dimBlock.x;
	dimGrid.y = (numCRows + dimBlock.y - 1) / dimBlock.y;
     if (strcmp (one,input_buf3) == 0)
{
    
   DCTI_Column_Inverse_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
}
  if (strcmp (two,input_buf3) == 0)
{
    
   DCTII_Column_Inverse_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
}
 if (strcmp (three,input_buf3) == 0)
{
    
   DCTIII_Column_Inverse_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
}
 if (strcmp (four,input_buf3) == 0)
{
    
   DCTIV_Column_Inverse_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
}
 
    plhs[0] = mxGPUCreateMxArrayOnGPU(B);   
    mxGPUDestroyGPUArray(B);
    
       }   
	}

    else if (!(mxIsGPUArray(prhs[0]))){
  
    if (mxGetClassID(prhs[0]) != mxSINGLE_CLASS) {
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Invalid input to MEX file, input(FIRST ARGUMENT) must be single precision (float).");
    }        
    if(mxIsComplex(prhs[0])){
       
       mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Incorrect input arguments!, input matrix must be real %s\n");
       
       }           
  	int numARows = (int)mxGetM(prhs[0]); 		// number of rows in the matrix A
	int numAColumns = (int)mxGetN(prhs[0]); 	// number of columns in the matrix A
	int numCRows;		// number of rows in the matrix C (you have to set this)
	int numCColumns;	// number of columns in the matrix C (you have to set this)
    	
		numCRows = numARows;
		numCColumns = numAColumns;
        
 if (numARows==1)
 {   
 printf("Attention, this is a row vector, please try Inverse Discrete Cosine Transform in row wise \n");
 return;
 }
    mxInitGPU();

	float  * hostA ; // The A matrix
	hostA = (float *)mxGetData(prhs[0]);
    plhs[0] = mxCreateNumericMatrix(numCRows, numCColumns, mxSINGLE_CLASS, mxREAL);
    float  *pointer = (float*)mxGetPr(plhs[0]);
     
     if (strcmp (one,input_buf3) == 0)
   {
  
       CalculateTransformDCTInverseColumnOne(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
	
   }
     if (strcmp (two,input_buf3) == 0)
   {
  
       CalculateTransformDCTInverseColumnTwo(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
	
   }
     if (strcmp (three,input_buf3) == 0)
   {
  
       CalculateTransformDCTInverseColumnThree(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
	
   }
     if (strcmp (four,input_buf3) == 0)
   {
  
       CalculateTransformDCTInverseColumnFour(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
	
      }	
   } 
}

    if (strcmp (row,input_buf4) == 0)
{
     
    if (mxIsGPUArray(prhs[0])) {

    mxGPUArray const *A;
    mxGPUArray *B;
    float const *d_A;
   
    float *d_B;
    int numARows, numAColumns, numCRows,  numCColumns;
    mxInitGPU();
    hipError_t error;
    int devID = 0;
    error = hipGetDevice(&devID);
    hipDeviceProp_t deviceProp;
     error = hipGetDeviceProperties(&deviceProp, devID);
      if (error != hipSuccess)
      {
          printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
          exit(EXIT_FAILURE);
      }
    int TILEDIM = (deviceProp.major < 2) ? 16 : 32;
    
    A = mxGPUCreateFromMxArray(prhs[0]);
       if(mxGPUGetComplexity(A) != mxREAL){
       mxGPUDestroyGPUArray(A);
       mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Incorrect input arguments!, input matrix must be real %s\n");
       
       }
    const mwSize *dims;
    dims=mxGPUGetDimensions(A);
    numARows = (int)dims[0]; /* gets number of rows of A */
    numAColumns = (int)dims[1]; /* gets number of columns of A */
    size_t pivot_dimensA[2] = {numARows,numAColumns};
    mwSize NrOfDim=mxGPUGetNumberOfDimensions(A);
    
  if (numAColumns==1)
 {   
 printf("Attention, this is a column vector, please try Inverse Discrete Cosine Transform in column wise \n");
 return;
 }
 
    numCRows = numARows;
	numCColumns = numAColumns;
    char const * const errId = "parallel:gpu:mexGPUExample:InvalidInput";
    char const * const errMsg = "Invalid input to MEX file, input(FIRST ARGUMENT) must be single precision (float).";

    if (mxGPUGetClassID(A) != mxSINGLE_CLASS) {
        mexErrMsgIdAndTxt(errId, errMsg);
    }

    d_A = (float const *)(mxGPUGetDataReadOnly(A));
    mxGPUDestroyGPUArray(A);
    B = mxGPUCreateGPUArray(NrOfDim, (mwSize*) pivot_dimensA, mxSINGLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    d_B = (float *)(mxGPUGetData(B));
    dim3 dimBlock;
	dim3 dimGrid;
    
     switch (TILEDIM){
        
        case 16:
            
     TILE_DIM= TILEDIM;
	 dimBlock.x=TILE_DIM;
	 dimBlock.y=TILE_DIM;
     dimBlock.z=1;
	dimGrid.x = (numCColumns + dimBlock.x - 1) / dimBlock.x;
	dimGrid.y = (numCRows + dimBlock.y - 1) / dimBlock.y;
 if (strcmp (one,input_buf3) == 0)
{
   DCTI_Row__InverseKernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);

   }
 if (strcmp (two,input_buf3) == 0)
{
   DCTII_Row__InverseKernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);

   }
 if (strcmp (three,input_buf3) == 0)
{
   DCTIII_Row__InverseKernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);

   }
 if (strcmp (four,input_buf3) == 0)
{
   DCTIV_Row__InverseKernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);

   }

    plhs[0] = mxGPUCreateMxArrayOnGPU(B);   
    mxGPUDestroyGPUArray(B);
    
        case 32:
            
     TILE_DIM= TILEDIM;
	 dimBlock.x=TILE_DIM;
	 dimBlock.y=TILE_DIM;
     dimBlock.z=1;
	dimGrid.x = (numCColumns + dimBlock.x - 1) / dimBlock.x;
	dimGrid.y = (numCRows + dimBlock.y - 1) / dimBlock.y;
    
     if (strcmp (one,input_buf3) == 0)
{
   DCTI_Row__InverseKernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);

   }
 if (strcmp (two,input_buf3) == 0)
{
   DCTII_Row__InverseKernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);

   }
 if (strcmp (three,input_buf3) == 0)
{
   DCTIII_Row__InverseKernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);

   }
 if (strcmp (four,input_buf3) == 0)
{
   DCTIV_Row__InverseKernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);

   }

    plhs[0] = mxGPUCreateMxArrayOnGPU(B);   
    mxGPUDestroyGPUArray(B);
       
       }    
	}

    else if (!(mxIsGPUArray(prhs[0]))){
            
    if (mxGetClassID(prhs[0]) != mxSINGLE_CLASS) {
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Invalid input to MEX file, input(FIRST ARGUMENT) must be single precision (float).");
    } 
    if(mxIsComplex(prhs[0])){
       
       mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Incorrect input arguments!, input matrix must be real %s\n");
       
       }    
  	int numARows = (int)mxGetM(prhs[0]); 		// number of rows in the matrix A
	int numAColumns = (int)mxGetN(prhs[0]); 	// number of columns in the matrix A
	
	int numCRows;		// number of rows in the matrix C (you have to set this)
	int numCColumns;	// number of columns in the matrix C (you have to set this)
	
    if (numAColumns==1)
    {   
    printf("Attention, this is a column vector, please try Inverse Discrete Cosine Transform in column wise \n");
    return;
    }
    mxInitGPU();
    numCRows = numARows;
	numCColumns = numAColumns;
	float  * hostA ; // The A matrix
	hostA = (float *)mxGetData(prhs[0]);
    plhs[0] = mxCreateNumericMatrix(numCRows, numCColumns, mxSINGLE_CLASS, mxREAL);
    float  *pointer =(float*) mxGetPr(plhs[0]);
 
     if (strcmp (one,input_buf3) == 0)
{
      CalculateTransformDCTInverseRowOne(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);  
 } 
     if (strcmp (two,input_buf3) == 0)
{
      CalculateTransformDCTInverseRowTwo(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);  
 } 
     if (strcmp (three,input_buf3) == 0)
{
      CalculateTransformDCTInverseRowThree(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);  
 } 
     if (strcmp (four,input_buf3) == 0)
{
      CalculateTransformDCTInverseRowFour(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);  
 } 

	
      }
    }
  } 
}


//SINE TRANSFORM
 if (strcmp (sine,input_buf1) == 0)
{

 if (strcmp (direct,input_buf2) == 0)
{   
  if (strcmp (column,input_buf4) == 0)
{   

    if (mxIsGPUArray(prhs[0])) {
	
    mxGPUArray const *A;
    mxGPUArray *B;
    float const *d_A;
    float *d_B;
    int numARows, numAColumns, numCRows,  numCColumns;
    mxInitGPU();
    hipError_t error;
    int devID = 0;
    error = hipGetDevice(&devID);
    hipDeviceProp_t deviceProp;
     error = hipGetDeviceProperties(&deviceProp, devID);
      if (error != hipSuccess)
      {
          printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
          exit(EXIT_FAILURE);
      }
    int TILEDIM = (deviceProp.major < 2) ? 16 : 32;

    A = mxGPUCreateFromMxArray(prhs[0]);
       if(mxGPUGetComplexity(A) != mxREAL){
       mxGPUDestroyGPUArray(A);
       mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Incorrect input arguments!, input matrix must be real %s\n");
       
       }
    const mwSize *dims;
   dims=mxGPUGetDimensions(A);
   numARows = (int)dims[0]; /* gets number of rows of A */
   numAColumns = (int)dims[1]; /* gets number of columns of A */
   size_t pivot_dimensA[2] = {numARows,numAColumns};
   mwSize NrOfDim=mxGPUGetNumberOfDimensions(A);
		numCRows = numARows;
		numCColumns = numAColumns;
        
  if (numARows==1)
 {   
 printf("Attention, this is a row vector, please try Discrete Sine Transform in row wise \n");
 return;
 }
 
    char const * const errId = "parallel:gpu:mexGPUExample:InvalidInput";
    char const * const errMsg = "Invalid input to MEX file, input(FIRST ARGUMENT) must be single precision (float).";

    if (mxGPUGetClassID(A) != mxSINGLE_CLASS) {
        mexErrMsgIdAndTxt(errId, errMsg);
    }

    d_A = (float const *)(mxGPUGetDataReadOnly(A));
    mxGPUDestroyGPUArray(A);
    B = mxGPUCreateGPUArray(NrOfDim, (mwSize*) pivot_dimensA, mxSINGLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    d_B = (float *)(mxGPUGetData(B));

    dim3 dimBlock;
	dim3 dimGrid;
    
     switch (TILEDIM){
        
        case 16:
            
     TILE_DIM= TILEDIM;
	 dimBlock.x=TILE_DIM;
	 dimBlock.y=TILE_DIM;
     dimBlock.z=1;
	dimGrid.x = (numCColumns + dimBlock.x - 1) / dimBlock.x;
	dimGrid.y = (numCRows + dimBlock.y - 1) / dimBlock.y;
 if (strcmp (one,input_buf3) == 0)
{
     
   DSTI_Column_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
 }  
 if (strcmp (two,input_buf3) == 0)
{
     
   DSTII_Column_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
 }
 if (strcmp (three,input_buf3) == 0)
{
     
   DSTIII_Column_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
 }
 if (strcmp (four,input_buf3) == 0)
{
     
   DSTIV_Column_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
 }

    plhs[0] = mxGPUCreateMxArrayOnGPU(B);  
    mxGPUDestroyGPUArray(B);
    
            case 32:
            
     TILE_DIM= TILEDIM;
	 dimBlock.x=TILE_DIM;
	 dimBlock.y=TILE_DIM;
     dimBlock.z=1;
	dimGrid.x = (numCColumns + dimBlock.x - 1) / dimBlock.x;
	dimGrid.y = (numCRows + dimBlock.y - 1) / dimBlock.y;
    
     if (strcmp (one,input_buf3) == 0)
{
     
   DSTI_Column_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
 }  
 if (strcmp (two,input_buf3) == 0)
{
     
   DSTII_Column_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
 }
 if (strcmp (three,input_buf3) == 0)
{
     
   DSTIII_Column_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
 }
 if (strcmp (four,input_buf3) == 0)
{
     
   DSTIV_Column_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
 }

    plhs[0] = mxGPUCreateMxArrayOnGPU(B);  
    mxGPUDestroyGPUArray(B);
     
       }   
	}

    else if (!(mxIsGPUArray(prhs[0]))){
            
    if (mxGetClassID(prhs[0]) != mxSINGLE_CLASS) {
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Invalid input to MEX file, input(FIRST ARGUMENT) must be single precision (float).");
    } 
    if(mxIsComplex(prhs[0])){
       
       mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Incorrect input arguments!, input matrix must be real %s\n");
       
       }
            
  	int numARows = (int)mxGetM(prhs[0]); 		// number of rows in the matrix A
	int numAColumns = (int)mxGetN(prhs[0]); 	// number of columns in the matrix A
	int numCRows;		// number of rows in the matrix C (you have to set this)
	int numCColumns;	// number of columns in the matrix C (you have to set this)
    	
		numCRows = numARows;
		numCColumns = numAColumns;
        
 if (numARows==1)
 {   
 printf("Attention, this is a row vector, please try Discrete Sine Transform in row wise \n");
 return;
 }
    mxInitGPU();
	float  * hostA ; // The A matrix
	hostA = (float *)mxGetData(prhs[0]);
    
    plhs[0] = mxCreateNumericMatrix(numCRows, numCColumns, mxSINGLE_CLASS, mxREAL);
    float  *pointer = (float*)mxGetPr(plhs[0]);
     
     if (strcmp (one,input_buf3) == 0)
    {
     
       CalculateTransformDSTColumnOne(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
	
    }
     if (strcmp (two,input_buf3) == 0)
    {
     
       CalculateTransformDSTColumnTwo(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
	
    }
     if (strcmp (three,input_buf3) == 0)
    {
     
       CalculateTransformDSTColumnThree(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
	
    }
     if (strcmp (four,input_buf3) == 0)
    {
     
       CalculateTransformDSTColumnFour(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
	
    }

      } 
    } 
  
     if (strcmp (row,input_buf4) == 0)
    {
    
    if (mxIsGPUArray(prhs[0])) {
		
    mxGPUArray const *A;
    mxGPUArray *B;
    float const *d_A;
    float *d_B;
    int numARows, numAColumns, numCRows,  numCColumns;
    mxInitGPU();
    hipError_t error;
    int devID = 0;
    error = hipGetDevice(&devID);
    hipDeviceProp_t deviceProp;
     error = hipGetDeviceProperties(&deviceProp, devID);
      if (error != hipSuccess)
      {
          printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
          exit(EXIT_FAILURE);
      }
    int TILEDIM = (deviceProp.major < 2) ? 16 : 32;

    A = mxGPUCreateFromMxArray(prhs[0]);
       if(mxGPUGetComplexity(A) != mxREAL){
       mxGPUDestroyGPUArray(A);
       mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Incorrect input arguments!, input matrix must be real %s\n");
       
       }
   const mwSize *dims;
   dims=mxGPUGetDimensions(A);
   numARows = (int)dims[0]; /* gets number of rows of A */
   numAColumns = (int)dims[1]; /* gets number of columns of A */
   size_t pivot_dimensA[2] = {numARows,numAColumns};
   mwSize NrOfDim=mxGPUGetNumberOfDimensions(A);
   
 if (numAColumns==1)
 {   
 printf("Attention, this is a column vector, please try Discrete Sine Transform in column wise \n");
 return;
 }
    numCRows = numARows;
	numCColumns = numAColumns;

    char const * const errId = "parallel:gpu:mexGPUExample:InvalidInput";
    char const * const errMsg = "Invalid input to MEX file, input(FIRST ARGUMENT) must be single precision (float).";
    
    if (mxGPUGetClassID(A) != mxSINGLE_CLASS) {
        mexErrMsgIdAndTxt(errId, errMsg);
    }

    d_A = (float const *)(mxGPUGetDataReadOnly(A));
    mxGPUDestroyGPUArray(A);
    B = mxGPUCreateGPUArray(NrOfDim, (mwSize*) pivot_dimensA, mxSINGLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    d_B = (float *)(mxGPUGetData(B));

    dim3 dimBlock;
	dim3 dimGrid;
    
     switch (TILEDIM){
        
        case 16:
            
     TILE_DIM= TILEDIM;
	 dimBlock.x=TILE_DIM;
	 dimBlock.y=TILE_DIM;
     dimBlock.z=1;
	 dimGrid.x = (numCColumns + dimBlock.x - 1) / dimBlock.x;
	 dimGrid.y = (numCRows + dimBlock.y - 1) / dimBlock.y;
 if (strcmp (one,input_buf3) == 0)
{
    
   DSTI_Row_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
}
 if (strcmp (two,input_buf3) == 0)
{
    
   DSTII_Row_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
}
 if (strcmp (three,input_buf3) == 0)
{
    
   DSTIII_Row_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
}
 if (strcmp (four,input_buf3) == 0)
{
    
   DSTIV_Row_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
}
   
    plhs[0] = mxGPUCreateMxArrayOnGPU(B);  
    mxGPUDestroyGPUArray(B);
     
      case 32:
            
     TILE_DIM= TILEDIM;
	 dimBlock.x=TILE_DIM;
	 dimBlock.y=TILE_DIM;
     dimBlock.z=1;
	 dimGrid.x = (numCColumns + dimBlock.x - 1) / dimBlock.x;
	 dimGrid.y = (numCRows + dimBlock.y - 1) / dimBlock.y;
     if (strcmp (one,input_buf3) == 0)
{
    
   DSTI_Row_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
}
 if (strcmp (two,input_buf3) == 0)
{
    
   DSTII_Row_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
}
 if (strcmp (three,input_buf3) == 0)
{
    
   DSTIII_Row_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
}
 if (strcmp (four,input_buf3) == 0)
{
    
   DSTIV_Row_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
}
     
    plhs[0] = mxGPUCreateMxArrayOnGPU(B);   
    mxGPUDestroyGPUArray(B);
    
      }
	}
 
    else if (!(mxIsGPUArray(prhs[0]))){
            
    if (mxGetClassID(prhs[0]) != mxSINGLE_CLASS) {
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Invalid input to MEX file, input(FIRST ARGUMENT) must be single precision (float).");
    } 
    if(mxIsComplex(prhs[0])){
       
       mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Incorrect input arguments!, input matrix must be real %s\n");
       
       }
       
  	int numARows = (int)mxGetM(prhs[0]); 		// number of rows in the matrix A
	int numAColumns = (int)mxGetN(prhs[0]); 	// number of columns in the matrix A
	
	int numCRows;		// number of rows in the matrix C (you have to set this)
	int numCColumns;	// number of columns in the matrix C (you have to set this)
	
	numCRows = numARows;
	numCColumns = numAColumns;
	float  * hostA ; // The A matrix
	
 if (numAColumns==1)
 {   
 printf("Attention, this is a column vector, please try Discrete Sine Transform in column wise \n");
 return;
 }
   
    mxInitGPU();
	hostA = (float *)mxGetData(prhs[0]);
    plhs[0] = mxCreateNumericMatrix(numCRows, numCColumns, mxSINGLE_CLASS, mxREAL);
    float  *pointer = (float*)mxGetPr(plhs[0]);
    
     if (strcmp (one,input_buf3) == 0)
{  
      CalculateTransformDSTRowOne(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
      
      } 
       if (strcmp (two,input_buf3) == 0)
{  
      CalculateTransformDSTRowTwo(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
      
      }
       if (strcmp (three,input_buf3) == 0)
{  
      CalculateTransformDSTRowThree(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
      
      }
       if (strcmp (four,input_buf3) == 0)
{  
      CalculateTransformDSTRowFour(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
      
      }
  
      } 
   }  
}

    if (strcmp (inverse,input_buf2) == 0)
     {
    if (strcmp (column,input_buf4) == 0)
     {      
    if (mxIsGPUArray(prhs[0])) {

    mxGPUArray const *A;
    mxGPUArray *B;
    float const *d_A;
    float *d_B;
    int numARows, numAColumns, numCRows,  numCColumns;
    mxInitGPU();
    hipError_t error;
    int devID = 0;
    error = hipGetDevice(&devID);
    hipDeviceProp_t deviceProp;
     error = hipGetDeviceProperties(&deviceProp, devID);
      if (error != hipSuccess)
      {
          printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
          exit(EXIT_FAILURE);
      }
    int TILEDIM = (deviceProp.major < 2) ? 16 : 32;

    A = mxGPUCreateFromMxArray(prhs[0]);
       if(mxGPUGetComplexity(A) != mxREAL){
       mxGPUDestroyGPUArray(A);
       mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Incorrect input arguments!, input matrix must be real %s\n");
       
       }
    const mwSize *dims;
    dims=mxGPUGetDimensions(A);
    numARows = (int)dims[0]; /* gets number of rows of A */
    numAColumns = (int)dims[1]; /* gets number of columns of A */
    size_t pivot_dimensA[2] = {numARows,numAColumns};
    mwSize NrOfDim=mxGPUGetNumberOfDimensions(A);
    numCRows = numARows;
    numCColumns = numAColumns;
   
  if (numARows==1)
 {   
 printf("Attention, this is a row vector, please try Inverse Discrete Sine Transform in row wise \n");
 return;
 }
 
    char const * const errId = "parallel:gpu:mexGPUExample:InvalidInput";
    char const * const errMsg = "Invalid input to MEX file, input(FIRST ARGUMENT) must be single precision (float).";

    if (mxGPUGetClassID(A) != mxSINGLE_CLASS) {
        mexErrMsgIdAndTxt(errId, errMsg);
    }
    d_A = (float const *)(mxGPUGetDataReadOnly(A));
    mxGPUDestroyGPUArray(A);
    B = mxGPUCreateGPUArray(NrOfDim, (mwSize*) pivot_dimensA, mxSINGLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);    
    d_B = (float *)(mxGPUGetData(B));
    
    dim3 dimBlock;
	dim3 dimGrid;
    
     switch (TILEDIM){
        
        case 16:
            
     TILE_DIM= TILEDIM;
	 dimBlock.x=TILE_DIM;
	 dimBlock.y=TILE_DIM;
     dimBlock.z=1;
	 dimGrid.x = (numCColumns + dimBlock.x - 1) / dimBlock.x;
	 dimGrid.y = (numCRows + dimBlock.y - 1) / dimBlock.y;
 if (strcmp (one,input_buf3) == 0)
{
    
   DSTI_Column_Inverse_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
}
  if (strcmp (two,input_buf3) == 0)
{
    
   DSTII_Column_Inverse_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
}
 if (strcmp (three,input_buf3) == 0)
{
    
   DSTIII_Column_Inverse_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
}
 if (strcmp (four,input_buf3) == 0)
{
    
   DSTIV_Column_Inverse_Kernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
}

    plhs[0] = mxGPUCreateMxArrayOnGPU(B);     
    mxGPUDestroyGPUArray(B);
    
      case 32:
            
     TILE_DIM= TILEDIM;
	 dimBlock.x=TILE_DIM;
	 dimBlock.y=TILE_DIM;
     dimBlock.z=1;
	 dimGrid.x = (numCColumns + dimBlock.x - 1) / dimBlock.x;
	 dimGrid.y = (numCRows + dimBlock.y - 1) / dimBlock.y; 
    
     if (strcmp (one,input_buf3) == 0)
{
    
   DSTI_Column_Inverse_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
}
  if (strcmp (two,input_buf3) == 0)
{
    
   DSTII_Column_Inverse_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
}
 if (strcmp (three,input_buf3) == 0)
{
    
   DSTIII_Column_Inverse_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
}
 if (strcmp (four,input_buf3) == 0)
{
    
   DSTIV_Column_Inverse_Kernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);
   
   
}

    plhs[0] = mxGPUCreateMxArrayOnGPU(B);         
    mxGPUDestroyGPUArray(B);
    
      }     
	}

    else if (!(mxIsGPUArray(prhs[0]))){
            
    if (mxGetClassID(prhs[0]) != mxSINGLE_CLASS) {
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Invalid input to MEX file, input(FIRST ARGUMENT) must be single precision (float).");
    }    
    if(mxIsComplex(prhs[0])){
       
       mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Incorrect input arguments!, input matrix must be real %s\n");
       
       }
       
  	int numARows = (int)mxGetM(prhs[0]); 		// number of rows in the matrix A
	int numAColumns = (int)mxGetN(prhs[0]); 	// number of columns in the matrix A
	int numCRows;		// number of rows in the matrix C (you have to set this)
	int numCColumns;	// number of columns in the matrix C (you have to set this)
    	
		numCRows = numARows;
		numCColumns = numAColumns;
        
 if (numARows==1)
 {   
 printf("Attention, this is a row vector, please try Inverse Discrete Sine Transform in row wise \n");
 return;
 }
    mxInitGPU();
	float  * hostA ; // The A matrix
	hostA = (float *)mxGetData(prhs[0]);
    
    plhs[0] = mxCreateNumericMatrix(numCRows, numCColumns, mxSINGLE_CLASS, mxREAL);
    float  *pointer = (float*)mxGetPr(plhs[0]);
    
     if (strcmp (one,input_buf3) == 0)
   {
  
       CalculateTransformDSTInverseColumnOne(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
	
   }
     if (strcmp (two,input_buf3) == 0)
   {
  
       CalculateTransformDSTInverseColumnTwo(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
	
   }
     if (strcmp (three,input_buf3) == 0)
   {
  
       CalculateTransformDSTInverseColumnThree(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
	
   }
     if (strcmp (four,input_buf3) == 0)
   {
  
       CalculateTransformDSTInverseColumnFour(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);
	
   }

   } 
} 

    if (strcmp (row,input_buf4) == 0)
    {
    
    if (mxIsGPUArray(prhs[0])) {

    mxGPUArray const *A;
    mxGPUArray *B;
    float const *d_A;
    float *d_B;
    int numARows, numAColumns, numCRows,  numCColumns;
    mxInitGPU();
    hipError_t error;
    int devID = 0;
    error = hipGetDevice(&devID);
    hipDeviceProp_t deviceProp;
     error = hipGetDeviceProperties(&deviceProp, devID);
    if (error != hipSuccess)
      {
          printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
          exit(EXIT_FAILURE);
      }
    int TILEDIM = (deviceProp.major < 2) ? 16 : 32;  

    A = mxGPUCreateFromMxArray(prhs[0]);
       if(mxGPUGetComplexity(A) != mxREAL){
       mxGPUDestroyGPUArray(A);
       mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Incorrect input arguments!, input matrix must be real %s\n");
       
       }
    const mwSize *dims;
    dims=mxGPUGetDimensions(A);
    numARows = (int)dims[0]; /* gets number of rows of A */
    numAColumns = (int)dims[1]; /* gets number of columns of A */
    size_t pivot_dimensA[2] = {numARows,numAColumns};
    mwSize NrOfDim=mxGPUGetNumberOfDimensions(A);
    
    if (numAColumns==1)
 {   
 printf("Attention, this is a column vector, please try Inverse Discrete Sine Transform in column wise \n");
 return;
 }
 
    numCRows = numARows;
	numCColumns = numAColumns;
    
    char const * const errId = "parallel:gpu:mexGPUExample:InvalidInput";
    char const * const errMsg = "Invalid input to MEX file, input(FIRST ARGUMENT) must be single precision (float).";

    if (mxGPUGetClassID(A) != mxSINGLE_CLASS) {
        mexErrMsgIdAndTxt(errId, errMsg);
    }
    
    d_A = (float const *)(mxGPUGetDataReadOnly(A));    
    mxGPUDestroyGPUArray(A);
    B = mxGPUCreateGPUArray(NrOfDim, (mwSize*) pivot_dimensA, mxSINGLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE); 
    d_B = (float *)(mxGPUGetData(B));
    
    dim3 dimBlock;
	dim3 dimGrid;
    
     switch (TILEDIM){
        
        case 16:
            
     TILE_DIM= TILEDIM;
	 dimBlock.x=TILE_DIM;
	 dimBlock.y=TILE_DIM;
     dimBlock.z=1;
	 dimGrid.x = (numCColumns + dimBlock.x - 1) / dimBlock.x;
	 dimGrid.y = (numCRows + dimBlock.y - 1) / dimBlock.y;
 if (strcmp (one,input_buf3) == 0)
{
   DSTI_Row__InverseKernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);

   }
 if (strcmp (two,input_buf3) == 0)
{
   DSTII_Row__InverseKernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);

   }
 if (strcmp (three,input_buf3) == 0)
{
   DSTIII_Row__InverseKernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);

   }
 if (strcmp (four,input_buf3) == 0)
{
   DSTIV_Row__InverseKernel_GPUA <16> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);

   }

    plhs[0] = mxGPUCreateMxArrayOnGPU(B);        
    mxGPUDestroyGPUArray(B);
    
          case 32:
            
     TILE_DIM= TILEDIM;
	 dimBlock.x=TILE_DIM;
	 dimBlock.y=TILE_DIM;
     dimBlock.z=1;
	 dimGrid.x = (numCColumns + dimBlock.x - 1) / dimBlock.x;
	 dimGrid.y = (numCRows + dimBlock.y - 1) / dimBlock.y;
    
    if (strcmp (one,input_buf3) == 0)
{
   DSTI_Row__InverseKernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);

   }
 if (strcmp (two,input_buf3) == 0)
{
   DSTII_Row__InverseKernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);

   }
 if (strcmp (three,input_buf3) == 0)
{
   DSTIII_Row__InverseKernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);

   }
 if (strcmp (four,input_buf3) == 0)
{
   DSTIV_Row__InverseKernel_GPUA <32> << <dimGrid, dimBlock >> >(d_A, d_B, numARows, numAColumns, numCRows, numCColumns);

   }

    plhs[0] = mxGPUCreateMxArrayOnGPU(B);    
    mxGPUDestroyGPUArray(B); 
        
      }     
	}
 
    else if (!(mxIsGPUArray(prhs[0]))){
            
    if (mxGetClassID(prhs[0]) != mxSINGLE_CLASS) {
         mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Invalid input to MEX file, input(FIRST ARGUMENT) must be single precision (float).");
    }  
    if(mxIsComplex(prhs[0])){
       
       mexErrMsgIdAndTxt( "MATLAB:mexatexit:invalidInput",
                "Incorrect input arguments!, input matrix must be real %s\n");
       
       }    
    
  	int numARows = (int)mxGetM(prhs[0]); 		// number of rows in the matrix A
	int numAColumns = (int)mxGetN(prhs[0]); 	// number of columns in the matrix A
	
	int numCRows;		// number of rows in the matrix C (you have to set this)
	int numCColumns;	// number of columns in the matrix C (you have to set this)
		
    if (numAColumns==1)
    {   
    printf("Attention, this is a column vector, please try Inverse Discrete Sine Transform in column wise \n");
    return;
    }
    mxInitGPU();
    numCRows = numARows;
	numCColumns = numAColumns;
	float  * hostA ; // The A matrix
	hostA = (float *)mxGetData(prhs[0]);
    
    plhs[0] = mxCreateNumericMatrix(numCRows, numCColumns, mxSINGLE_CLASS, mxREAL);
    float  *pointer =(float*) mxGetPr(plhs[0]);
    
     if (strcmp (one,input_buf3) == 0)
{
      CalculateTransformDSTInverseRowOne(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);  
 } 
     if (strcmp (two,input_buf3) == 0)
{
      CalculateTransformDSTInverseRowTwo(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);  
 } 
     if (strcmp (three,input_buf3) == 0)
{
      CalculateTransformDSTInverseRowThree(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);  
 } 
     if (strcmp (four,input_buf3) == 0)
{
      CalculateTransformDSTInverseRowFour(hostA, pointer, numARows, numAColumns, numCRows, numCColumns);  
 } 

      }
    }
  } 
}

}
